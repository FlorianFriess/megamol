#include "hip/hip_runtime.h"
/*
* VolumeMeshRenderer-sort.cu
*
* Copyright (C) 2012 by Universitaet Stuttgart (VIS).
* Alle Rechte vorbehalten.
*/
#ifndef MEGAMOLPROTEIN_VOLUMEMESHRENDERER_SORT_CU_INCLUDED
#define MEGAMOLPROTEIN_VOLUMEMESHRENDERER_SORT_CU_INCLUDED

#include "VolumeMeshRenderer.cuh"
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/unique.h>
#include <thrust/iterator/constant_iterator.h>
#include <cutil_math.h>
#include <hip/device_functions.h>

/*
 * Note:
 * This is a VERY slow compiling piece of code (because of thrust::sort).
 * Its in an extra file so that small changes on other parts of VolumeMeshRenderer
 * wont lead to huge compilation times.
 */
extern "C"
hipError_t CentroidReduce(uint* centroidLabelsCount, uint* centroidLabels, float4* centroidSums, uint* centroidCounts, uint* vertexLabels, float4* vertices, uint vertexCount)
{
    uint* vertexLabelsEnd = vertexLabels + vertexCount;
    // Sort (reduce needs consecutive keys).
    thrust::sort_by_key(thrust::device_ptr<uint>(vertexLabels), thrust::device_ptr<uint>(vertexLabelsEnd), 
        thrust::device_ptr<float4>(vertices));
    // Count.
    thrust::reduce_by_key(thrust::device_ptr<uint>(vertexLabels), thrust::device_ptr<uint>(vertexLabelsEnd),
        thrust::constant_iterator<uint>(1), thrust::device_ptr<uint>(centroidLabels),
        thrust::device_ptr<uint>(centroidCounts));
    // Sum.
    uint* centroidLabelsEnd = thrust::reduce_by_key(thrust::device_ptr<uint>(vertexLabels), thrust::device_ptr<uint>(vertexLabelsEnd),
        thrust::device_ptr<float4>(vertices), thrust::device_ptr<uint>(centroidLabels),
        thrust::device_ptr<float4>(centroidSums)).first.get();
    if (centroidLabelsEnd >= centroidLabels) {
        *centroidLabelsCount = centroidLabelsEnd - centroidLabels;
    } else{
        *centroidLabelsCount = 0;
    }
    return hipGetLastError();
}

extern "C"
hipError_t ComputeFeatureBBox( float* fBBoxMinX, float* fBBoxMinY, float* fBBoxMinZ, float* fBBoxMaxX, float* fBBoxMaxY, float* fBBoxMaxZ,
        uint* triaLabelsMinX, uint* triaLabelsMinY, uint* triaLabelsMinZ, uint* triaLabelsMaxX, uint* triaLabelsMaxY, uint* triaLabelsMaxZ,
        uint triaCount) {
    // compute the bboxes of all features
    
    // sort the min values
    thrust::sort_by_key( thrust::device_ptr<float>(fBBoxMinX), thrust::device_ptr<float>(fBBoxMinX + triaCount), thrust::device_ptr<uint>(triaLabelsMinX));
    thrust::sort_by_key( thrust::device_ptr<float>(fBBoxMinY), thrust::device_ptr<float>(fBBoxMinY + triaCount), thrust::device_ptr<uint>(triaLabelsMinY));
    thrust::sort_by_key( thrust::device_ptr<float>(fBBoxMinZ), thrust::device_ptr<float>(fBBoxMinZ + triaCount), thrust::device_ptr<uint>(triaLabelsMinZ));
    // sort the max values
    thrust::sort_by_key( thrust::device_ptr<float>(fBBoxMaxX), thrust::device_ptr<float>(fBBoxMaxX + triaCount), thrust::device_ptr<uint>(triaLabelsMaxX), thrust::greater<float>());
    thrust::sort_by_key( thrust::device_ptr<float>(fBBoxMaxY), thrust::device_ptr<float>(fBBoxMaxY + triaCount), thrust::device_ptr<uint>(triaLabelsMaxY), thrust::greater<float>());
    thrust::sort_by_key( thrust::device_ptr<float>(fBBoxMaxZ), thrust::device_ptr<float>(fBBoxMaxZ + triaCount), thrust::device_ptr<uint>(triaLabelsMaxZ), thrust::greater<float>());
    // sort the min values by label
    thrust::stable_sort_by_key( thrust::device_ptr<uint>(triaLabelsMinX), thrust::device_ptr<uint>(triaLabelsMinX + triaCount), thrust::device_ptr<float>(fBBoxMinX));
    thrust::stable_sort_by_key( thrust::device_ptr<uint>(triaLabelsMinY), thrust::device_ptr<uint>(triaLabelsMinY + triaCount), thrust::device_ptr<float>(fBBoxMinY));
    thrust::stable_sort_by_key( thrust::device_ptr<uint>(triaLabelsMinZ), thrust::device_ptr<uint>(triaLabelsMinZ + triaCount), thrust::device_ptr<float>(fBBoxMinZ));
    // sort the max values by label
    thrust::stable_sort_by_key( thrust::device_ptr<uint>(triaLabelsMaxX), thrust::device_ptr<uint>(triaLabelsMaxX + triaCount), thrust::device_ptr<float>(fBBoxMaxX));
    thrust::stable_sort_by_key( thrust::device_ptr<uint>(triaLabelsMaxY), thrust::device_ptr<uint>(triaLabelsMaxY + triaCount), thrust::device_ptr<float>(fBBoxMaxY));
    thrust::stable_sort_by_key( thrust::device_ptr<uint>(triaLabelsMaxZ), thrust::device_ptr<uint>(triaLabelsMaxZ + triaCount), thrust::device_ptr<float>(fBBoxMaxZ));
    // get the min/max x/y/z-value per feature
    thrust::unique_by_key( thrust::device_ptr<uint>(triaLabelsMinX), thrust::device_ptr<uint>(triaLabelsMinX + triaCount), thrust::device_ptr<float>(fBBoxMinX));
    thrust::unique_by_key( thrust::device_ptr<uint>(triaLabelsMinY), thrust::device_ptr<uint>(triaLabelsMinY + triaCount), thrust::device_ptr<float>(fBBoxMinY));
    thrust::unique_by_key( thrust::device_ptr<uint>(triaLabelsMinZ), thrust::device_ptr<uint>(triaLabelsMinZ + triaCount), thrust::device_ptr<float>(fBBoxMinZ));
    thrust::unique_by_key( thrust::device_ptr<uint>(triaLabelsMaxX), thrust::device_ptr<uint>(triaLabelsMaxX + triaCount), thrust::device_ptr<float>(fBBoxMaxX));
    thrust::unique_by_key( thrust::device_ptr<uint>(triaLabelsMaxY), thrust::device_ptr<uint>(triaLabelsMaxY + triaCount), thrust::device_ptr<float>(fBBoxMaxY));
    thrust::unique_by_key( thrust::device_ptr<uint>(triaLabelsMaxZ), thrust::device_ptr<uint>(triaLabelsMaxZ + triaCount), thrust::device_ptr<float>(fBBoxMaxZ));

    return hipGetLastError();
}

extern "C"
hipError_t SortPrevTetraLabel( int2* labelPair, uint tetrahedronCount, int &labelCount) {
    thrust::sort( thrust::device_ptr<int2>(labelPair), thrust::device_ptr<int2>(labelPair + tetrahedronCount), lessInt2X());
    const int numberOfUniqueValues = thrust::unique( thrust::device_ptr<int2>(labelPair), thrust::device_ptr<int2>(labelPair + tetrahedronCount), equalInt2()) - thrust::device_ptr<int2>(labelPair);
    labelCount = numberOfUniqueValues;

    return hipGetLastError();
}

#endif // MEGAMOLPROTEIN_VOLUMEMESHRENDERER_SORT_CU_INCLUDED
