#include "hip/hip_runtime.h"
/*
* VolumeMeshRenderer-sort.cu
*
* Copyright (C) 2012 by Universitaet Stuttgart (VIS).
* Alle Rechte vorbehalten.
*/
#ifndef MEGAMOLPROTEIN_VOLUMEMESHRENDERER_SORT_CU_INCLUDED
#define MEGAMOLPROTEIN_VOLUMEMESHRENDERER_SORT_CU_INCLUDED

#include "VolumeMeshRenderer.cuh"
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/unique.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/scan.h>
#include <thrust/sequence.h>
#include "cuda_helper.h"

/*
 * Note:
 * This is a VERY slow compiling piece of code (because of thrust::sort).
 * Its in an extra file so that small changes on other parts of VolumeMeshRenderer
 * wont lead to huge compilation times.
 */
extern "C"
hipError_t CentroidReduce(uint* centroidLabelsCount, uint* centroidLabels, float4* centroidSums, uint* centroidCounts, uint* vertexLabels, float4* vertices, uint vertexCount)
{
    uint* vertexLabelsEnd = vertexLabels + vertexCount;
    // Sort (reduce needs consecutive keys).
    thrust::sort_by_key(thrust::device_ptr<uint>(vertexLabels), thrust::device_ptr<uint>(vertexLabelsEnd), 
        thrust::device_ptr<float4>(vertices));
    // Count.
    thrust::reduce_by_key(thrust::device_ptr<uint>(vertexLabels), thrust::device_ptr<uint>(vertexLabelsEnd),
        thrust::constant_iterator<uint>(1), thrust::device_ptr<uint>(centroidLabels),
        thrust::device_ptr<uint>(centroidCounts));
    // Sum.
    uint* centroidLabelsEnd = thrust::reduce_by_key(thrust::device_ptr<uint>(vertexLabels), thrust::device_ptr<uint>(vertexLabelsEnd),
        thrust::device_ptr<float4>(vertices), thrust::device_ptr<uint>(centroidLabels),
        thrust::device_ptr<float4>(centroidSums)).first.get();
    if (centroidLabelsEnd >= centroidLabels) {
        *centroidLabelsCount = centroidLabelsEnd - centroidLabels;
    } else{
        *centroidLabelsCount = 0;
    }
    return hipGetLastError();
}

extern "C"
hipError_t ComputeFeatureBBox( float* fBBoxMinX, float* fBBoxMinY, float* fBBoxMinZ, float* fBBoxMaxX, float* fBBoxMaxY, float* fBBoxMaxZ,
        uint* triaLabelsMinX, uint* triaLabelsMinY, uint* triaLabelsMinZ, uint* triaLabelsMaxX, uint* triaLabelsMaxY, uint* triaLabelsMaxZ,
        uint triaCount) {
    // compute the bboxes of all features
    
    // sort the min values
    thrust::sort_by_key( thrust::device_ptr<float>(fBBoxMinX), thrust::device_ptr<float>(fBBoxMinX + triaCount), thrust::device_ptr<uint>(triaLabelsMinX));
    thrust::sort_by_key( thrust::device_ptr<float>(fBBoxMinY), thrust::device_ptr<float>(fBBoxMinY + triaCount), thrust::device_ptr<uint>(triaLabelsMinY));
    thrust::sort_by_key( thrust::device_ptr<float>(fBBoxMinZ), thrust::device_ptr<float>(fBBoxMinZ + triaCount), thrust::device_ptr<uint>(triaLabelsMinZ));
    // sort the max values
    thrust::sort_by_key( thrust::device_ptr<float>(fBBoxMaxX), thrust::device_ptr<float>(fBBoxMaxX + triaCount), thrust::device_ptr<uint>(triaLabelsMaxX), thrust::greater<float>());
    thrust::sort_by_key( thrust::device_ptr<float>(fBBoxMaxY), thrust::device_ptr<float>(fBBoxMaxY + triaCount), thrust::device_ptr<uint>(triaLabelsMaxY), thrust::greater<float>());
    thrust::sort_by_key( thrust::device_ptr<float>(fBBoxMaxZ), thrust::device_ptr<float>(fBBoxMaxZ + triaCount), thrust::device_ptr<uint>(triaLabelsMaxZ), thrust::greater<float>());
    // sort the min values by label
    thrust::stable_sort_by_key( thrust::device_ptr<uint>(triaLabelsMinX), thrust::device_ptr<uint>(triaLabelsMinX + triaCount), thrust::device_ptr<float>(fBBoxMinX));
    thrust::stable_sort_by_key( thrust::device_ptr<uint>(triaLabelsMinY), thrust::device_ptr<uint>(triaLabelsMinY + triaCount), thrust::device_ptr<float>(fBBoxMinY));
    thrust::stable_sort_by_key( thrust::device_ptr<uint>(triaLabelsMinZ), thrust::device_ptr<uint>(triaLabelsMinZ + triaCount), thrust::device_ptr<float>(fBBoxMinZ));
    // sort the max values by label
    thrust::stable_sort_by_key( thrust::device_ptr<uint>(triaLabelsMaxX), thrust::device_ptr<uint>(triaLabelsMaxX + triaCount), thrust::device_ptr<float>(fBBoxMaxX));
    thrust::stable_sort_by_key( thrust::device_ptr<uint>(triaLabelsMaxY), thrust::device_ptr<uint>(triaLabelsMaxY + triaCount), thrust::device_ptr<float>(fBBoxMaxY));
    thrust::stable_sort_by_key( thrust::device_ptr<uint>(triaLabelsMaxZ), thrust::device_ptr<uint>(triaLabelsMaxZ + triaCount), thrust::device_ptr<float>(fBBoxMaxZ));
    // get the min/max x/y/z-value per feature
    thrust::unique_by_key( thrust::device_ptr<uint>(triaLabelsMinX), thrust::device_ptr<uint>(triaLabelsMinX + triaCount), thrust::device_ptr<float>(fBBoxMinX));
    thrust::unique_by_key( thrust::device_ptr<uint>(triaLabelsMinY), thrust::device_ptr<uint>(triaLabelsMinY + triaCount), thrust::device_ptr<float>(fBBoxMinY));
    thrust::unique_by_key( thrust::device_ptr<uint>(triaLabelsMinZ), thrust::device_ptr<uint>(triaLabelsMinZ + triaCount), thrust::device_ptr<float>(fBBoxMinZ));
    thrust::unique_by_key( thrust::device_ptr<uint>(triaLabelsMaxX), thrust::device_ptr<uint>(triaLabelsMaxX + triaCount), thrust::device_ptr<float>(fBBoxMaxX));
    thrust::unique_by_key( thrust::device_ptr<uint>(triaLabelsMaxY), thrust::device_ptr<uint>(triaLabelsMaxY + triaCount), thrust::device_ptr<float>(fBBoxMaxY));
    thrust::unique_by_key( thrust::device_ptr<uint>(triaLabelsMaxZ), thrust::device_ptr<uint>(triaLabelsMaxZ + triaCount), thrust::device_ptr<float>(fBBoxMaxZ));

    return hipGetLastError();
}

extern "C"
hipError_t SortPrevTetraLabel( int2* labelPair, uint tetrahedronCount, int &labelCount) {
    thrust::sort( thrust::device_ptr<int2>(labelPair), thrust::device_ptr<int2>(labelPair + tetrahedronCount), lessInt2X());
    const int numberOfUniqueValues = thrust::unique( thrust::device_ptr<int2>(labelPair), thrust::device_ptr<int2>(labelPair + tetrahedronCount), equalInt2()) - thrust::device_ptr<int2>(labelPair);
    labelCount = numberOfUniqueValues;

    return hipGetLastError();
}

extern "C"
hipError_t TriangleVerticesToIndexList( float4* featureVertices, uint* featureVertexIdx, uint* featureVertexCnt, uint* featureVertexStartIdx, uint* featureVertexIdxNew, uint fLength, uint &vertexCnt) {
    thrust::sequence( thrust::device_ptr<uint>(featureVertexIdx), thrust::device_ptr<uint>(featureVertexIdx + fLength));
    thrust::fill_n( thrust::device_ptr<uint>(featureVertexCnt), fLength, 1);
    thrust::stable_sort_by_key( thrust::device_ptr<float4>(featureVertices), 
        thrust::device_ptr<float4>(featureVertices + fLength), 
        thrust::device_ptr<uint>(featureVertexIdx), less_float4());
    float4* new_end = thrust::reduce_by_key( thrust::device_ptr<float4>(featureVertices), 
        thrust::device_ptr<float4>(featureVertices + fLength), 
        thrust::device_ptr<uint>(featureVertexCnt),
        thrust::device_ptr<float4>(featureVertices), 
        thrust::device_ptr<uint>(featureVertexCnt), equal_float4()).first.get();
    vertexCnt = (new_end - featureVertices);
    thrust::exclusive_scan( thrust::device_ptr<uint>(featureVertexCnt), thrust::device_ptr<uint>(featureVertexCnt + vertexCnt), thrust::device_ptr<uint>(featureVertexStartIdx));
    WriteTriangleVertexIndexList( featureVertexIdx, featureVertexCnt, featureVertexStartIdx, featureVertexIdxNew, fLength, vertexCnt);

    return hipGetLastError();
}

#endif // MEGAMOLPROTEIN_VOLUMEMESHRENDERER_SORT_CU_INCLUDED
