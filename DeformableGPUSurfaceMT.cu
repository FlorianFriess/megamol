#include "hip/hip_runtime.h"
//
// DeformableGPUSurfaceMT.cpp
//
// Copyright (C) 2013 by University of Stuttgart (VISUS).
// All rights reserved.
//
// Created on : Sep 17, 2013
// Author     : scharnkn
//

#include "stdafx.h"

#include <glh/glh_extensions.h>
#include "DeformableGPUSurfaceMT.h"
#ifdef WITH_CUDA
#include "ogl_error_check.h"
#include "cuda_error_check.h"
//#include "ComparativeSurfacePotentialRenderer.cuh"
//#include "ComparativeSurfacePotentialRenderer_inline_device_functions.cuh"
#include "HostArr.h"
#include "DiffusionSolver.h"
//#include "constantGridParams.cuh"
#include "CUDAGrid.cuh"
#include "cuda_helper.h"

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#define USE_TIMER

using namespace megamol;
using namespace megamol::protein;


/**
 * 'Safe' inverse sqrt, that prevents dividing by zero
 *
 * @param x The input value
 * @return The inverse sqrt if x>0, 0.0 otherwise
 */
inline __device__ float safeRsqrtf(float x) {
    if (x > 0.0) {
        return 1.0f/sqrtf(x);
    } else {
        return 0.0f;
    }
}

/**
 * 'Safe' normalize function for float3 that uses safe rsqrt
 *
 * @param v The input vector to be normalized
 * @return The normalized vector v
 */
inline __device__ float safeInvLength(float3 v) {
    return safeRsqrtf(dot(v, v));
}

/**
 * 'Safe' normalize function for float2 that uses safe rsqrt
 *
 * @param v The input vector to be normalized
 * @return The normalized vector v
 */
inline __device__ float2 safeNormalize(float2 v) {
    float invLen = safeRsqrtf(dot(v, v));
    return v * invLen;
}

/**
 * 'Safe' normalize function for float3 that uses safe rsqrt
 *
 * @param v The input vector to be normalized
 * @return The normalized vector v
 */
inline __device__ float3 safeNormalize(float3 v) {
    float invLen = safeRsqrtf(dot(v, v));
    return v * invLen;
}


////////////////////////////////////////////////////////////////////////////////
//  Inline device functions                                                   //
////////////////////////////////////////////////////////////////////////////////

/**
 * @return Returns the thread index based on the current CUDA grid dimensions
 */
inline __device__ uint GetThreadIdx() {
    return __umul24(__umul24(blockIdx.y, gridDim.x) + blockIdx.x, blockDim.x) +
            threadIdx.x;
}


////////////////////////////////////////////////////////////////////////////////
//  Global device functions                                                   //
////////////////////////////////////////////////////////////////////////////////


/**
 * Computes the gradient of a given scalar field using central differences.
 * Border areas are omitted.
 *
 * @param[out] grad_D  The gradient field
 * @param[in]  field_D The scalar field
 */
__global__ void calcVolGradient_D(float4 *grad_D, float *field_D) {

    const uint idx = ::GetThreadIdx();

    // Get grid coordinates
    uint3 gridCoord = make_uint3(
            idx % gridSize_D.x,
            (idx / gridSize_D.x) % gridSize_D.y,
            (idx / gridSize_D.x) / gridSize_D.y);

    // Omit border cells (gradient remains zero)
    if (gridCoord.x == 0) return;
    if (gridCoord.y == 0) return;
    if (gridCoord.z == 0) return;
    if (gridCoord.x >= gridSize_D.x - 1) return;
    if (gridCoord.y >= gridSize_D.y - 1) return;
    if (gridCoord.z >= gridSize_D.z - 1) return;

    float3 grad;

    grad.x =
            field_D[GetPosIdxByGridCoords(make_uint3(gridCoord.x+1, gridCoord.y+0, gridCoord.z+0))]-
            field_D[GetPosIdxByGridCoords(make_uint3(gridCoord.x-1, gridCoord.y+0, gridCoord.z+0))];

    grad.y =
            field_D[GetPosIdxByGridCoords(make_uint3(gridCoord.x, gridCoord.y+1, gridCoord.z+0))]-
            field_D[GetPosIdxByGridCoords(make_uint3(gridCoord.x, gridCoord.y-1, gridCoord.z+0))];

    grad.z =
            field_D[GetPosIdxByGridCoords(make_uint3(gridCoord.x, gridCoord.y+0, gridCoord.z+1))]-
            field_D[GetPosIdxByGridCoords(make_uint3(gridCoord.x, gridCoord.y+0, gridCoord.z-1))];

    grad = safeNormalize(grad);

    grad_D[idx].x = grad.x;
    grad_D[idx].y = grad.y;
    grad_D[idx].z = grad.z;
}


/**
 * Computes the gradient of a given scalar field using central differences.
 * Border areas are omitted.
 *
 * @param[out] grad_D  The gradient field
 * @param[in]  field_D The scalar field
 * @param[in]  field_D The distance field
 */
__global__ void calcVolGradientWithDistField_D(float4 *grad_D, float *field_D,
        float *distField_D, float minDist, float isovalue) {

    const uint idx = ::GetThreadIdx();

    // Get grid coordinates
    uint3 gridCoord = ::GetGridCoordsByPosIdx(idx);

    // Omit border cells (gradient remains zero)
    if (gridCoord.x == 0) return;
    if (gridCoord.y == 0) return;
    if (gridCoord.z == 0) return;
    if (gridCoord.x >= gridSize_D.x - 1) return;
    if (gridCoord.y >= gridSize_D.y - 1) return;
    if (gridCoord.z >= gridSize_D.z - 1) return;

    float distSample = ::SampleFieldAt_D<float>(gridCoord, distField_D);
    float volSample = ::SampleFieldAt_D<float>(gridCoord, field_D);

    float3 grad = make_float3(0.0, 0.0, 0.0);

    if (distSample > minDist) {
        grad.x =
                distField_D[GetPosIdxByGridCoords(make_uint3(gridCoord.x+1, gridCoord.y+0, gridCoord.z+0))]-
                distField_D[GetPosIdxByGridCoords(make_uint3(gridCoord.x+0, gridCoord.y+0, gridCoord.z+0))];

        grad.y =
                distField_D[GetPosIdxByGridCoords(make_uint3(gridCoord.x, gridCoord.y+1, gridCoord.z+0))]-
                distField_D[GetPosIdxByGridCoords(make_uint3(gridCoord.x, gridCoord.y+0, gridCoord.z+0))];

        grad.z =
                distField_D[GetPosIdxByGridCoords(make_uint3(gridCoord.x, gridCoord.y+0, gridCoord.z+1))]-
                distField_D[GetPosIdxByGridCoords(make_uint3(gridCoord.x, gridCoord.y+0, gridCoord.z+0))];

        if (volSample < isovalue) {
            grad.x *= -1.0;
            grad.y *= -1.0;
            grad.z *= -1.0;
        }

    } else {

        grad.x =
                field_D[GetPosIdxByGridCoords(make_uint3(gridCoord.x+1, gridCoord.y+0, gridCoord.z+0))]-
                field_D[GetPosIdxByGridCoords(make_uint3(gridCoord.x+0, gridCoord.y+0, gridCoord.z+0))];

        grad.y =
                field_D[GetPosIdxByGridCoords(make_uint3(gridCoord.x, gridCoord.y+1, gridCoord.z+0))]-
                field_D[GetPosIdxByGridCoords(make_uint3(gridCoord.x, gridCoord.y+0, gridCoord.z+0))];

        grad.z =
                field_D[GetPosIdxByGridCoords(make_uint3(gridCoord.x, gridCoord.y+0, gridCoord.z+1))]-
                field_D[GetPosIdxByGridCoords(make_uint3(gridCoord.x, gridCoord.y+0, gridCoord.z+0))];
    }


    grad = safeNormalize(grad);

    grad_D[idx].x = grad.x;
    grad_D[idx].y = grad.y;
    grad_D[idx].z = grad.z;
}


/**
 * Computes a distance field based on the vertex positions.
 *
 * @param[in]  vertexPos_D The vertex data buffer (device memory)
 * @param[out] distField_D The distance field buffer (device memory)
 * @param[in]  vertexCnt   The number of vertices
 * @param[in]  dataArrOffs The vertex position offset for the vertex data buffer
 * @param[in]  dataArrSize The stride of the vertex data buffer
 */
__global__ void computeDistField_D(
        float *vertexPos_D,
        float *distField_D,
        uint vertexCnt,
        uint dataArrOffs,
        uint dataArrSize) {

    // TODO This is very slow since it basically bruteforces all vertex
    //      positions and stores the distance to the nearest one.

    const uint idx = GetThreadIdx();

    if (idx >= gridSize_D.x*gridSize_D.y*gridSize_D.z) {
        return;
    }

    // Get world space position of gridPoint
    uint3 gridCoords = GetGridCoordsByPosIdx(idx);
    float3 latticePos = TransformToWorldSpace(make_float3(
            gridCoords.x,
            gridCoords.y,
            gridCoords.z));

    // Loop through all vertices to find minimal distance
    float3 pos = make_float3(vertexPos_D[0], vertexPos_D[1], vertexPos_D[2]);
    float len;
    len = (latticePos.x-pos.x)*(latticePos.x-pos.x)+
          (latticePos.y-pos.y)*(latticePos.y-pos.y)+
          (latticePos.z-pos.z)*(latticePos.z-pos.z);
    float dist2 = len;


    for (uint i = 0; i < vertexCnt; ++i) {
        pos = make_float3(
                vertexPos_D[dataArrSize*i+dataArrOffs+0],
                vertexPos_D[dataArrSize*i+dataArrOffs+1],
                vertexPos_D[dataArrSize*i+dataArrOffs+2]);
        len = (latticePos.x-pos.x)*(latticePos.x-pos.x)+
              (latticePos.y-pos.y)*(latticePos.y-pos.y)+
              (latticePos.z-pos.z)*(latticePos.z-pos.z);
        dist2 = min(dist2, len);
    }

    distField_D[idx] = sqrt(dist2);
}


/**
 * Writes a flag for every vertex that is adjacent to a corrupt triangles.
 *
 * @param[in,out] vertexData_D              The buffer with the vertex data
 * @param[in]     vertexDataStride          The stride for the vertex data
 *                                          buffer
 * @param[in]     vertexDataOffsPos         The position offset in the vertex
 *                                          data buffer
 * @param[in]     vertexDataOffsCorruptFlag The corruption flag offset in the
 *                                          vertex data buffer
 * @param[in]     triangleVtxIdx_D          Array with triangle vertex indices
 * @param[in]     volume_D                  The target volume defining the
 *                               iso-surface
 * @param[in]     externalForcesScl_D       Array with the scale factor for the external force
 * @param[in]     triangleCnt               The number of triangles
 * @param[in]     minDispl                  Minimum force scale to keep going
 * @param[in]     isoval                    The iso-value defining the iso-surface
 *
 * TODO
 */
__global__ void FlagCorruptTriangleVertices_D(
        float *vertexFlag_D,
        float *vertexData_D,
        uint vertexDataStride,
        uint vertexDataOffsPos,
        uint vertexDataOffsNormal,
        uint *triangleVtxIdx_D,
        float *targetVol_D,
        uint triangleCnt,
        float isoval) {

    const uint idx = ::GetThreadIdx();
    if (idx >= triangleCnt) {
        return;
    }

    /* Alternative 1: Sample volume at triangle midpoint */

    const uint baseIdx0 = vertexDataStride*triangleVtxIdx_D[3*idx+0];
    const uint baseIdx1 = vertexDataStride*triangleVtxIdx_D[3*idx+1];
    const uint baseIdx2 = vertexDataStride*triangleVtxIdx_D[3*idx+2];
    const float3 p0 = make_float3(vertexData_D[baseIdx0+vertexDataOffsPos+0],
                                  vertexData_D[baseIdx0+vertexDataOffsPos+1],
                                  vertexData_D[baseIdx0+vertexDataOffsPos+2]);
    const float3 p1 = make_float3(vertexData_D[baseIdx1+vertexDataOffsPos+0],
                                  vertexData_D[baseIdx1+vertexDataOffsPos+1],
                                  vertexData_D[baseIdx1+vertexDataOffsPos+2]);
    const float3 p2 = make_float3(vertexData_D[baseIdx2+vertexDataOffsPos+0],
                                  vertexData_D[baseIdx2+vertexDataOffsPos+1],
                                  vertexData_D[baseIdx2+vertexDataOffsPos+2]);
    // Sample volume at midpoint
    const float3 midPoint = (p0+p1+p2)/3.0;
    const float volSampleMidPoint = ::SampleFieldAtPosTricub_D<float>(midPoint, targetVol_D);
    float flag = float(::fabs(volSampleMidPoint-isoval) > 0.3);
    vertexFlag_D[triangleVtxIdx_D[3*idx+0]] = flag;
    vertexFlag_D[triangleVtxIdx_D[3*idx+1]] = flag;
    vertexFlag_D[triangleVtxIdx_D[3*idx+2]] = flag;

    /* Alternative 2: calc variance of angle between normals */

//    const uint baseIdx0 = vertexDataStride*triangleVtxIdx_D[3*idx+0];
//    const uint baseIdx1 = vertexDataStride*triangleVtxIdx_D[3*idx+1];
//    const uint baseIdx2 = vertexDataStride*triangleVtxIdx_D[3*idx+2];
//    const float3 n0 = make_float3(vertexData_D[baseIdx0+vertexDataOffsNormal+0],
//                                  vertexData_D[baseIdx0+vertexDataOffsNormal+1],
//                                  vertexData_D[baseIdx0+vertexDataOffsNormal+2]);
//    const float3 n1 = make_float3(vertexData_D[baseIdx1+vertexDataOffsNormal+0],
//                                  vertexData_D[baseIdx1+vertexDataOffsNormal+1],
//                                  vertexData_D[baseIdx1+vertexDataOffsNormal+2]);
//    const float3 n2 = make_float3(vertexData_D[baseIdx2+vertexDataOffsNormal+0],
//                                  vertexData_D[baseIdx2+vertexDataOffsNormal+1],
//                                  vertexData_D[baseIdx2+vertexDataOffsNormal+2]);
//    // Sample volume at midpoint
//    const float3 avgNormal = (n0+n1+n2)/3.0;
//    float dot0 = clamp(dot(n0, avgNormal), 0.0, 1.0);
//    float dot1 = clamp(dot(n1, avgNormal), 0.0, 1.0);
//    float dot2 = clamp(dot(n2, avgNormal), 0.0, 1.0);
//    float maxDot = max(dot0, max(dot1, dot2));
//    float flag = float(maxDot > 0.9);
//    vertexFlag_D[triangleVtxIdx_D[3*idx+0]] = flag;
//    vertexFlag_D[triangleVtxIdx_D[3*idx+1]] = flag;
//    vertexFlag_D[triangleVtxIdx_D[3*idx+2]] = flag;
}


/**
 * Initializes the scale factor for the external forces with either -1.0 (if the
 * starting position of the vector is inside the isosurface, or 1.0 (vice
 * versa).
 *
 * @param[in] arr_D       The external forces data buffer
 * @param[in] volume_D    The volume the isosurface is extracted from
 * @param[in] vertexPos_D The vertex data buffer
 * @param[in] nElements   The number of vertices
 * @param[in] isoval      The isovalue that defines the isosurface
 * @param[in] dataArrOffs The offset for vertex positions in the vertex
 *                        data buffer
 * @param[in] dataArrSize The stride of the vertex data buffer
 */
__global__ void initExternalForceScl_D (
        float *arr_D,
        float *volume_D,
        float *vertexPos_D,
        uint nElements,
        float isoval,
        uint dataArrOffs,
        uint dataArrSize) {

    const uint idx = GetThreadIdx();

    if (idx >= nElements) {
        return;
    }

    float3 pos = make_float3(
            vertexPos_D[dataArrSize*idx+dataArrOffs+0],
            vertexPos_D[dataArrSize*idx+dataArrOffs+1],
            vertexPos_D[dataArrSize*idx+dataArrOffs+2]);

    // If the sampled value is smaller than isoval, we are outside the
    // isosurface TODO Make this smarter
    if (SampleFieldAtPosTrilin_D<float>(pos, volume_D) <= isoval) {
        arr_D[idx] = 1.0;
    } else {
        arr_D[idx] = -1.0;
    }
}

__global__ void MeshLaplacian_D(
        float *in_D,
        uint inOffs,
        uint inStride,
        int *vertexNeighbours_D,
        uint maxNeighbours,
        uint vertexCnt,
        float *out_D,
        uint outOffs,
        uint outStride) {

    const uint idx = ::GetThreadIdx();
    if (idx >= vertexCnt) {
        return;
    }

    // Get initial position from global device memory
     float3 inOwn = make_float3(
             in_D[inStride*idx+inOffs+0],
             in_D[inStride*idx+inOffs+1],
             in_D[inStride*idx+inOffs+2]);

    uint activeNeighbourCnt = 0;
    float3 out = make_float3(0.0, 0.0, 0.0);
    for(int i = 0; i < maxNeighbours; ++i) {
        int isIdxValid = int(vertexNeighbours_D[maxNeighbours*idx+i] >= 0); // Check if idx != -1
        float3 in;
        int tmpIdx = isIdxValid*vertexNeighbours_D[maxNeighbours*idx+i]; // Map negative indices to 0
        in.x = in_D[inStride*tmpIdx+inOffs+0];
        in.y = in_D[inStride*tmpIdx+inOffs+1];
        in.z = in_D[inStride*tmpIdx+inOffs+2];
        out += (in - inOwn)*isIdxValid;
        activeNeighbourCnt += 1.0f*isIdxValid;
    }
    out /= activeNeighbourCnt; // Represents internal force

    out_D[outStride*idx+outOffs+0] = 1;
    out_D[outStride*idx+outOffs+1] = 1;
    out_D[outStride*idx+outOffs+2] = 1;

}


/**
 * Updates the positions of all vertices based on external and internal forces.
 * The external force is computed on the fly based on a the given volume.
 * Samples are aquired using tricubic interpolation.
 *
 * @param[in]      targetVolume_D         The volume the isosurface is extracted
 *                                        from
 * @param[in,out]  vertexPosMapped_D      The vertex data buffer
 * @param[in]      vertexExternalForces_D The external force and scale factor
 *                                        (in 'w') for all vertices
 * @param[in]      vertexNeighbours_D     The neighbour indices of all vertices
 * @param[in]      gradient_D             Array with the gradient
 * @param[in]      vtxNormals_D           The current normals of all vertices
 * @param[in]      vertexCount            The number of vertices
 * @param[in]      externalWeight         Weighting factor for the external
 *                                        forces. The factor for internal forces
 *                                        is implicitely defined by
 *                                        1.0-'externalWeight'
 * @param[in]      forcesScl              General scale factor for the final
 *                                        combined force
 * @param[in]      stiffness              The stiffness of the springs defining
 *                                        the internal forces
 * @param[in]      isoval                 The isovalue defining the isosurface
 * @param[in]      minDispl               The minimum displacement for the
 *                                        vertices to be updated
 * @param[in]      dataArrOffs            The vertex position offset in the
 *                                        vertex data buffer
 * @param[in]      dataArrSize            The stride of the vertex data buffer TODO
 */
__global__ void UpdateVtxPos_D(
        float *targetVolume_D,
        float *vertexPosMapped_D,
        float *vertexExternalForcesScl_D,
        float4 *gradient_D,
        float3 *laplacian_D,
        float3 *laplacian2_D,
        uint vertexCnt,
        float externalWeight,
        float forcesScl,
        float stiffness,
        float isoval,
        float minDispl,
        bool useCubicInterpolation,
        uint dataArrOffsPos,
        uint dataArrOffsNormal,
        uint dataArrSize) {

    const uint idx = GetThreadIdx();
    if (idx >= vertexCnt) {
        return;
    }

    const uint posBaseIdx = dataArrSize*idx+dataArrOffsPos;


    /* Retrieve stuff from global device memory */

    // Get initial position from global device memory
    float3 posOld = make_float3(
            vertexPosMapped_D[posBaseIdx+0],
            vertexPosMapped_D[posBaseIdx+1],
            vertexPosMapped_D[posBaseIdx+2]);

    // Get initial scale factor for external forces
    float externalForcesScl = vertexExternalForcesScl_D[idx];

    // Get partial derivatives
    float3 laplacian = laplacian_D[idx];
    float3 laplacian2 = laplacian2_D[idx];

    /* Update position */

    // No warp divergence here, since useCubicInterpolation is the same for all
    // threads
    const float sampleDens = useCubicInterpolation
                    ? SampleFieldAtPosTricub_D<float>(posOld, targetVolume_D)
                    : SampleFieldAtPosTrilin_D<float>(posOld, targetVolume_D);

    // Switch sign and scale down if necessary
    bool negative = externalForcesScl < 0;
    bool outside = sampleDens <= isoval;
    int switchSign = int((negative && outside)||(!negative && !outside));
    externalForcesScl = externalForcesScl*(1.0*(1-switchSign) - 1.0*switchSign);
    externalForcesScl *= (1.0*(1-switchSign) + 0.5*(switchSign));

    // Sample gradient by cubic interpolation
    float4 externalForceTmp = useCubicInterpolation
            ? SampleFieldAtPosTricub_D<float4>(posOld, gradient_D)
            : SampleFieldAtPosTrilin_D<float4>(posOld, gradient_D);

    float3 externalForce;
    externalForce.x = externalForceTmp.x;
    externalForce.y = externalForceTmp.y;
    externalForce.z = externalForceTmp.z;

    externalForce = safeNormalize(externalForce);
    externalForce *= forcesScl*externalForcesScl*externalWeight;

    // Umbrella internal force
    float3 posNew = posOld + externalForce +
            (1.0-externalWeight)*forcesScl*((1.0 - stiffness)*laplacian - stiffness*laplacian2);

    /* Write back to global device memory */

    vertexPosMapped_D[posBaseIdx+0] = posNew.x;
    vertexPosMapped_D[posBaseIdx+1] = posNew.y;
    vertexPosMapped_D[posBaseIdx+2] = posNew.z;

    // Write external forces scale factor back to global device memory
    vertexExternalForcesScl_D[idx] = externalForcesScl;
}


/*
 * DeformableGPUSurfaceMT::DeformableGPUSurfaceMT
 */
DeformableGPUSurfaceMT::DeformableGPUSurfaceMT() : GPUSurfaceMT(),
        vboCorruptTriangleVertexFlag(0) {

}


/*
 * DeformableGPUSurfaceMT::DeformableGPUSurfaceMT
 */
DeformableGPUSurfaceMT::DeformableGPUSurfaceMT(const DeformableGPUSurfaceMT& other) :
    GPUSurfaceMT(other) {

    CudaSafeCall(this->vertexExternalForcesScl_D.Validate(other.vertexExternalForcesScl_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->vertexExternalForcesScl_D.Peek(),
            other.vertexExternalForcesScl_D.PeekConst(),
            this->vertexExternalForcesScl_D.GetCount()*sizeof(float),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->externalForces_D.Validate(other.externalForces_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->externalForces_D.Peek(),
            other.externalForces_D.PeekConst(),
            this->externalForces_D.GetCount()*sizeof(float4),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->laplacian_D.Validate(other.laplacian_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->laplacian_D.Peek(),
            other.laplacian_D.PeekConst(),
            this->laplacian_D.GetCount()*sizeof(float3),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->laplacian2_D.Validate(other.laplacian2_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->laplacian2_D.Peek(),
            other.laplacian2_D.PeekConst(),
            this->laplacian2_D.GetCount()*sizeof(float3),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->displLen_D.Validate(other.displLen_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->displLen_D.Peek(),
            other.displLen_D.PeekConst(),
            this->displLen_D.GetCount()*sizeof(float),
            hipMemcpyDeviceToDevice));

    /* Make deep copy of corrupt triangle flag buffer */

    if (other.vboCorruptTriangleVertexFlag) {
        // Destroy if necessary
        if (this->vboCorruptTriangleVertexFlag) {
            glBindBufferARB(GL_ARRAY_BUFFER, this->vboCorruptTriangleVertexFlag);
            glDeleteBuffersARB(1, &this->vboCorruptTriangleVertexFlag);
            glBindBufferARB(GL_ARRAY_BUFFER, 0);
            this->vboCorruptTriangleVertexFlag = 0;
        }

        // Create vertex buffer object for triangle indices
        glGenBuffersARB(1, &this->vboCorruptTriangleVertexFlag);

        CheckForGLError();

        // Map as copy buffer
        glBindBufferARB(GL_COPY_READ_BUFFER, other.vboCorruptTriangleVertexFlag);
        glBindBufferARB(GL_COPY_WRITE_BUFFER, this->vboCorruptTriangleVertexFlag);
        glBufferDataARB(GL_COPY_WRITE_BUFFER,
                sizeof(int)*this->vertexCnt*3, 0, GL_DYNAMIC_DRAW);
        // Copy data
        glCopyBufferSubData(GL_COPY_READ_BUFFER, GL_COPY_WRITE_BUFFER, 0, 0,
                sizeof(int)*this->vertexCnt*3);
        glBindBufferARB(GL_COPY_WRITE_BUFFER, 0);
        glBindBufferARB(GL_COPY_READ_BUFFER, 0);
        CheckForGLError();
    }
}


/*
 * DeformableGPUSurfaceMT::~DeformableGPUSurfaceMT
 */
DeformableGPUSurfaceMT::~DeformableGPUSurfaceMT() {
}


/*
 * DeformableGPUSurfaceMT::FlagCorruptTriangleVertices
 */
bool DeformableGPUSurfaceMT::FlagCorruptTriangleVertices(
        float *targetVol_D,
        int3 volDim,
        float3 volOrg,
        float3 volDelta,
        float isovalue) {

    using namespace vislib::sys;

    if (!this->InitCorruptFlagVBO(this->vertexCnt)) {
        return false;
    }

    // Init constant device params
    if (!initGridParams(volDim, volOrg, volDelta)) {
        Log::DefaultLog.WriteMsg(Log::LEVEL_ERROR,
                "%s: could not init constant device params",
                this->ClassName());
        return false;
    }

//    ::CheckForCudaErrorSync();

    hipGraphicsResource* cudaTokens[3];

    // Register memory with CUDA
    if (!CudaSafeCall(hipGraphicsGLRegisterBuffer(
            &cudaTokens[0],
            this->vboVtxData,
            cudaGraphicsMapFlagsNone))) {
        return false;
    }

//    ::CheckForCudaErrorSync();

    if (!CudaSafeCall(hipGraphicsGLRegisterBuffer(
            &cudaTokens[1],
            this->vboTriangleIdx,
            cudaGraphicsMapFlagsNone))) {
        return false;
    }

//    ::CheckForCudaErrorSync();

    if (!CudaSafeCall(hipGraphicsGLRegisterBuffer(
            &cudaTokens[2],
            this->vboCorruptTriangleVertexFlag,
            cudaGraphicsMapFlagsNone))) {
        return false;
    }

//    ::CheckForCudaErrorSync();

    // Map cuda ressource handles
    if (!CudaSafeCall(hipGraphicsMapResources(3, cudaTokens, 0))) {
        return false;
    }

//    ::CheckForCudaErrorSync();

    /* Get mapped pointers to the vertex data buffer */

    float *vboFlagPt;
    float *vboPt;
    size_t vboSize;
    unsigned int *vboTriangleIdxPt;

    if (!CudaSafeCall(hipGraphicsResourceGetMappedPointer(
            reinterpret_cast<void**>(&vboPt),
            &vboSize,
            cudaTokens[0]))) {
        return false;
    }

//    ::CheckForCudaErrorSync();

    if (!CudaSafeCall(hipGraphicsResourceGetMappedPointer(
            reinterpret_cast<void**>(&vboTriangleIdxPt),
            &vboSize,
            cudaTokens[1]))) {
        return false;
    }

//    ::CheckForCudaErrorSync();

    if (!CudaSafeCall(hipGraphicsResourceGetMappedPointer(
            reinterpret_cast<void**>(&vboFlagPt),
            &vboSize,
            cudaTokens[2]))) {
        return false;
    }

//    ::CheckForCudaErrorSync();

    // Call kernel
    FlagCorruptTriangleVertices_D <<< this->Grid(this->triangleCnt, 256), 256 >>> (
            vboFlagPt,
            vboPt,
            AbstractGPUSurface::vertexDataStride,
            AbstractGPUSurface::vertexDataOffsPos,
            AbstractGPUSurface::vertexDataOffsNormal,
            vboTriangleIdxPt,
            targetVol_D,
            this->triangleCnt,
            isovalue);

//    ::CheckForCudaErrorSync();

    if (!CudaSafeCall(hipGetLastError())) {
        return false;
    }

//    ::CheckForCudaErrorSync();

    if (!CudaSafeCall(hipGraphicsUnmapResources(3, cudaTokens, 0))) {
        return false;
    }

//    ::CheckForCudaErrorSync();

    if (!CudaSafeCall(hipGraphicsUnregisterResource(cudaTokens[0]))) {
        return false;
    }

//    ::CheckForCudaErrorSync();

    if (!CudaSafeCall(hipGraphicsUnregisterResource(cudaTokens[1]))) {
        return false;
    }

//    ::CheckForCudaErrorSync();

    if (!CudaSafeCall(hipGraphicsUnregisterResource(cudaTokens[2]))) {
        return false;
    }

    return true;
}


/*
 * DeformableGPUSurfaceMT::InitCorruptFlagVBO
 */
bool DeformableGPUSurfaceMT::InitCorruptFlagVBO(size_t vertexCnt) {

    // Destroy if necessary
    if (this->vboCorruptTriangleVertexFlag) {
        glBindBufferARB(GL_ELEMENT_ARRAY_BUFFER, this->vboCorruptTriangleVertexFlag);
        glDeleteBuffersARB(1, &this->vboCorruptTriangleVertexFlag);
        this->vboCorruptTriangleVertexFlag = 0;
    }

    // Create vertex buffer object for corrupt vertex flag
    glGenBuffersARB(1, &this->vboCorruptTriangleVertexFlag);
    glBindBufferARB(GL_ARRAY_BUFFER, this->vboCorruptTriangleVertexFlag);
    glBufferDataARB(GL_ARRAY_BUFFER, sizeof(float)*3*vertexCnt, 0, GL_DYNAMIC_DRAW);
    glBindBufferARB(GL_ARRAY_BUFFER, 0);

    return CheckForGLError();
}


/*
 * DeformableGPUSurfaceMT::initExtForcesGradient
 */
bool DeformableGPUSurfaceMT::initExtForcesGradient(float *volTarget_D,
        int3 volDim, float3 volOrg, float3 volDelta) {
    using namespace vislib::sys;

    int volSize = volDim.x*volDim.y*volDim.z;

    // Init constant device params
    if (!initGridParams(volDim, volOrg, volDelta)) {
        Log::DefaultLog.WriteMsg(Log::LEVEL_ERROR,
                "%s: could not init constant device params",
                this->ClassName());
        return false;
    }

    // Allocate memory
    if (!CudaSafeCall(this->externalForces_D.Validate(volSize*4))) {
        Log::DefaultLog.WriteMsg(Log::LEVEL_ERROR,
                "%s: could not allocate memory",
                this->ClassName());
        return false;
    }

    // Init with zero
    if (!CudaSafeCall(this->externalForces_D.Set(0))) {
        Log::DefaultLog.WriteMsg(Log::LEVEL_ERROR,
                "%s: could not init memory",
                this->ClassName());
        return false;
    }

#ifdef USE_CUDA_TIMER
    float dt_ms;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipEventRecord(event1, 0);
#endif

    // Calculate gradient using finite differences
    calcVolGradient_D <<< this->Grid(volSize, 256), 256 >>> (
            (float4*)this->externalForces_D.Peek(), volTarget_D);

#ifdef USE_CUDA_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("CUDA time for 'CalcVolGradient_D':                     %.10f sec\n",
            dt_ms/1000.0f);
#endif

    return true;

}


/*
 * DeformableGPUSurfaceMT::initExtForcesDistfield
 */
bool DeformableGPUSurfaceMT::initExtForcesDistfield(
        float *volume_D,
        float *vertexBuffer_D,
        int3 volDim,
        float3 volOrg,
        float3 volDelta,
        float distfieldDist,
        float isovalue) {

    using namespace vislib::sys;

    int volSize = volDim.x*volDim.y*volDim.z;

    // Compute distance field
    if (!CudaSafeCall(this->distField_D.Validate(volSize))) {
        return false;
    }

#ifdef USE_TIMER
    float dt_ms;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipEventRecord(event1, 0);
#endif

    computeDistField_D <<< Grid(volSize, 256), 256 >>> (
            vertexBuffer_D,
            this->distField_D.Peek(),
            this->vertexCnt,
            this->vertexDataOffsPos,
            this->vertexDataStride);

#ifdef USE_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("CUDA time for 'ComputeDistField_D':                    %.10f sec\n",
            dt_ms/1000.0f);
#endif

    // Compute gradient
    if (!CudaSafeCall(this->externalForces_D.Validate(volSize*4))) {
        return false;
    }
    if (!CudaSafeCall(this->externalForces_D.Set(0))) {
        return false;
    }

#ifdef USE_TIMER
    hipEventRecord(event1, 0);
#endif

    // Calculate gradient using finite differences
    calcVolGradientWithDistField_D <<< Grid(volSize, 256), 256 >>> (
            (float4*)this->externalForces_D.Peek(),
            volume_D,
            this->distField_D.Peek(), distfieldDist, isovalue);

#ifdef USE_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("CUDA time for 'CalcVolGradientWithDistField_D':        %.10f sec\n",
            dt_ms/1000.0f);
#endif

    return CudaSafeCall(hipGetLastError());
}


bool DeformableGPUSurfaceMT::initExtForcesGVF(
        float *volumeTarget_D,
        const unsigned int *cellStatesTarget_D,
        int3 volDim,
        float3 volOrg,
        float3 volDelta,
        float isovalue,
        float gvfScl,
        unsigned int gvfIt) {

    int volSize = volDim.x*volDim.y*volDim.z;

    // Compute external forces
    if (!CudaSafeCall(this->externalForces_D.Validate(volSize*4))) {
        return false;
    }
    if (!CudaSafeCall(this->externalForces_D.Set(0))) {
        return false;
    }
    if (!CudaSafeCall(this->gvfTmp_D.Validate(volSize*4))) {
        return false;
    }
    if (!CudaSafeCall(this->gvfTmp_D.Set(0))) {
        return false;
    }
    if (!CudaSafeCall(this->gvfConstData_D.Validate(volSize*4))) {
        return false;
    }
    if (!CudaSafeCall(this->gvfConstData_D.Set(0))) {
        return false;
    }

    // Initialize device constants
    DiffusionSolver::grid grid_H;
    grid_H.size = volDim;
    grid_H.delta = volDelta;
    grid_H.org = volOrg;
    if (!CudaSafeCall(DiffusionSolver::InitDevConstants(grid_H, isovalue))) {
        return false;
    }

    // Use GVF
    if (!DiffusionSolver::CalcGVF(
            volumeTarget_D,
            this->gvfConstData_D.Peek(),
            cellStatesTarget_D,
            volDim,
            isovalue,
            this->externalForces_D.Peek(),
            this->gvfTmp_D.Peek(),
            gvfIt,
            gvfScl)) {
        return false;
    }

    return true;
}


/*
 * DeformableGPUSurfaceMT::initExtForcesTwoWayGVF
 */
bool DeformableGPUSurfaceMT::initExtForcesTwoWayGVF(
        float *volumeSource_D,
        float *volumeTarget_D,
        const unsigned int *cellStatesSource_D,
        const unsigned int *cellStatesTarget_D,
        int3 volDim,
        float3 volOrg,
        float3 volDelta,
        float isovalue,
        float gvfScl,
        unsigned int gvfIt) {

    using namespace vislib::sys;

    int volSize = volDim.x*volDim.y*volDim.z;

    // Compute external forces
    if (!CudaSafeCall(this->externalForces_D.Validate(volSize*4))) {
        return false;
    }
    if (!CudaSafeCall(this->externalForces_D.Set(0))) {
        return false;
    }
    if (!CudaSafeCall(this->gvfTmp_D.Validate(volSize*4))) {
        return false;
    }
    if (!CudaSafeCall(this->gvfTmp_D.Set(0))) {
        return false;
    }
    if (!CudaSafeCall(this->gvfConstData_D.Validate(volSize*4))) {
        return false;
    }
    if (!CudaSafeCall(this->gvfConstData_D.Set(0))) {
        return false;
    }

    // Initialize device constants
    DiffusionSolver::grid grid_H;
    grid_H.size = volDim;
    grid_H.delta = volDelta;
    grid_H.org = volOrg;
    if (!CudaSafeCall(DiffusionSolver::InitDevConstants(grid_H, isovalue))) {
        return false;
    }

    // Calculate two way gvf by using isotropic diffusion
    if (!DiffusionSolver::CalcTwoWayGVF(
           volumeSource_D,
           volumeTarget_D,
           cellStatesSource_D,
           cellStatesTarget_D,
           volDim,
           volOrg,
           volDelta,
           isovalue,
           this->gvfConstData_D.Peek(),
           this->externalForces_D.Peek(),
           this->gvfTmp_D.Peek(),
           gvfIt,
           gvfScl)) {
        return false;
    }

    return true;
}


/*
 * DeformableGPUSurfaceMT::InitGridParams
 */
bool DeformableGPUSurfaceMT::InitGridParams(uint3 gridSize, float3 org, float3 delta) {
    hipMemcpyToSymbol(HIP_SYMBOL(gridSize_D), &gridSize, sizeof(uint3));
    hipMemcpyToSymbol(HIP_SYMBOL(gridOrg_D), &org, sizeof(float3));
    hipMemcpyToSymbol(HIP_SYMBOL(gridDelta_D), &delta, sizeof(float3));
//    printf("Init grid with org %f %f %f, delta %f %f %f, dim %u %u %u\n", org.x,
//            org.y, org.z, delta.x, delta.y, delta.z, gridSize.x, gridSize.y,
//            gridSize.z);
    return CudaSafeCall(hipGetLastError());
}


/*
 * DeformableGPUSurfaceMT::MorphToVolumeGradient
 */
bool DeformableGPUSurfaceMT::MorphToVolumeGradient(
        float *volume_D,
        int3 volDim,
        float3 volOrg,
        float3 volDelta,
        float isovalue,
        InterpolationMode interpMode,
        size_t maxIt,
        float surfMappedMinDisplScl,
        float springStiffness,
        float forceScl,
        float externalForcesWeight) {

    using vislib::sys::Log;

    if ((!this->triangleIdxReady)||(!this->neighboursReady)) {
        return false;
    }

    if (volume_D == NULL) {
        return false;
    }

    if (!initExtForcesGradient(volume_D,
            volDim, volOrg, volDelta)) {
        return false;
    }

    // Register memory with CUDA
    if (!CudaSafeCall(hipGraphicsGLRegisterBuffer(
            &this->vertexDataResource, this->vboVtxData,
            cudaGraphicsMapFlagsNone))) {
        return false;
    }

    if (!CudaSafeCall(hipGraphicsMapResources(1, &this->vertexDataResource, 0))) {
        return false;
    }

    // Get mapped pointers to the vertex data buffers
    float *vboPt;
    size_t vboSize;
    if (!CudaSafeCall(hipGraphicsResourceGetMappedPointer(
            reinterpret_cast<void**>(&vboPt), // The mapped pointer
            &vboSize,              // The size of the accessible data
            this->vertexDataResource))) {                 // The mapped resource
        return false;
    }


//        // DEBUG Print normals
//        HostArr<float> vertexBuffer;
//        vertexBuffer.Validate(this->vertexDataStride*this->vertexCnt*sizeof(float));
//        if (!CudaSafeCall(hipMemcpy(vertexBuffer.Peek(), vboPt,
//                this->vertexDataStride*this->vertexCnt*sizeof(float), hipMemcpyDeviceToHost))) {
//            return false;
//        }
//        for (int i = 0; i < this->vertexCnt; i+=3) {
//    //        if (uint(abs(vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+0]))>= this->vertexCnt) {
//                        printf("%i: pos %f %f %f, normal %f %f %f, texcoord %f %f %f\n", i,
//                                vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsPos+0],
//                                vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsPos+1],
//                                vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsPos+2],
//                                vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+0],
//                                vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+1],
//                                vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+2],
//                                vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsTexCoord+0],
//                                vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsTexCoord+1],
//                                vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsTexCoord+2],
//                                this->vertexCnt);
//    //        }
//        }
//        vertexBuffer.Release();
//        // end DEBUG

    // Init forces scale factor with -1 or 1, depending on whether they start
    // outside or inside the isosurface
    if (!CudaSafeCall(this->vertexExternalForcesScl_D.Validate(this->vertexCnt))) {
        return false;
    }
#ifdef USE_TIMER
    float dt_ms;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipEventRecord(event1, 0);
#endif

    initExternalForceScl_D <<< Grid(this->vertexCnt, 256), 256 >>> (
            this->vertexExternalForcesScl_D.Peek(),
            volume_D,
            vboPt,
            this->vertexCnt,
            isovalue,
            this->vertexDataOffsPos,
            this->vertexDataStride);

#ifdef USE_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("CUDA time for 'InitExternalForceScl_D':                %.10f sec\n",
            dt_ms/1000.0f);
#endif

    // Iterations for new position
    if (!this->updateVtxPos(
            volume_D,
            vboPt,
            volDim,
            volOrg,
            volDelta,
            isovalue,
            (interpMode == INTERP_CUBIC),
            maxIt,
            surfMappedMinDisplScl,
            springStiffness,
            forceScl,
            externalForcesWeight)) {
        return false;
    }

    if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->vertexDataResource, 0))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
        return false;
    }

    return true;
}


/*
 * DeformableGPUSurfaceMT::MorphToVolumeDistfield
 */
bool DeformableGPUSurfaceMT::MorphToVolumeDistfield(
        float *volume_D,
        int3 volDim,
        float3 volOrg,
        float3 volDelta,
        float isovalue,
        InterpolationMode interpMode,
        size_t maxIt,
        float surfMappedMinDisplScl,
        float springStiffness,
        float forceScl,
        float externalForcesWeight,
        float distfieldDist) {

    using vislib::sys::Log;

    if ((!this->triangleIdxReady)||(!this->neighboursReady)) {
        return false;
    }

    if (volume_D == NULL) {
        return false;
    }

    // Register memory with CUDA
    if (!CudaSafeCall(hipGraphicsGLRegisterBuffer(
            &this->vertexDataResource, this->vboVtxData,
            cudaGraphicsMapFlagsNone))) {
        return false;
    }

    if (!CudaSafeCall(hipGraphicsMapResources(1, &this->vertexDataResource, 0))) {
        return false;
    }

    // Get mapped pointers to the vertex data buffers
    float *vboPt;
    size_t vboSize;
    if (!CudaSafeCall(hipGraphicsResourceGetMappedPointer(
            reinterpret_cast<void**>(&vboPt), // The mapped pointer
            &vboSize,              // The size of the accessible data
            this->vertexDataResource))) {                 // The mapped resource
        return false;
    }


    if (!this->initExtForcesDistfield(
            volume_D,
            vboPt,
            volDim,
            volOrg,
            volDelta,
            distfieldDist,
            isovalue)) {
        return false;
    }


    // Init forces scale factor with -1 or 1, depending on whether they start
    // outside or inside the isosurface
    if (!CudaSafeCall(this->vertexExternalForcesScl_D.Validate(this->vertexCnt))) {
        return false;
    }
#ifdef USE_TIMER
    float dt_ms;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipEventRecord(event1, 0);
#endif

    initExternalForceScl_D <<< Grid(this->vertexCnt, 256), 256 >>> (
            this->vertexExternalForcesScl_D.Peek(),
            volume_D,
            vboPt,
            this->vertexCnt,
            isovalue,
            this->vertexDataOffsPos,
            this->vertexDataStride);

#ifdef USE_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("CUDA time for 'InitExternalForceScl_D':                %.10f sec\n",
            dt_ms/1000.0f);
#endif

    // Iterations for new position
    if (!this->updateVtxPos(
            volume_D,
            vboPt,
            volDim,
            volOrg,
            volDelta,
            isovalue,
            (interpMode == INTERP_CUBIC),
            maxIt,
            surfMappedMinDisplScl,
            springStiffness,
            forceScl,
            externalForcesWeight)) {
        return false;
    }

    if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->vertexDataResource, 0))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
        return false;
    }

    return true;
}


/*
 * DeformableGPUSurfaceMT::MorphToVolumeGVF
 */
bool DeformableGPUSurfaceMT::MorphToVolumeGVF(float *volumeSource_D,
        float *volumeTarget_D,
        const unsigned int *targetCubeStates_D,
        int3 volDim,
        float3 volOrg,
        float3 volDelta,
        float isovalue,
        InterpolationMode interpMode,
        size_t maxIt,
        float surfMappedMinDisplScl,
        float springStiffness,
        float forceScl,
        float externalForcesWeight,
        float gvfScl,
        unsigned int gvfIt) {

    using vislib::sys::Log;

    if ((!this->triangleIdxReady)||(!this->neighboursReady)) {
        return false;
    }

    if (volumeTarget_D == NULL) {
        return false;
    }

    if (!this->initExtForcesGVF(
            volumeTarget_D,
            targetCubeStates_D,
            volDim,
            volOrg,
            volDelta,
            isovalue,
            gvfScl,
            gvfIt)) {
        return false;
    }

    // Register memory with CUDA
    if (!CudaSafeCall(hipGraphicsGLRegisterBuffer(
            &this->vertexDataResource, this->vboVtxData,
            cudaGraphicsMapFlagsNone))) {
        return false;
    }

    if (!CudaSafeCall(hipGraphicsMapResources(1, &this->vertexDataResource, 0))) {
        return false;
    }

    // Get mapped pointers to the vertex data buffers
    float *vboPt;
    size_t vboSize;
    if (!CudaSafeCall(hipGraphicsResourceGetMappedPointer(
            reinterpret_cast<void**>(&vboPt), // The mapped pointer
            &vboSize,              // The size of the accessible data
            this->vertexDataResource))) {                 // The mapped resource
        return false;
    }

    // Init forces scale factor with -1 or 1, depending on whether they start
    // outside or inside the isosurface
    if (!CudaSafeCall(this->vertexExternalForcesScl_D.Validate(this->vertexCnt))) {
        return false;
    }
#ifdef USE_TIMER
    float dt_ms;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipEventRecord(event1, 0);
#endif

    initExternalForceScl_D <<< Grid(this->vertexCnt, 256), 256 >>> (
            this->vertexExternalForcesScl_D.Peek(),
            volumeTarget_D,
            vboPt,
            this->vertexCnt,
            isovalue,
            this->vertexDataOffsPos,
            this->vertexDataStride);

#ifdef USE_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("CUDA time for 'InitExternalForceScl_D':                %.10f sec\n",
            dt_ms/1000.0f);
#endif

    // Iterations for new position
    if (!this->updateVtxPos(
            volumeTarget_D,
            vboPt,
            volDim,
            volOrg,
            volDelta,
            isovalue,
            (interpMode == INTERP_CUBIC),
            maxIt,
            surfMappedMinDisplScl,
            springStiffness,
            forceScl,
            externalForcesWeight)) {
        return false;
    }

    if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->vertexDataResource, 0))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
        return false;
    }

    return true;
}


/*
 * DeformableGPUSurfaceMT::MorphToVolumeTwoWayGVF
 */
bool DeformableGPUSurfaceMT::MorphToVolumeTwoWayGVF(
        float *volumeSource_D,
        float *volumeTarget_D,
        const unsigned int *cellStatesSource_D,
        const unsigned int *cellStatesTarget_D,
        int3 volDim,
        float3 volOrg,
        float3 volDelta,
        float isovalue,
        InterpolationMode interpMode,
        size_t maxIt,
        float surfMappedMinDisplScl,
        float springStiffness,
        float forceScl,
        float externalForcesWeight,
        float gvfScl,
        unsigned int gvfIt) {

    using vislib::sys::Log;

    if ((!this->triangleIdxReady)||(!this->neighboursReady)) {
        return false;
    }

    if ((volumeTarget_D == NULL)||(volumeSource_D == NULL)) {
        return false;
    }

    if (!this->initExtForcesTwoWayGVF(
            volumeSource_D,
            volumeTarget_D,
            cellStatesSource_D,
            cellStatesTarget_D,
            volDim, volOrg, volDelta,
            isovalue, gvfScl, gvfIt)) {
        return false;
    }

    // Register memory with CUDA
    if (!CudaSafeCall(hipGraphicsGLRegisterBuffer(
            &this->vertexDataResource, this->vboVtxData,
            cudaGraphicsMapFlagsNone))) {
        return false;
    }

    if (!CudaSafeCall(hipGraphicsMapResources(1, &this->vertexDataResource, 0))) {
        return false;
    }

    // Get mapped pointers to the vertex data buffers
    float *vboPt;
    size_t vboSize;
    if (!CudaSafeCall(hipGraphicsResourceGetMappedPointer(
            reinterpret_cast<void**>(&vboPt), // The mapped pointer
            &vboSize,              // The size of the accessible data
            this->vertexDataResource))) {                 // The mapped resource
        return false;
    }

    // Init forces scale factor with -1 or 1, depending on whether they start
    // outside or inside the isosurface
    if (!CudaSafeCall(this->vertexExternalForcesScl_D.Validate(this->vertexCnt))) {
        return false;
    }
#ifdef USE_TIMER
    float dt_ms;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipEventRecord(event1, 0);
#endif

    initExternalForceScl_D <<< Grid(this->vertexCnt, 256), 256 >>> (
            this->vertexExternalForcesScl_D.Peek(),
            volumeTarget_D,
            vboPt,
            this->vertexCnt,
            isovalue,
            this->vertexDataOffsPos,
            this->vertexDataStride);

#ifdef USE_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("CUDA time for 'InitExternalForceScl_D':                %.10f sec\n",
            dt_ms/1000.0f);
#endif

    // Iterations for new position
    if (!this->updateVtxPos(
            volumeTarget_D,
            vboPt,
            volDim,
            volOrg,
            volDelta,
            isovalue,
            (interpMode == INTERP_CUBIC),
            maxIt,
            surfMappedMinDisplScl,
            springStiffness,
            forceScl,
            externalForcesWeight)) {
        return false;
    }

    if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->vertexDataResource, 0))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
        return false;
    }

    return true;
}


/*
 * DeformableGPUSurfaceMT::operator=
 */
DeformableGPUSurfaceMT& DeformableGPUSurfaceMT::operator=(const DeformableGPUSurfaceMT &rhs) {
    GPUSurfaceMT::operator =(rhs);


    CudaSafeCall(this->vertexExternalForcesScl_D.Validate(rhs.vertexExternalForcesScl_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->vertexExternalForcesScl_D.Peek(),
            rhs.vertexExternalForcesScl_D.PeekConst(),
            this->vertexExternalForcesScl_D.GetCount()*sizeof(float),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->externalForces_D.Validate(rhs.externalForces_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->externalForces_D.Peek(),
            rhs.externalForces_D.PeekConst(),
            this->externalForces_D.GetCount()*sizeof(float),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->laplacian_D.Validate(rhs.laplacian_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->laplacian_D.Peek(),
            rhs.laplacian_D.PeekConst(),
            this->laplacian_D.GetCount()*sizeof(float3),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->laplacian2_D.Validate(rhs.laplacian2_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->laplacian2_D.Peek(),
            rhs.laplacian2_D.PeekConst(),
            this->laplacian2_D.GetCount()*sizeof(float3),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->displLen_D.Validate(rhs.displLen_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->displLen_D.Peek(),
            rhs.displLen_D.PeekConst(),
            this->displLen_D.GetCount()*sizeof(float),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->gvfTmp_D.Validate(rhs.gvfTmp_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->gvfTmp_D.Peek(),
            rhs.gvfTmp_D.PeekConst(),
            this->gvfTmp_D.GetCount()*sizeof(float),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->gvfConstData_D.Validate(rhs.gvfConstData_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->gvfConstData_D.Peek(),
            rhs.gvfConstData_D.PeekConst(),
            this->gvfConstData_D.GetCount()*sizeof(float),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->distField_D.Validate(rhs.distField_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->distField_D.Peek(),
            rhs.distField_D.PeekConst(),
            this->distField_D.GetCount()*sizeof(float),
            hipMemcpyDeviceToDevice));

    /* Make deep copy of corrupt triangle flag buffer */

    if (rhs.vboCorruptTriangleVertexFlag) {
        // Destroy if necessary
        if (this->vboCorruptTriangleVertexFlag) {
            glBindBufferARB(GL_ARRAY_BUFFER, this->vboCorruptTriangleVertexFlag);
            glDeleteBuffersARB(1, &this->vboCorruptTriangleVertexFlag);
            glBindBufferARB(GL_ARRAY_BUFFER, 0);
            this->vboCorruptTriangleVertexFlag = 0;
        }

        // Create vertex buffer object for triangle indices
        glGenBuffersARB(1, &this->vboCorruptTriangleVertexFlag);

        CheckForGLError();

        // Map as copy buffer
        glBindBufferARB(GL_COPY_READ_BUFFER, rhs.vboCorruptTriangleVertexFlag);
        glBindBufferARB(GL_COPY_WRITE_BUFFER, this->vboCorruptTriangleVertexFlag);
        glBufferDataARB(GL_COPY_WRITE_BUFFER,
                sizeof(int)*this->vertexCnt*3, 0, GL_DYNAMIC_DRAW);
        // Copy data
        glCopyBufferSubData(GL_COPY_READ_BUFFER, GL_COPY_WRITE_BUFFER, 0, 0,
                sizeof(int)*this->vertexCnt*3);
        glBindBufferARB(GL_COPY_WRITE_BUFFER, 0);
        glBindBufferARB(GL_COPY_READ_BUFFER, 0);
        CheckForGLError();
    }

    return *this;
}


/*
 * DeformableGPUSurfaceMT::Release
 */
void DeformableGPUSurfaceMT::Release() {
    GPUSurfaceMT::Release();
    CudaSafeCall(this->vertexExternalForcesScl_D.Release());
    CudaSafeCall(this->gvfTmp_D.Release());
    CudaSafeCall(this->gvfConstData_D.Release());
    CudaSafeCall(this->laplacian_D.Release());
    CudaSafeCall(this->laplacian2_D.Release());
    CudaSafeCall(this->displLen_D.Release());
    CudaSafeCall(this->distField_D.Release());
    CudaSafeCall(this->externalForces_D.Release());
}


/*
 * DeformableGPUSurfaceMT::updateVtxPos
 */
bool DeformableGPUSurfaceMT::updateVtxPos(
        float* volTarget_D,
        float* vertexBuffer_D,
        int3 volDim,
        float3 volOrg,
        float3 volDelta,
        float isovalue,
        bool useCubicInterpolation,
        size_t maxIt,
        float surfMappedMinDisplScl,
        float springStiffness,
        float forceScl,
        float externalForcesWeight) {

    using namespace vislib::sys;


//    // DEBUG Print normals
//    HostArr<float> vertexBuffer;
//    vertexBuffer.Validate(this->vertexDataStride*this->vertexCnt*sizeof(float));
//    if (!CudaSafeCall(hipMemcpy(vertexBuffer.Peek(), vertexBuffer_D,
//            this->vertexDataStride*this->vertexCnt*sizeof(float), hipMemcpyDeviceToHost))) {
//        return false;
//    }
//    for (int i = 0; i < this->vertexCnt; i+=3) {
////        if (uint(abs(vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+0]))>= this->vertexCnt) {
//                    printf("%i: pos %f %f %f, normal %f %f %f, texcoord %f %f %f\n", i,
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsPos+0],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsPos+1],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsPos+2],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+0],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+1],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+2],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsTexCoord+0],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsTexCoord+1],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsTexCoord+2],
//                            this->vertexCnt);
////        }
//    }
//    vertexBuffer.Release();
//    // end DEBUG


    // Init constant device params
    if (!initGridParams(volDim, volOrg, volDelta)) {
        Log::DefaultLog.WriteMsg(Log::LEVEL_ERROR,
                "%s: could not init constant device params",
                this->ClassName());
        return false;
    }

    if (!CudaSafeCall(this->laplacian_D.Validate(this->vertexCnt))) {
        return false;
    }
    if (!CudaSafeCall(this->laplacian_D.Set(0))) {
        return false;
    }

    if (!CudaSafeCall(this->laplacian2_D.Validate(this->vertexCnt))) {
        return false;
    }
    if (!CudaSafeCall(this->laplacian2_D.Set(0))) {
        return false;
    }

    if (!CudaSafeCall(this->displLen_D.Validate(this->vertexCnt))) {
        return false;
    }
    if (!CudaSafeCall(this->displLen_D.Set(0xff))) {
        return false;
    }

#ifdef USE_TIMER
    float dt_ms;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipEventRecord(event1, 0);
#endif

    // TODO Timer
    for (uint i = 0; i < maxIt; ++i) {

        // Calc laplacian
        printf("vertex count %u\n", this->vertexCnt);
        MeshLaplacian_D <<< this->Grid(this->vertexCnt, 256), 256 >>> (
                vertexBuffer_D,
                this->vertexDataOffsPos,
                this->vertexDataStride,
                this->vertexNeighbours_D.Peek(),
                18,
                this->vertexCnt,
                (float*)this->laplacian_D.Peek(),
                3,
                0);

        ::CheckForCudaErrorSync();

//        // DEBUG Print laplacian
//        HostArr<float3> laplacian;
//        laplacian.Validate(this->laplacian_D.GetCount());
//        this->laplacian_D.CopyToHost(laplacian.Peek());
//        for (int i = 0; i < this->laplacian_D.GetCount(); ++i) {
//            printf("laplacian %f %f %f\n", laplacian.Peek()[i].x,
//                    laplacian.Peek()[i].y,
//                    laplacian.Peek()[i].z);
//        }
//        laplacian.Release();
//        // END DEBUG

        ::CheckForCudaErrorSync();

        // Calc laplacian^2
        MeshLaplacian_D <<< this->Grid(this->vertexCnt, 256), 256 >>> (
                (float*)this->laplacian_D.Peek(),
                3,
                0,
                this->vertexNeighbours_D.Peek(),
                18,
                this->vertexCnt,
                (float*)this->laplacian2_D.Peek(),
                3,
                0);

        ::CheckForCudaErrorSync();

        // Update vertex position
        UpdateVtxPos_D <<< this->Grid(this->vertexCnt, 256), 256 >>> (
                volTarget_D,
                vertexBuffer_D,
                this->vertexExternalForcesScl_D.Peek(),
                (float4*)this->externalForces_D.Peek(),
                this->laplacian_D.Peek(),
                this->laplacian2_D.Peek(),
                this->vertexCnt,
                externalForcesWeight,
                forceScl,
                springStiffness,
                isovalue,
                surfMappedMinDisplScl,
                useCubicInterpolation,
                this->vertexDataOffsPos,
                this->vertexDataOffsNormal,
                this->vertexDataStride);

        ::CheckForCudaErrorSync();
    }

#ifdef USE_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&dt_ms, event1, event2);
    Log::DefaultLog.WriteMsg(Log::LEVEL_INFO,
            "%s: Time for mapping (%u iterations, %u vertices): %f sec\n",
            "DeformableGPUSurfaceMT",
            maxIt, this->vertexCnt, dt_ms/1000.0f);
#endif

    return CudaSafeCall(hipGetLastError());
}
#endif // WITH_CUDA

