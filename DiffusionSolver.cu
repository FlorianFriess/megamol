#include "hip/hip_runtime.h"
//
// DiffusionSolver.cpp
//
// Copyright (C) 2013 by University of Stuttgart (VISUS).
// All rights reserved.
//
// Created on : Sep 16, 2013
// Author     : scharnkn
//

#include "stdafx.h"
#include "DiffusionSolver.h"

#ifdef WITH_CUDA

#include "cuda_error_check.h"
#include "cuda_helper.h"

using namespace megamol;
using namespace megamol::protein;

#define USE_TIMER
#define USE_CUDA_TIMER


////////////////////////////////////////////////////////////////////////////////
//  Grid utility functions offering coversion of different indices. The term  //
//  'cell' refers to grid centers rather than grid corners. Thus, there are   //
//  gridSize - 1 cells in every dimension.                                    //
////////////////////////////////////////////////////////////////////////////////

/**
 * Answers the grid position index associated with the given coordinates.
 *
 * @param v0 The coordinates
 * @return The index
 */
inline __device__ uint GetPosIdxByGridCoords(uint3 v0, uint3 voldim) {
    return voldim.x*(voldim.y*v0.z + v0.y) + v0.x;
}

/**
 * Answers the cell index associated with the given coordinates.
 *
 * @param v0 The coordinates
 * @return The index
 */
inline __device__ uint GetCellIdxByGridCoords(int3 v0, uint3 voldim) {
    return (voldim.x-1)*((voldim.y-1)*v0.z + v0.y) + v0.x;
}

/**
 * Answers the grid position coordinates associated with a given cell index.
 * The returned position is the left/lower/back corner of the cell
 *
 * @param index The index
 * @return The coordinates
 */
inline __device__ uint3 GetGridCoordsByCellIdx(uint index, uint3 voldim) {
    return make_uint3(index % (voldim.x-1),
                      (index / (voldim.x-1)) % (voldim.y-1),
                      (index / (voldim.x-1)) / (voldim.y-1));
}

/**
 * Answers the cell coordinates associated with a given grid position index.
 *
 * @param index The index
 * @return The coordinates
 */
inline __device__ uint3 GetGridCoordsByPosIdx(uint index, uint3 voldim) {
    return make_uint3(index % voldim.x,
                      (index / voldim.x) % voldim.y,
                      (index / voldim.x) / voldim.y);
}


/*
 * updateGVF_D
 */
__global__ void updateGVF_D(
        float *gvf_D,
        float *gvfConstData_D, // b, c1, c2, c3
        float scl,
        uint3 voldim) {

    const uint idx = __umul24(__umul24(blockIdx.y, gridDim.x) +
            blockIdx.x, blockDim.x) + threadIdx.x;

    uint volsize = voldim.x*voldim.y*voldim.z;
    if (idx >= volsize) return;

    float3 gvf, gvfOld, gvfAdj[6];
    uint idxAdj[6];

    // Get grid coordinates
    uint3 gridC = make_uint3(
            idx % voldim.x,
            (idx / voldim.x) % voldim.y,
            (idx / voldim.x) / voldim.y);

    // Get const data
    float b = gvfConstData_D[4*idx+0];
    float c1 = gvfConstData_D[4*idx+1];
    float c2 = gvfConstData_D[4*idx+2];
    float c3 = gvfConstData_D[4*idx+3];

    /* Update isotropic diffusion for all vector components */

    // Get indices of adjacent values
    idxAdj[0] = ::GetPosIdxByGridCoords(make_uint3(clamp(uint(gridC.x-1), uint(0), voldim.x), gridC.y, gridC.z), voldim);
    idxAdj[1] = ::GetPosIdxByGridCoords(make_uint3(clamp(uint(gridC.x+1), uint(0), voldim.x), gridC.y, gridC.z), voldim);
    idxAdj[2] = ::GetPosIdxByGridCoords(make_uint3(gridC.x, clamp(uint(gridC.y-1), uint(0), voldim.y), gridC.z), voldim);
    idxAdj[3] = ::GetPosIdxByGridCoords(make_uint3(gridC.x, clamp(uint(gridC.y+1), uint(0), voldim.y), gridC.z), voldim);
    idxAdj[4] = ::GetPosIdxByGridCoords(make_uint3(gridC.x, gridC.y, clamp(uint(gridC.z-1), uint(0), voldim.z)), voldim);
    idxAdj[5] = ::GetPosIdxByGridCoords(make_uint3(gridC.x, gridC.y, clamp(uint(gridC.z+1), uint(0), voldim.z)), voldim);

    if (idxAdj[0] >= volsize) return;
    if (idxAdj[1] >= volsize) return;
    if (idxAdj[2] >= volsize) return;
    if (idxAdj[3] >= volsize) return;
    if (idxAdj[4] >= volsize) return;
    if (idxAdj[5] >= volsize) return;

    // Get adjacent gvf values
    gvfOld = make_float3(gvf_D[4*idx+0], gvf_D[4*idx+1], gvf_D[4*idx+2]);
    gvfAdj[0] = make_float3(gvf_D[4*idxAdj[0]+0], gvf_D[4*idxAdj[0]+1], gvf_D[4*idxAdj[0]+2]);
    gvfAdj[1] = make_float3(gvf_D[4*idxAdj[1]+0], gvf_D[4*idxAdj[1]+1], gvf_D[4*idxAdj[1]+2]);
    gvfAdj[2] = make_float3(gvf_D[4*idxAdj[2]+0], gvf_D[4*idxAdj[2]+1], gvf_D[4*idxAdj[2]+2]);
    gvfAdj[3] = make_float3(gvf_D[4*idxAdj[3]+0], gvf_D[4*idxAdj[3]+1], gvf_D[4*idxAdj[3]+2]);
    gvfAdj[4] = make_float3(gvf_D[4*idxAdj[4]+0], gvf_D[4*idxAdj[4]+1], gvf_D[4*idxAdj[4]+2]);
    gvfAdj[5] = make_float3(gvf_D[4*idxAdj[5]+0], gvf_D[4*idxAdj[5]+1], gvf_D[4*idxAdj[5]+2]);

    // Compute diffusion
    gvf.x = (1.0-b)*gvfOld.x;
    gvf.x += (gvfAdj[0].x + gvfAdj[1].x + gvfAdj[2].x + gvfAdj[3].x +
              gvfAdj[4].x + gvfAdj[5].x -6*gvfOld.x)*scl;
    gvf.x += c1;

    gvf.y = (1.0-b)*gvfOld.y;
    gvf.y += (gvfAdj[0].y + gvfAdj[1].y + gvfAdj[2].y + gvfAdj[3].y +
            gvfAdj[4].y + gvfAdj[5].y -6*gvfOld.y)*scl;
    gvf.y += c2;

    gvf.z = (1.0-b)*gvfOld.z;
    gvf.z += (gvfAdj[0].z + gvfAdj[1].z + gvfAdj[2].z + gvfAdj[3].z +
            gvfAdj[4].z + gvfAdj[5].z -6*gvfOld.z)*scl;
    gvf.z += c3;

    gvf.x = gvfAdj[4].x + c1;
    gvf.y = gvfAdj[4].y + c2;
    gvf.z = gvfAdj[4].z + c3;

    gvf.x = gvfAdj[4].x;
    gvf.y = gvfAdj[4].y;
    gvf.z = gvfAdj[4].z;

    gvf.x = c1;
    gvf.y = c2;
    gvf.z = c3;

    float len = length(gvf);
    if (len) gvf /= len;

    __syncthreads();
    gvf_D[4*idx+0] = gvf.x;
    gvf_D[4*idx+1] = gvf.y;
    gvf_D[4*idx+2] = gvf.z;
}


/*
 * DiffusionSolver::CalcGVF
 */
bool DiffusionSolver::CalcGVF(const float *startVol, float *gvfConstData_D,
        const unsigned int *cellStates_D,
        float *grad_D, size_t dim[3], float isovalue,
        float *gvf_D, unsigned int maxIt, float scl) {

    // TODO Use the correct starting volume!

    // Initialize the GVF field with the gradient in the starting regions
    if (!DiffusionSolver::initGVF(startVol, dim, cellStates_D, isovalue, grad_D,
            gvfConstData_D)) {
        return false;
    }

    uint volsize = dim[0]*dim[1]*dim[2];
    uint3 voldim = make_uint3(dim[0], dim[1], dim[2]);

    for (unsigned int it = 0; it < maxIt; ++it) {

#ifdef USE_CUDA_TIMER
        float dt_ms;
        hipEvent_t event1, event2;
        hipEventCreate(&event1);
        hipEventCreate(&event2);
        hipEventRecord(event1, 0);
#endif
        // Calculate gradient using finite differences
        updateGVF_D <<< DiffusionSolver::Grid(volsize, 256), 256 >>> (
                gvf_D, gvfConstData_D, scl, voldim);

        if (hipGetLastError() != hipSuccess) {
            return false;
        }

#ifdef USE_CUDA_TIMER
        hipEventRecord(event2, 0);
        hipEventSynchronize(event1);
        hipEventSynchronize(event2);
        hipEventElapsedTime(&dt_ms, event1, event2);
        printf("CUDA time for 'updateGVF_D':                       %.10f sec\n",
                dt_ms/1000.0f);
        hipEventRecord(event1, 0);
#endif
    }

    return true;
}


/*
 * calcGradient_D
 * Computes the gradient of the given volume.
 */
__global__ void calcGradient_D(const float *vol_D, float *grad_D,
        const unsigned int *cellStates_D, uint3 voldim) {

    const uint idx = __umul24(__umul24(blockIdx.y, gridDim.x) +
            blockIdx.x, blockDim.x) + threadIdx.x;

    uint volsize = voldim.x*voldim.y*voldim.z;
    if (idx >= volsize) return;



    int3 cellC;
    uint cellIdx;
    int active = 0;

    // Get grid coordinates
    int3 gridC = make_int3(
            idx % (voldim.x),
            (idx / voldim.x) % voldim.y,
            (idx / voldim.x) / voldim.y);

    int3 voldimI = make_int3(int(voldim.x), int(voldim.y), int(voldim.z));

    /* Check all eight adjactent cells */

    // (-1, -1, -1)
    cellC = make_int3(
            clamp(gridC.x-1, 0, voldimI.x-1),
            clamp(gridC.y-1, 0, voldimI.y-1),
            clamp(gridC.z-1, 0, voldimI.z-1));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    active = cellStates_D[cellIdx];
    // (-1, 1, -1)
    cellC = make_int3(
            clamp(gridC.x-1, 0, voldimI.x-1),
            clamp(gridC.y, 0, voldimI.y-1),
            clamp(gridC.z-1, 0, voldimI.z-1));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    active = cellStates_D[cellIdx];
    // (-1, 1, 1)
    cellC = make_int3(
            clamp(gridC.x-1, 0, voldimI.x-1),
            clamp(gridC.y, 0, voldimI.y-1),
            clamp(gridC.z, 0, voldimI.z-1));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    active = cellStates_D[cellIdx];
    // (-1, -1, 1)
    cellC = make_int3(
            clamp(gridC.x-1, 0, voldimI.x-1),
            clamp(gridC.y-1, 0, voldimI.y-1),
            clamp(gridC.z, 0, voldimI.z-1));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    active = cellStates_D[cellIdx];
    // (1, -1, -1)
    cellC = make_int3(
            clamp(gridC.x, 0, voldimI.x-1),
            clamp(gridC.y-1, 0, voldimI.y-1),
            clamp(gridC.z-1, 0, voldimI.z-1));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    active = cellStates_D[cellIdx];
    // (1, 1, -1)
    cellC = make_int3(
            clamp(gridC.x, 0, voldimI.x-1),
            clamp(gridC.y, 0, voldimI.y-1),
            clamp(gridC.z-1, 0, voldimI.z-1));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    active = cellStates_D[cellIdx];
    // (1, 1, 1)
    cellC = make_int3(
            clamp(gridC.x, 0, voldimI.x-1),
            clamp(gridC.y, 0, voldimI.y-1),
            clamp(gridC.z, 0, voldimI.z-1));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    active = cellStates_D[cellIdx];
    // (1, -1, 1)
    cellC = make_int3(
            clamp(gridC.x, 0, voldimI.x-1),
            clamp(gridC.y-1, 0, voldimI.y-1),
            clamp(gridC.z, 0, voldimI.z-1));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    active = cellStates_D[cellIdx];

    /* Sample gradient if necessary */

    if (active) {
        float3 grad;
        uint3 x1, x2;

        x1 = make_uint3(clamp(gridC.x+1, 0, voldimI.x-1), gridC.y, gridC.z);
        x2 = make_uint3(clamp(gridC.x-1, 0, voldimI.x-1), gridC.y, gridC.z);
        grad.x = vol_D[voldim.x*(voldim.y*x1.z + x1.y) + x1.x]-
                vol_D[voldim.x*(voldim.y*x2.z + x2.y) + x2.x];

        x1 = make_uint3(gridC.x, clamp(gridC.y+1, 0, voldimI.y-1), gridC.z);
        x2 = make_uint3(gridC.x, clamp(gridC.y-1, 0, voldimI.y-1), gridC.z);
        grad.y = vol_D[voldim.x*(voldim.y*x1.z + x1.y) + x1.x]-
                vol_D[voldim.x*(voldim.y*x2.z + x2.y) + x2.x];

        x1 = make_uint3(gridC.x, gridC.y, clamp(gridC.z+1, 0, voldimI.z-1));
        x2 = make_uint3(gridC.x, gridC.y, clamp(gridC.z-1, 0, voldimI.z-1));
        grad.z = vol_D[voldim.x*(voldim.y*x1.z + x1.y) + x1.x]-
                vol_D[voldim.x*(voldim.y*x2.z + x2.y) + x2.x];

        float len = length(grad);
        if (len > 0.0) grad/= len;
        grad_D[4*idx+0] = grad.x;
        grad_D[4*idx+1] = grad.y;
        grad_D[4*idx+2] = grad.z;
    }
}


/*
 * prepareGVFDiffusion
 */
__global__ void prepareGVFDiffusion_D(
        float *grad_D,
        float *gvfConstData_D, // b, c1, c2, c3
        uint3 voldim) {

    const uint idx = __umul24(__umul24(blockIdx.y, gridDim.x) +
            blockIdx.x, blockDim.x) + threadIdx.x;

    uint volsize = voldim.x*voldim.y*voldim.z;
    if (idx >= volsize) return;

    // Get grid coordinates
    uint3 gridCoord = make_uint3(
            idx % voldim.x,
            (idx / voldim.x) % voldim.y,
            (idx / voldim.x) / voldim.y);

    // Get vector field
    float3 grad;
    grad.x = grad_D[4*idx+0];
    grad.y = grad_D[4*idx+1];
    grad.z = grad_D[4*idx+2];

    // Compute len^2
    float len = grad.x*grad.x + grad.y*grad.y + grad.z*grad.z;

    // Write b to device memory
    gvfConstData_D[4*idx+0] = len;
    // Write c1, c2, and c3 to device memory
    gvfConstData_D[4*idx+1] = len*grad.x;
    gvfConstData_D[4*idx+2] = len*grad.y;
    gvfConstData_D[4*idx+3] = len*grad.z;

}


/*
 * DiffusionSolver::initGVFCuda
 */
bool DiffusionSolver::initGVF(const float *startVol, size_t dim[3],
        const unsigned int *cellStates_D,
        float isovalue, float *grad_D, float *gvfConstData_D) {

    size_t volsize = dim[0]*dim[1]*dim[2];
    uint3 voldim = make_uint3(dim[0], dim[1], dim[2]);

#ifdef USE_CUDA_TIMER
    float dt_ms;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipEventRecord(event1, 0);
#endif

    // Calculate gradient using finite differences
    calcGradient_D <<< DiffusionSolver::Grid(volsize, 256), 256 >>> (
            startVol, grad_D, cellStates_D, voldim);

#ifdef USE_CUDA_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("CUDA time for 'calcGradient_D':                        %.10f sec\n",
            dt_ms/1000.0f);
    hipEventRecord(event1, 0);
#endif

    // Precompute b,c1,c2, and c3
    prepareGVFDiffusion_D <<< DiffusionSolver::Grid(volsize, 256), 256 >>> (
            grad_D, gvfConstData_D, voldim);

#ifdef USE_CUDA_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("CUDA time for 'prepareGVFDiffusion_D':                 %.10f sec\n",
            dt_ms/1000.0f);
#endif

    return true;
}


/**
 * Returns a 1D grid definition based on the given threadsPerBlock value.
 *
 * @param size             The minimum number of threads
 * @param threadsPerBlock  The number of threads per block
 * @return The grid dimensions
 */
extern "C" dim3 DiffusionSolver::Grid(const unsigned int size, const int threadsPerBlock) {
    //TODO: remove hardcoded hardware capabilities :(
    // see: http://code.google.com/p/thrust/source/browse/thrust/detail/backend/cuda/arch.inl
    //   and http://code.google.com/p/thrust/source/browse/thrust/detail/backend/cuda/detail/safe_scan.inl
    //   for refactoring.
    // Get maximum grid size of CUDA device.
    //hipDevice_t device;
    //hipDeviceGet(&device, 0);
    //CUdevprop deviceProps;
    //cuDeviceGetProperties(&deviceProps, device);
    //this->gridSize = dim3(deviceProps.maxGridSize[0],
    //  deviceProps.maxGridSize[1],
    //  deviceProps.maxGridSize[2]);
    const dim3 maxGridSize(65535, 65535, 0);
    const int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    dim3 grid(blocksPerGrid, 1, 1);

    return grid;
}

#endif // WITH_CUDA
