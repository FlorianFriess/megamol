#include "hip/hip_runtime.h"
//
// DiffusionSolver.cpp
//
// Copyright (C) 2013 by University of Stuttgart (VISUS).
// All rights reserved.
//
// Created on : Sep 16, 2013
// Author     : scharnkn
//

#include "stdafx.h"
#include "DiffusionSolver.h"

#ifdef WITH_CUDA

#include "cuda_error_check.h"
#include "cuda_helper.h"

using namespace megamol;
using namespace megamol::protein;

#define USE_TIMER
//#define USE_CUDA_TIMER


/**
 * Samples the field at a given (integer) grid position.
 *
 * @param x,y,z Coordinates of the position
 * @return The sampled value of the field
 */
template <typename T>
inline __device__ T SampleFieldAt_D(uint x, uint y, uint z, T *field_D, uint3 dim) {
    return field_D[dim.x*(dim.y*z+y)+x];
}


/**
 * Answers the grid position index associated with the given coordinates.
 *
 * @param v0 The coordinates
 * @return The index
 */
inline __device__ uint GetPosIdxByGridCoords(uint3 v0, uint3 voldim) {
    return voldim.x*(voldim.y*v0.z + v0.y) + v0.x;
}

/**
 * Answers the cell index associated with the given coordinates.
 *
 * @param v0 The coordinates
 * @return The index
 */
inline __device__ uint GetCellIdxByGridCoords(int3 v0, uint3 voldim) {
    return (voldim.x-1)*((voldim.y-1)*v0.z + v0.y) + v0.x;
}

/**
 * Answers the grid position coordinates associated with a given cell index.
 * The returned position is the left/lower/back corner of the cell
 *
 * @param index The index
 * @return The coordinates
 */
inline __device__ uint3 GetGridCoordsByCellIdx(uint index, uint3 voldim) {
    return make_uint3(index % (voldim.x-1),
                      (index / (voldim.x-1)) % (voldim.y-1),
                      (index / (voldim.x-1)) / (voldim.y-1));
}

/**
 * Answers the cell coordinates associated with a given grid position index.
 *
 * @param index The index
 * @return The coordinates
 */
inline __device__ uint3 GetGridCoordsByPosIdx(uint index, uint3 voldim) {
    return make_uint3(index % voldim.x,
                      (index / voldim.x) % voldim.y,
                      (index / voldim.x) / voldim.y);
}


/*
 * updateGVF_D
 */
__global__ void updateGVF_D(
        float *gvfIn_D,
        float *gvfOut_D,
        float *gvfConstData_D, // b, c1, c2, c3
        float scl,
        uint3 voldim) {

    const uint idx = __umul24(__umul24(blockIdx.y, gridDim.x) +
            blockIdx.x, blockDim.x) + threadIdx.x;

    uint volsize = voldim.x*voldim.y*voldim.z;
    if (idx >= volsize) return;

    float3 gvf, gvfOld, gvfAdj[6];
    uint idxAdj[6];

    // Get grid coordinates
    uint3 gridC = make_uint3(
            idx % voldim.x,
            (idx / voldim.x) % voldim.y,
            (idx / voldim.x) / voldim.y);

    // Get const data
    float b = gvfConstData_D[4*idx+0];
    float c1 = gvfConstData_D[4*idx+1];
    float c2 = gvfConstData_D[4*idx+2];
    float c3 = gvfConstData_D[4*idx+3];

    /* Update isotropic diffusion for all vector components */

    // Get indices of adjacent values
    idxAdj[0] = ::GetPosIdxByGridCoords(make_uint3(uint(clamp(int(gridC.x)-1, 0, int(voldim.x-1))), gridC.y, gridC.z), voldim);
    idxAdj[1] = ::GetPosIdxByGridCoords(make_uint3(uint(clamp(int(gridC.x)+1, 0, int(voldim.x-1))), gridC.y, gridC.z), voldim);
    idxAdj[2] = ::GetPosIdxByGridCoords(make_uint3(gridC.x, uint(clamp(int(gridC.y)-1, 0, int(voldim.y-1))), gridC.z), voldim);
    idxAdj[3] = ::GetPosIdxByGridCoords(make_uint3(gridC.x, uint(clamp(int(gridC.y)+1, 0, int(voldim.y-1))), gridC.z), voldim);
    idxAdj[4] = ::GetPosIdxByGridCoords(make_uint3(gridC.x, gridC.y, uint(clamp(int(gridC.z)-1, 0, int(voldim.z-1)))), voldim);
    //idxAdj[4] = 0;
    idxAdj[5] = ::GetPosIdxByGridCoords(make_uint3(gridC.x, gridC.y, uint(clamp(int(gridC.z)+1, 0, int(voldim.z-1)))), voldim);

    // Get adjacent gvf values
    gvfOld = make_float3(gvfIn_D[4*idx+0], gvfIn_D[4*idx+1], gvfIn_D[4*idx+2]);
    gvfAdj[0] = make_float3(gvfIn_D[4*idxAdj[0]+0], gvfIn_D[4*idxAdj[0]+1], gvfIn_D[4*idxAdj[0]+2]);
    gvfAdj[1] = make_float3(gvfIn_D[4*idxAdj[1]+0], gvfIn_D[4*idxAdj[1]+1], gvfIn_D[4*idxAdj[1]+2]);
    gvfAdj[2] = make_float3(gvfIn_D[4*idxAdj[2]+0], gvfIn_D[4*idxAdj[2]+1], gvfIn_D[4*idxAdj[2]+2]);
    gvfAdj[3] = make_float3(gvfIn_D[4*idxAdj[3]+0], gvfIn_D[4*idxAdj[3]+1], gvfIn_D[4*idxAdj[3]+2]);
    gvfAdj[4] = make_float3(gvfIn_D[4*idxAdj[4]+0], gvfIn_D[4*idxAdj[4]+1], gvfIn_D[4*idxAdj[4]+2]);
    gvfAdj[5] = make_float3(gvfIn_D[4*idxAdj[5]+0], gvfIn_D[4*idxAdj[5]+1], gvfIn_D[4*idxAdj[5]+2]);

    // Compute diffusion
    gvf.x = (1.0-b)*gvfOld.x;
    gvf.x += (gvfAdj[0].x + gvfAdj[1].x + gvfAdj[2].x + gvfAdj[3].x +
              gvfAdj[4].x + gvfAdj[5].x -6*gvfOld.x)*scl;
    gvf.x += c1;

    gvf.y = (1.0-b)*gvfOld.y;
    gvf.y += (gvfAdj[0].y + gvfAdj[1].y + gvfAdj[2].y + gvfAdj[3].y +
            gvfAdj[4].y + gvfAdj[5].y -6*gvfOld.y)*scl;
    gvf.y += c2;

    gvf.z = (1.0-b)*gvfOld.z;
    gvf.z += (gvfAdj[0].z + gvfAdj[1].z + gvfAdj[2].z + gvfAdj[3].z +
            gvfAdj[4].z + gvfAdj[5].z -6*gvfOld.z)*scl;
    gvf.z += c3;

    float len = length(gvf);
    if (len > 0.0f) gvf /= len;

    //__syncthreads();
    gvfOut_D[4*idx+0] = gvf.x;
    gvfOut_D[4*idx+1] = gvf.y;
    gvfOut_D[4*idx+2] = gvf.z;
}


/*
 * DiffusionSolver::CalcGVF
 */
bool DiffusionSolver::CalcGVF(const float *startVol, float *gvfConstData_D,
        const unsigned int *cellStates_D,
        float *grad_D, size_t dim[3], float isovalue,
        float *gvfIn_D, float *gvfOut_D, unsigned int maxIt, float scl) {

    // Initialize the GVF field with the gradient in the starting regions
    if (!DiffusionSolver::initGVF(startVol, dim, cellStates_D, isovalue, grad_D,
            gvfConstData_D)) {
        return false;
    }

    uint volsize = dim[0]*dim[1]*dim[2];
    uint3 voldim = make_uint3(dim[0], dim[1], dim[2]);


    for (unsigned int it=(maxIt%2); it < maxIt+(maxIt%2); ++it) {

#ifdef USE_CUDA_TIMER
        float dt_ms;
        hipEvent_t event1, event2;
        hipEventCreate(&event1);
        hipEventCreate(&event2);
        hipEventRecord(event1, 0);
#endif
        if (it%2 == 0) {
            // Update diffusion
            updateGVF_D <<< DiffusionSolver::Grid(volsize, 256), 256 >>> (
                    gvfIn_D, gvfOut_D, gvfConstData_D, scl, voldim);

            if (hipGetLastError() != hipSuccess) {
                return false;
            }
        } else {
            // Update diffusion
            updateGVF_D <<< DiffusionSolver::Grid(volsize, 256), 256 >>> (
                    gvfOut_D, gvfIn_D, gvfConstData_D, scl, voldim);

            if (hipGetLastError() != hipSuccess) {
                return false;
            }
        }

#ifdef USE_CUDA_TIMER
        hipEventRecord(event2, 0);
        hipEventSynchronize(event1);
        hipEventSynchronize(event2);
        hipEventElapsedTime(&dt_ms, event1, event2);
        printf("CUDA time for 'updateGVF_D':                       %.10f sec\n",
                dt_ms/1000.0f);
        hipEventRecord(event1, 0);
#endif
    }

    return true;
}


/*
 * DiffusionSolver::CalcTwoWayGVF
 */
bool DiffusionSolver::CalcTwoWayGVF(
        const float *volSource_D,
        const float *volTarget_D,
        float *gvfConstData_D,
        const unsigned int *cellStatesSource_D,
        const unsigned int *cellStatesTarget_D,
        float *grad_D,
        size_t dim[3],
        float isovalue,
        float *gvfIn_D,
        float *gvfOut_D,
        unsigned int maxIt,
        float scl) {

    // Initialize the GVF field with the gradient in the starting regions
    if (!DiffusionSolver::initTwoWayGVF(volSource_D, volTarget_D, dim,
            cellStatesSource_D, cellStatesTarget_D, isovalue, grad_D,
            gvfConstData_D)) {
        return false;
    }

    uint volsize = dim[0]*dim[1]*dim[2];
    uint3 voldim = make_uint3(dim[0], dim[1], dim[2]);


    for (unsigned int it=(maxIt%2); it < maxIt+(maxIt%2); ++it) {

#ifdef USE_CUDA_TIMER
        float dt_ms;
        hipEvent_t event1, event2;
        hipEventCreate(&event1);
        hipEventCreate(&event2);
        hipEventRecord(event1, 0);
#endif
        if (it%2 == 0) {
            // Update diffusion
            updateGVF_D <<< DiffusionSolver::Grid(volsize, 256), 256 >>> (
                    gvfIn_D, gvfOut_D, gvfConstData_D, scl, voldim);

            if (hipGetLastError() != hipSuccess) {
                return false;
            }
        } else {
            // Update diffusion
            updateGVF_D <<< DiffusionSolver::Grid(volsize, 256), 256 >>> (
                    gvfOut_D, gvfIn_D, gvfConstData_D, scl, voldim);

            if (hipGetLastError() != hipSuccess) {
                return false;
            }
        }

#ifdef USE_CUDA_TIMER
        hipEventRecord(event2, 0);
        hipEventSynchronize(event1);
        hipEventSynchronize(event2);
        hipEventElapsedTime(&dt_ms, event1, event2);
        printf("CUDA time for 'updateGVF_D':                       %.10f sec\n",
                dt_ms/1000.0f);
        hipEventRecord(event1, 0);
#endif
    }

    return true;
}


/*
 * calcGradient_D
 * Computes the gradient of the given volume.
 */
__global__ void calcGradient_D(const float *vol_D, float *grad_D,
        const unsigned int *cellStates_D, uint3 voldim, float isovalue) {

    const uint idx = __umul24(__umul24(blockIdx.y, gridDim.x) +
            blockIdx.x, blockDim.x) + threadIdx.x;

    uint volsize = voldim.x*voldim.y*voldim.z;
    if (idx >= volsize) return;

    uint nCells = (voldim.x-1)*(voldim.y-1)*(voldim.z-1);

    int3 cellC;
    uint cellIdx;
    int active = 0;

    // Get grid coordinates
    int3 gridC = make_int3(
            idx % (voldim.x),
            (idx / voldim.x) % voldim.y,
            (idx / voldim.x) / voldim.y);

    int3 voldimI = make_int3(int(voldim.x), int(voldim.y), int(voldim.z));

    /* Check all eight adjacent cells */

    // (-1, -1, -1)
    cellC = make_int3(
            clamp(gridC.x-1, 0, voldimI.x-2),
            clamp(gridC.y-1, 0, voldimI.y-2),
            clamp(gridC.z-1, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    active |= cellStates_D[cellIdx];

    // (-1, 1, -1)
    cellC = make_int3(
            clamp(gridC.x-1, 0, voldimI.x-2),
            clamp(gridC.y, 0, voldimI.y-2),
            clamp(gridC.z-1, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    active |= cellStates_D[cellIdx];

    // (-1, 1, 1)
    cellC = make_int3(
            clamp(gridC.x-1, 0, voldimI.x-2),
            clamp(gridC.y, 0, voldimI.y-2),
            clamp(gridC.z, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    active |= cellStates_D[cellIdx];

    // (-1, -1, 1)
    cellC = make_int3(
            clamp(gridC.x-1, 0, voldimI.x-2),
            clamp(gridC.y-1, 0, voldimI.y-2),
            clamp(gridC.z, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    active |= cellStates_D[cellIdx];

    // (1, -1, -1)
    cellC = make_int3(
            clamp(gridC.x, 0, voldimI.x-2),
            clamp(gridC.y-1, 0, voldimI.y-2),
            clamp(gridC.z-1, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    active |= cellStates_D[cellIdx];

    // (1, 1, -1)
    cellC = make_int3(
            clamp(gridC.x, 0, voldimI.x-2),
            clamp(gridC.y, 0, voldimI.y-2),
            clamp(gridC.z-1, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    active |= cellStates_D[cellIdx];

    // (1, 1, 1)
    cellC = make_int3(
            clamp(gridC.x, 0, voldimI.x-2),
            clamp(gridC.y, 0, voldimI.y-2),
            clamp(gridC.z, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    active |= cellStates_D[cellIdx];

    // (1, -1, 1)
    cellC = make_int3(
            clamp(gridC.x, 0, voldimI.x-2),
            clamp(gridC.y-1, 0, voldimI.y-2),
            clamp(gridC.z, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    active |= cellStates_D[cellIdx];

    /* Sample gradient if necessary */

    if (active) {
        float3 grad;
        uint3 x1, x2;

        x1 = make_uint3(clamp(gridC.x+1, 0, voldimI.x-1), gridC.y, gridC.z);
        x2 = make_uint3(clamp(gridC.x-1, 0, voldimI.x-1), gridC.y, gridC.z);
        grad.x = vol_D[voldim.x*(voldim.y*x1.z + x1.y) + x1.x]-
                vol_D[voldim.x*(voldim.y*x2.z + x2.y) + x2.x];

        x1 = make_uint3(gridC.x, clamp(gridC.y+1, 0, voldimI.y-1), gridC.z);
        x2 = make_uint3(gridC.x, clamp(gridC.y-1, 0, voldimI.y-1), gridC.z);
        grad.y = vol_D[voldim.x*(voldim.y*x1.z + x1.y) + x1.x]-
                vol_D[voldim.x*(voldim.y*x2.z + x2.y) + x2.x];

        x1 = make_uint3(gridC.x, gridC.y, clamp(gridC.z+1, 0, voldimI.z-1));
        x2 = make_uint3(gridC.x, gridC.y, clamp(gridC.z-1, 0, voldimI.z-1));
        grad.z = vol_D[voldim.x*(voldim.y*x1.z + x1.y) + x1.x]-
                vol_D[voldim.x*(voldim.y*x2.z + x2.y) + x2.x];

        float len = length(grad);
        if (len > 0.0) grad/= len;

        grad_D[4*idx+0] = grad.x;
        grad_D[4*idx+1] = grad.y;
        grad_D[4*idx+2] = grad.z;
    } else {
        grad_D[4*idx+0] = 0.0;
        grad_D[4*idx+1] = 0.0;
        grad_D[4*idx+2] = 0.0;
    }
}


/*
 * prepareGVFDiffusion
 */
__global__ void prepareGVFDiffusion_D(
        float *grad_D,
        float *gvfConstData_D, // b, c1, c2, c3
        uint3 voldim) {

    const uint idx = __umul24(__umul24(blockIdx.y, gridDim.x) +
            blockIdx.x, blockDim.x) + threadIdx.x;

    uint volsize = voldim.x*voldim.y*voldim.z;
    if (idx >= volsize) return;

    // Get grid coordinates
    uint3 gridCoord = make_uint3(
            idx % voldim.x,
            (idx / voldim.x) % voldim.y,
            (idx / voldim.x) / voldim.y);

    // Get vector field
    float3 grad;
    grad.x = grad_D[4*idx+0];
    grad.y = grad_D[4*idx+1];
    grad.z = grad_D[4*idx+2];

    // Compute len^2
    float len = grad.x*grad.x + grad.y*grad.y + grad.z*grad.z;

    // Write b to device memory
    gvfConstData_D[4*idx+0] = len;
    // Write c1, c2, and c3 to device memory
    gvfConstData_D[4*idx+1] = len*grad.x;
    gvfConstData_D[4*idx+2] = len*grad.y;
    gvfConstData_D[4*idx+3] = len*grad.z;

}


/*
 * DiffusionSolver::initGVFCuda
 */
bool DiffusionSolver::initGVF(const float *startVol, size_t dim[3],
        const unsigned int *cellStates_D,
        float isovalue, float *grad_D, float *gvfConstData_D) {

    size_t volsize = dim[0]*dim[1]*dim[2];
    uint3 voldim = make_uint3(dim[0], dim[1], dim[2]);

#ifdef USE_CUDA_TIMER
    float dt_ms;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipEventRecord(event1, 0);
#endif

    // Calculate gradient using finite differences
    calcGradient_D <<< DiffusionSolver::Grid(volsize, 256), 256 >>> (
            startVol, grad_D, cellStates_D, voldim, isovalue);

#ifdef USE_CUDA_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("CUDA time for 'calcGradient_D':                        %.10f sec\n",
            dt_ms/1000.0f);
    hipEventRecord(event1, 0);
#endif

    // Precompute b,c1,c2, and c3
    prepareGVFDiffusion_D <<< DiffusionSolver::Grid(volsize, 256), 256 >>> (
            grad_D, gvfConstData_D, voldim);

#ifdef USE_CUDA_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("CUDA time for 'prepareGVFDiffusion_D':                 %.10f sec\n",
            dt_ms/1000.0f);
#endif

    return true;
}


/*
 * calcGradient_D
 * Computes the gradient of the given volume.
 */
__global__ void calcTwoWayGradient_D(
        const float *sourceVol_D,
        const float *targetVol_D,
        float *grad_D,
        const unsigned int *cellStatesSource_D,
        const unsigned int *cellStatesTarget_D,
        uint3 voldim,
        float isovalue) {

    const uint idx = __umul24(__umul24(blockIdx.y, gridDim.x) +
            blockIdx.x, blockDim.x) + threadIdx.x;

    uint volsize = voldim.x*voldim.y*voldim.z;
    if (idx >= volsize) return;

    uint nCells = (voldim.x-1)*(voldim.y-1)*(voldim.z-1);

    int3 cellC;
    uint cellIdx;
    int activeSource = 0, activeTarget=0;

    // Get grid coordinates
    int3 gridC = make_int3(
            idx % (voldim.x),
            (idx / voldim.x) % voldim.y,
            (idx / voldim.x) / voldim.y);

    int3 voldimI = make_int3(int(voldim.x), int(voldim.y), int(voldim.z));

    /* Check all eight adjacent cells in source volume */

    // (-1, -1, -1)
    cellC = make_int3(
            clamp(gridC.x-1, 0, voldimI.x-2),
            clamp(gridC.y-1, 0, voldimI.y-2),
            clamp(gridC.z-1, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    activeSource |= cellStatesSource_D[cellIdx];

    // (-1, 1, -1)
    cellC = make_int3(
            clamp(gridC.x-1, 0, voldimI.x-2),
            clamp(gridC.y, 0, voldimI.y-2),
            clamp(gridC.z-1, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    activeSource |= cellStatesSource_D[cellIdx];

    // (-1, 1, 1)
    cellC = make_int3(
            clamp(gridC.x-1, 0, voldimI.x-2),
            clamp(gridC.y, 0, voldimI.y-2),
            clamp(gridC.z, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    activeSource |= cellStatesSource_D[cellIdx];

    // (-1, -1, 1)
    cellC = make_int3(
            clamp(gridC.x-1, 0, voldimI.x-2),
            clamp(gridC.y-1, 0, voldimI.y-2),
            clamp(gridC.z, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    activeSource |= cellStatesSource_D[cellIdx];

    // (1, -1, -1)
    cellC = make_int3(
            clamp(gridC.x, 0, voldimI.x-2),
            clamp(gridC.y-1, 0, voldimI.y-2),
            clamp(gridC.z-1, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    activeSource |= cellStatesSource_D[cellIdx];

    // (1, 1, -1)
    cellC = make_int3(
            clamp(gridC.x, 0, voldimI.x-2),
            clamp(gridC.y, 0, voldimI.y-2),
            clamp(gridC.z-1, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    activeSource |= cellStatesSource_D[cellIdx];

    // (1, 1, 1)
    cellC = make_int3(
            clamp(gridC.x, 0, voldimI.x-2),
            clamp(gridC.y, 0, voldimI.y-2),
            clamp(gridC.z, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    activeSource |= cellStatesSource_D[cellIdx];

    // (1, -1, 1)
    cellC = make_int3(
            clamp(gridC.x, 0, voldimI.x-2),
            clamp(gridC.y-1, 0, voldimI.y-2),
            clamp(gridC.z, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    activeSource |= cellStatesSource_D[cellIdx];


    /* Check all adjacent cells in the target volume */

    // (-1, -1, -1)
    cellC = make_int3(
            clamp(gridC.x-1, 0, voldimI.x-2),
            clamp(gridC.y-1, 0, voldimI.y-2),
            clamp(gridC.z-1, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    activeTarget |= cellStatesTarget_D[cellIdx];

    // (-1, 1, -1)
    cellC = make_int3(
            clamp(gridC.x-1, 0, voldimI.x-2),
            clamp(gridC.y, 0, voldimI.y-2),
            clamp(gridC.z-1, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    activeTarget |= cellStatesTarget_D[cellIdx];

    // (-1, 1, 1)
    cellC = make_int3(
            clamp(gridC.x-1, 0, voldimI.x-2),
            clamp(gridC.y, 0, voldimI.y-2),
            clamp(gridC.z, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    activeTarget |= cellStatesTarget_D[cellIdx];

    // (-1, -1, 1)
    cellC = make_int3(
            clamp(gridC.x-1, 0, voldimI.x-2),
            clamp(gridC.y-1, 0, voldimI.y-2),
            clamp(gridC.z, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    activeTarget |= cellStatesTarget_D[cellIdx];

    // (1, -1, -1)
    cellC = make_int3(
            clamp(gridC.x, 0, voldimI.x-2),
            clamp(gridC.y-1, 0, voldimI.y-2),
            clamp(gridC.z-1, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    activeTarget |= cellStatesTarget_D[cellIdx];

    // (1, 1, -1)
    cellC = make_int3(
            clamp(gridC.x, 0, voldimI.x-2),
            clamp(gridC.y, 0, voldimI.y-2),
            clamp(gridC.z-1, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    activeTarget |= cellStatesTarget_D[cellIdx];

    // (1, 1, 1)
    cellC = make_int3(
            clamp(gridC.x, 0, voldimI.x-2),
            clamp(gridC.y, 0, voldimI.y-2),
            clamp(gridC.z, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    activeTarget |= cellStatesTarget_D[cellIdx];

    // (1, -1, 1)
    cellC = make_int3(
            clamp(gridC.x, 0, voldimI.x-2),
            clamp(gridC.y-1, 0, voldimI.y-2),
            clamp(gridC.z, 0, voldimI.z-2));
    cellIdx = ::GetCellIdxByGridCoords(cellC, voldim);
    activeTarget |= cellStatesTarget_D[cellIdx];


    /* Sample gradients  */

    float3 gradSource,gradTarget;
    uint3 x1, x2;

    x1 = make_uint3(clamp(gridC.x+1, 0, voldimI.x-1), gridC.y, gridC.z);
    x2 = make_uint3(clamp(gridC.x-1, 0, voldimI.x-1), gridC.y, gridC.z);
    gradSource.x = sourceVol_D[voldim.x*(voldim.y*x1.z + x1.y) + x1.x]-
            sourceVol_D[voldim.x*(voldim.y*x2.z + x2.y) + x2.x];

    x1 = make_uint3(gridC.x, clamp(gridC.y+1, 0, voldimI.y-1), gridC.z);
    x2 = make_uint3(gridC.x, clamp(gridC.y-1, 0, voldimI.y-1), gridC.z);
    gradSource.y = sourceVol_D[voldim.x*(voldim.y*x1.z + x1.y) + x1.x]-
            sourceVol_D[voldim.x*(voldim.y*x2.z + x2.y) + x2.x];

    x1 = make_uint3(gridC.x, gridC.y, clamp(gridC.z+1, 0, voldimI.z-1));
    x2 = make_uint3(gridC.x, gridC.y, clamp(gridC.z-1, 0, voldimI.z-1));
    gradSource.z = targetVol_D[voldim.x*(voldim.y*x1.z + x1.y) + x1.x]-
            targetVol_D[voldim.x*(voldim.y*x2.z + x2.y) + x2.x];

    float len = length(gradSource);
    if (len > 0.0) gradSource /= len;

    x1 = make_uint3(clamp(gridC.x+1, 0, voldimI.x-1), gridC.y, gridC.z);
    x2 = make_uint3(clamp(gridC.x-1, 0, voldimI.x-1), gridC.y, gridC.z);
    gradTarget.x = targetVol_D[voldim.x*(voldim.y*x1.z + x1.y) + x1.x]-
            targetVol_D[voldim.x*(voldim.y*x2.z + x2.y) + x2.x];

    x1 = make_uint3(gridC.x, clamp(gridC.y+1, 0, voldimI.y-1), gridC.z);
    x2 = make_uint3(gridC.x, clamp(gridC.y-1, 0, voldimI.y-1), gridC.z);
    gradTarget.y = targetVol_D[voldim.x*(voldim.y*x1.z + x1.y) + x1.x]-
            targetVol_D[voldim.x*(voldim.y*x2.z + x2.y) + x2.x];

    x1 = make_uint3(gridC.x, gridC.y, clamp(gridC.z+1, 0, voldimI.z-1));
    x2 = make_uint3(gridC.x, gridC.y, clamp(gridC.z-1, 0, voldimI.z-1));
    gradTarget.z = targetVol_D[voldim.x*(voldim.y*x1.z + x1.y) + x1.x]-
            targetVol_D[voldim.x*(voldim.y*x2.z + x2.y) + x2.x];

    len = length(gradTarget);
    if (len > 0.0) gradTarget /= len;


    /* Compute final gradient sample based on active cells */

    float3 gradFinal;
    gradFinal.x = activeSource*gradSource.x + activeTarget*gradTarget.x;
    gradFinal.y = activeSource*gradSource.y + activeTarget*gradTarget.y;
    gradFinal.z = activeSource*gradSource.z + activeTarget*gradTarget.z;

    len = length(gradFinal);
    if (len > 0.0) gradFinal /= len;

    grad_D[4*idx+0] = gradFinal.x;
    grad_D[4*idx+1] = gradFinal.y;
    grad_D[4*idx+2] = gradFinal.z;
}


/*
 * DiffusionSolver::initGVFCuda
 */
bool DiffusionSolver::initTwoWayGVF(
        const float *sourceVol_D,
        const float *targetVol_D,
        size_t dim[3],
        const unsigned int *cellStatesSource_D,
        const unsigned int *cellStatesTarget_D,
        float isovalue,
        float *grad_D,
        float *gvfConstData_D) {

    size_t volsize = dim[0]*dim[1]*dim[2];
    uint3 voldim = make_uint3(dim[0], dim[1], dim[2]);

#ifdef USE_CUDA_TIMER
    float dt_ms;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipEventRecord(event1, 0);
#endif

    // Calculate gradient using finite differences
    calcTwoWayGradient_D <<< DiffusionSolver::Grid(volsize, 256), 256 >>> (
            sourceVol_D,
            targetVol_D,
            grad_D,
            cellStatesSource_D,
            cellStatesTarget_D,
            voldim,
            isovalue);

#ifdef USE_CUDA_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("CUDA time for 'calcGradient_D':                        %.10f sec\n",
            dt_ms/1000.0f);
    hipEventRecord(event1, 0);
#endif

    // Precompute b,c1,c2, and c3
    prepareGVFDiffusion_D <<< DiffusionSolver::Grid(volsize, 256), 256 >>> (
            grad_D, gvfConstData_D, voldim);

#ifdef USE_CUDA_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("CUDA time for 'prepareGVFDiffusion_D':                 %.10f sec\n",
            dt_ms/1000.0f);
#endif

    return true;
}


/**
 * Returns a 1D grid definition based on the given threadsPerBlock value.
 *
 * @param size             The minimum number of threads
 * @param threadsPerBlock  The number of threads per block
 * @return The grid dimensions
 */
extern "C" dim3 DiffusionSolver::Grid(const unsigned int size, const int threadsPerBlock) {
    //TODO: remove hardcoded hardware capabilities :(
    // see: http://code.google.com/p/thrust/source/browse/thrust/detail/backend/cuda/arch.inl
    //   and http://code.google.com/p/thrust/source/browse/thrust/detail/backend/cuda/detail/safe_scan.inl
    //   for refactoring.
    // Get maximum grid size of CUDA device.
    //hipDevice_t device;
    //hipDeviceGet(&device, 0);
    //CUdevprop deviceProps;
    //cuDeviceGetProperties(&deviceProps, device);
    //this->gridSize = dim3(deviceProps.maxGridSize[0],
    //  deviceProps.maxGridSize[1],
    //  deviceProps.maxGridSize[2]);
    const dim3 maxGridSize(65535, 65535, 0);
    const int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    dim3 grid(blocksPerGrid, 1, 1);

    return grid;
}

#endif // WITH_CUDA
