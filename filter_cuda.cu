#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include <hip/hip_vector_types.h>
#include <cutil_math.h>

#include "filter_cuda.cuh"

// Parameters in constant memory
__constant__ FilterParams fparams;


/*
 * calcFilterHashGridD
 */
__global__
void calcFilterHashGridD(unsigned int *gridHash,  
                         unsigned int *gridIndex,
                         float3       *atmPos) {
                             
    unsigned int idx = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    
    if(idx >= fparams.atmCntProt) 
        return;
    
    int3 gridPos = make_int3(floor((atmPos[idx].x - fparams.worldOrigin.x) / fparams.cellSize.x),
                             floor((atmPos[idx].y - fparams.worldOrigin.y) / fparams.cellSize.y),
                             floor((atmPos[idx].z - fparams.worldOrigin.z) / fparams.cellSize.z));

    // Wrap grid, assumes size is power of 2
    //gridPos.x = gridPos.x & (fparams.gridSize.x - 1);  
    //gridPos.y = gridPos.y & (fparams.gridSize.y - 1);
    //gridPos.z = gridPos.z & (fparams.gridSize.z - 1);*/

    // Calculate hash value
    gridHash[idx]  = __umul24(__umul24(gridPos.z, fparams.gridSize.y), 
        fparams.gridSize.x) + __umul24(gridPos.y, fparams.gridSize.x) + gridPos.x;

    // Init index array 
    gridIndex[idx] = idx;
}


/*
 * reorderFilterDataD
 */
__global__
void reorderFilterDataD(unsigned int *cellStart,        
                        unsigned int *cellEnd,      
                        unsigned int *gridHash, 
                        unsigned int *gridIndex,
                        float3       *atmPos,
                        float3       *atmPosSorted) {
                        
    extern __shared__ unsigned int sharedHash[];    // blockSize + 1 elements
    
    unsigned int idx = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
    unsigned int hash;

    if(idx < fparams.atmCntProt) {
        hash = gridHash[idx];

        // Load hash data into shared memory so that we can look 
        // at neighboring particle's hash value without loading
        // two hash values per thread
        sharedHash[threadIdx.x+1] = hash;

        if(idx > 0 && threadIdx.x == 0) {
            // first thread in block must load neighbor particle hash
            sharedHash[0] = gridHash[idx - 1];
        }
    }

    __syncthreads();
    
    if(idx < fparams.atmCntProt) {
        
        // If this particle has a different cell index to the previous
        // particle then it must be the first particle in the cell,
        // so store the index of this particle in the cell.
        // As it isn't the first particle, it must also be the cell end of
        // the previous particle's cell
        
        if(idx == 0 || hash != sharedHash[threadIdx.x]) {
            cellStart[hash] = idx;
            if(idx > 0)
                cellEnd[sharedHash[threadIdx.x]] = idx;
        }

        if(idx == fparams.atmCntProt - 1) {
            cellEnd[hash] = idx + 1;
        }

        // Now use the sorted index to reorder the pos data
        atmPosSorted[idx] = atmPos[gridIndex[idx]];
        
        // macro does either global read or texture fetch
        //float4 pos = FETCH( oldPos, sortedIndex);       
    }
}


/*
 * calcSolventVisibilityD
 */
__global__
void calcSolventVisibilityD(unsigned int *cellStart,
                            unsigned int *cellEnd,
                            float3       *atmPos,
                            float3       *atmPosProtSorted,
                            bool         *isSolventAtom,
                            int          *atomVisibility) {

    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx >= fparams.atmCnt) {
        return;
    }
    
    // Non-solvent atoms are visible
    if(!isSolventAtom[idx]) {
        atomVisibility[idx] = 1;
        return;
    }
    
    unsigned int startIdx, endIdx, hash, j;
    int3 neighbourPos;
    int x,y,z;
    
    // Get position of the atom
    float3 p = atmPos[idx];

    int3 gridPos = make_int3(floor((p.x - fparams.worldOrigin.x) / fparams.cellSize.x),
                             floor((p.y - fparams.worldOrigin.y) / fparams.cellSize.y),
                             floor((p.z - fparams.worldOrigin.z) / fparams.cellSize.z));
    
    // Examine neighbouring cells within the given range
    for(z = -fparams.discRange.z; z <= fparams.discRange.z; z++) {
            
        neighbourPos.z = (gridPos.z + z);
        if((neighbourPos.z < 0) || (neighbourPos.z >= fparams.gridSize.z)) 
            continue;
        
        for(y = -fparams.discRange.y; y <= fparams.discRange.y; y++) {
            
            neighbourPos.y = (gridPos.y + y);
            if((neighbourPos.y < 0) || (neighbourPos.y >= fparams.gridSize.y)) 
                continue;
            
            for(x = -fparams.discRange.x; x <= fparams.discRange.x; x++) {
                
                neighbourPos.x = gridPos.x + x;
                if((neighbourPos.x < 0) || (neighbourPos.x >= fparams.gridSize.x)) 
                    continue;
                    
                hash = __umul24(__umul24(neighbourPos.z, fparams.gridSize.y), 
                           fparams.gridSize.x) + __umul24(neighbourPos.y, 
                           fparams.gridSize.x) + neighbourPos.x;
                           
                // Note: startIndex/endIndex are referring to the position in
                // the sorted array
                startIdx = cellStart[hash];
                
                if(startIdx == 0xffffffff) {
                   continue; // Cell is empty - continue with next cell
                }
                else {
                
                    // Note: startIndex/endIndex are referring to the position in
                    // the sorted array
                    endIdx = cellEnd[hash];
                    
                    // If cell contains non-solvent atoms and is within inner
                    // range the atom is visible
                    if((abs(neighbourPos.x) <= fparams.innerDiscRange) &&
                    (abs(neighbourPos.y) <= fparams.innerDiscRange) &&
                    (abs(neighbourPos.z) <= fparams.innerDiscRange)) {
                    
                        atomVisibility[idx] = 1;
                        return; 
                    }
                    else {
            
                        // Iterate over all atoms in this cell
                        for(j = startIdx; j < endIdx; j++) {
                            if(length(atmPosProtSorted[j] - p) <= fparams.solvRange) {
                                
                                atomVisibility[idx] = 1;
                                return; 
                            }
                        }
                    }
                }
            }
        }
    }
}


extern "C" {
 
    
    /*
     * setFilterParams
     */
    void setFilterParams(FilterParams *hostParams) {
        // Copy parameters to constant memory
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(fparams), hostParams, sizeof(FilterParams)));
    }


    /*
     * calcHashGrid
     */
    void calcFilterHashGrid(unsigned int *gridHash,
                            unsigned int *gridIndex,
                            float        *atmPosProt,
                            unsigned int  atmCntProt) {
        
        // Compute grid size
        unsigned int numThreads = min(256, atmCntProt);
        unsigned int numBlocks  = ceil((float)atmCntProt/(float)numThreads);
    
        // Execute the kernel
        calcFilterHashGridD <<< numBlocks, numThreads >>> (gridHash,
                                                           gridIndex,
                                                           (float3*) atmPosProt);
        
        cutilCheckMsg("calcFilterHashGridD");
    }


    /*
     * reorderFilterData
     */
    void reorderFilterData(unsigned int *cellStart,
                           unsigned int *cellEnd,
                           unsigned int *gridHash,
                           unsigned int *gridIndex,
                           float        *atmPosProt,
                           float        *atmPosProtSorted,
                           unsigned int  atmCntProt) {
  
        // Compute grid size
        unsigned int numThreads = min(256, atmCntProt);
        unsigned int numBlocks  = ceil((float)atmCntProt/(float)numThreads);
        
        // Compute memory size
        unsigned int memSize = sizeof(unsigned int)*(numThreads+1);
        
        // Execute kernel
        reorderFilterDataD <<< numBlocks, numThreads, memSize >>> (cellStart,
                                                                   cellEnd,
                                                                   gridHash,
                                                                   gridIndex,
                                                                   (float3*) atmPosProt,
                                                                   (float3*) atmPosProtSorted);
            
        cutilCheckMsg("reorderFilterDataD");
    }
                                       
    
    /*
     * calcSolventVisibility
     */
    void calcSolventVisibility(unsigned int *cellStart,
                               unsigned int *cellEnd,
                               float        *atmPos,
                               float        *atmPosProtSorted,
                               bool         *isSolventAtom,
                               int          *atomVisibility,
                               unsigned int  atmCnt) {
    
        // Compute grid size
        unsigned int numThreads = min(256, atmCnt);
        unsigned int numBlocks  = ceil((float)atmCnt/(float)numThreads);                                     
                                         
        // Execute kernel
        calcSolventVisibilityD <<< numBlocks, numThreads >>> (cellStart,
                                                              cellEnd,
                                                              (float3*) atmPos,
                                                              (float3*) atmPosProtSorted,
                                                              isSolventAtom,
                                                              atomVisibility);
        
        cutilCheckMsg("calcSolventVisibilityD");                                                              
    }

} // extern "C"
