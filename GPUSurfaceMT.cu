#include "hip/hip_runtime.h"
//
// GPUSurfaceMT.cpp
//
// Copyright (C) 2013 by University of Stuttgart (VISUS).
// All rights reserved.
//
// Created on : Sep 17, 2013
// Author     : scharnkn
//

#include "stdafx.h"

#include "GPUSurfaceMT.h"

#ifdef WITH_CUDA

#include "cuda_error_check.h"

#include "ComparativeSurfacePotentialRenderer.cuh"
#include "HostArr.h"
#include "sort_triangles.cuh"

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

using namespace megamol;
using namespace megamol::protein;


/*
 * GPUSurfaceMT::GPUSurfaceMT
 */
GPUSurfaceMT::GPUSurfaceMT() : AbstractGPUSurface() , neighboursReady(false) {
}


/*
 * GPUSurfaceMT::GPUSurfaceMT
 */
GPUSurfaceMT::GPUSurfaceMT(const GPUSurfaceMT& other) : AbstractGPUSurface(other) {

    // Copy GPU memory

    CudaSafeCall(this->cubeStates_D.Validate(other.cubeStates_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->cubeStates_D.Peek(),
            other.cubeStates_D.PeekConst(),
            this->cubeStates_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->cubeOffsets_D.Validate(other.cubeOffsets_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->cubeOffsets_D.Peek(),
            other.cubeOffsets_D.PeekConst(),
            this->cubeOffsets_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->cubeMap_D.Validate(other.cubeMap_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->cubeMap_D.Peek(),
            other.cubeMap_D.PeekConst(),
            this->cubeMap_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->cubeMapInv_D.Validate(other.cubeMapInv_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->cubeMapInv_D.Peek(),
            other.cubeMapInv_D.PeekConst(),
            this->cubeMapInv_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->vertexStates_D.Validate(other.vertexStates_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->vertexStates_D.Peek(),
            other.vertexStates_D.PeekConst(),
            this->vertexStates_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->activeVertexPos_D.Validate(other.activeVertexPos_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->activeVertexPos_D.Peek(),
            other.activeVertexPos_D.PeekConst(),
            this->activeVertexPos_D.GetCount()*sizeof(float3),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->vertexIdxOffs_D.Validate(other.vertexIdxOffs_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->vertexIdxOffs_D.Peek(),
            other.vertexIdxOffs_D.PeekConst(),
            this->vertexIdxOffs_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->vertexMap_D.Validate(other.vertexMap_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->vertexMap_D.Peek(),
            other.vertexMap_D.PeekConst(),
            this->vertexMap_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->vertexMapInv_D.Validate(other.vertexMapInv_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->vertexMapInv_D.Peek(),
            other.vertexMapInv_D.PeekConst(),
            this->vertexMapInv_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->vertexNeighbours_D.Validate(other.vertexNeighbours_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->vertexNeighbours_D.Peek(),
            other.vertexNeighbours_D.PeekConst(),
            this->vertexNeighbours_D.GetCount()*sizeof(int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->verticesPerTetrahedron_D.Validate(other.verticesPerTetrahedron_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->verticesPerTetrahedron_D.Peek(),
            other.verticesPerTetrahedron_D.PeekConst(),
            this->verticesPerTetrahedron_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->tetrahedronVertexOffsets_D.Validate(other.tetrahedronVertexOffsets_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->tetrahedronVertexOffsets_D.Peek(),
            other.tetrahedronVertexOffsets_D.PeekConst(),
            this->tetrahedronVertexOffsets_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->triangleCamDistance_D.Validate(other.triangleCamDistance_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->triangleCamDistance_D.Peek(),
            other.triangleCamDistance_D.PeekConst(),
            this->triangleCamDistance_D.GetCount()*sizeof(float),
            hipMemcpyDeviceToDevice));

    // The number of active cells
    this->activeCellCnt = other.activeCellCnt;

    // Check whether neighbors have been computed
    this->neighboursReady = other.neighboursReady;
}


/*
 * GPUSurfaceMT::~GPUSurfaceMT
 */
GPUSurfaceMT::~GPUSurfaceMT() {
}


/*
 * DeformableGPUSurfaceMT::ComputeVertexPositions
 */
bool GPUSurfaceMT::ComputeVertexPositions(
        float *volume_D,
        int3 volDim,
        float3 volOrg,
        float3 volDelta,
        float isovalue) {

    using vislib::sys::Log;

    size_t gridCellCnt = (volDim.x-1)*(volDim.y-1)*(volDim.z-1);


    /* Init grid parameters */

    if (!CudaSafeCall(InitVolume(
            make_uint3(volDim.x, volDim.y, volDim.z),
            volOrg,
            volDelta))) {
        return false;
    }

    if (!CudaSafeCall(InitVolume_surface_generation(
            make_uint3(volDim.x, volDim.y, volDim.z),
            volOrg,
            volDelta))) {
        return false;
    }

//    printf("Grid dims %u %u %u\n", volDim[0], volDim[1], volDim[2]);
//    printf("cell count %u\n", gridCellCnt);


    /* Find active grid cells */

    if (!CudaSafeCall(this->cubeStates_D.Validate(gridCellCnt))) {
        return false;
    }
    if (!CudaSafeCall(this->cubeOffsets_D.Validate(gridCellCnt))) {
        return false;
    }
    if (!CudaSafeCall(this->cubeStates_D.Set(0x00))) {
        return false;
    }
    if (!CudaSafeCall(this->cubeOffsets_D.Set(0x00))) {
        return false;
    }
    if (!CudaSafeCall(FindActiveGridCells(
            this->cubeStates_D.Peek(),
            this->cubeOffsets_D.Peek(),
            gridCellCnt,
            isovalue,
            volume_D))) {
        return false;
    }

//    // DEBUG Print Cube states and offsets
//    HostArr<unsigned int> cubeStates;
//    HostArr<unsigned int> cubeOffsets;
//    cubeStates.Validate(gridCellCnt);
//    cubeOffsets.Validate(gridCellCnt);
//    this->cubeStates_D.CopyToHost(cubeStates.Peek());
//    this->cubeOffsets_D.CopyToHost(cubeOffsets.Peek());
//    for (int i = 0; i < gridCellCnt; ++i) {
//        printf ("Cell %i: state %u, offs %u\n", i, cubeStates.Peek()[i],
//                cubeOffsets.Peek()[i]);
//    }
//    // END DEBUG


    /* Get number of active grid cells */

    this->activeCellCnt =
            this->cubeStates_D.GetAt(gridCellCnt-1) +
            this->cubeOffsets_D.GetAt(gridCellCnt-1);
    if (!CheckForCudaError()) {
        return false;
    }


//    printf("Active cell count %u\n", activeCellCnt); // DEBUG
//    printf("Reduction %f\n", 1.0 - static_cast<float>(activeCellCnt)/
//            static_cast<float>(gridCellCnt)); // DEBUG


    /* Prepare cube map */

    if (!CudaSafeCall(this->cubeMapInv_D.Validate(gridCellCnt))) {
        return false;
    }
    if (!CudaSafeCall(this->cubeMapInv_D.Set(0x00))) {
        return false;
    }
    if (!CudaSafeCall(this->cubeMap_D.Validate(this->activeCellCnt))) {
        return false;
    }
    if (!CudaSafeCall(CalcCubeMap(
            this->cubeMap_D.Peek(),
            this->cubeMapInv_D.Peek(),
            this->cubeOffsets_D.Peek(),
            this->cubeStates_D.Peek(),
            gridCellCnt))) {
        return false;
    }

//
//    // DEBUG Cube map
//    HostArr<unsigned int> cubeMap;
//    HostArr<unsigned int> cubeMapInv;
//    cubeMap.Validate(activeCellCnt);
//    cubeMapInv.Validate(gridCellCnt);
//    cubeMapInv_D.CopyToHost(cubeMapInv.Peek());
//    cubeMap_D.CopyToHost(cubeMap.Peek());
//    for (int i = 0; i < gridCellCnt; ++i) {
//        printf ("Cell %i: cubeMapInv %u\n", i, cubeMapInv.Peek()[i]);
//    }
//    for (int i = 0; i < activeCellCnt; ++i) {
//        printf ("Cell %i: cubeMap %u\n", i, cubeMap.Peek()[i]);
//    }
//    // END DEBUG


    /* Get vertex positions */

    if (!CudaSafeCall(this->vertexStates_D.Validate(7*this->activeCellCnt))) {
        return false;
    }
    if (!CudaSafeCall(this->activeVertexPos_D.Validate(7*this->activeCellCnt))) {
        return false;
    }
    if (!CudaSafeCall(this->vertexIdxOffs_D.Validate(7*this->activeCellCnt))) {
        return false;
    }
    if (!CudaSafeCall(this->vertexStates_D.Set(0x00))) {
        return false;
    }
    if (!CudaSafeCall(this->activeVertexPos_D.Set(0x00))) {
        return false;
    }
    if (!CudaSafeCall(this->vertexIdxOffs_D.Set(0x00))) {
        return false;
    }
    if (!CudaSafeCall(CalcVertexPositions(
            this->vertexStates_D.Peek(),
            this->activeVertexPos_D.Peek(),
            this->vertexIdxOffs_D.Peek(),
            this->cubeMap_D.Peek(),
            this->activeCellCnt,
            isovalue,
            volume_D))) {
        return false;
    }

//    // DEBUG Print active vertex positions
//    HostArr<float3> activeVertexPos;
//    HostArr<unsigned int> vertexStates;
//    HostArr<unsigned int> vertexIdxOffsets;
//    activeVertexPos.Validate(7*this->activeCellCnt);
//    vertexIdxOffsets.Validate(7*this->activeCellCnt);
//    vertexStates.Validate(7*activeCellCnt);
//    hipMemcpy(vertexStates.Peek(), this->vertexStates_D.Peek(), 7*activeCellCnt*sizeof(unsigned int),
//            hipMemcpyDeviceToHost);
//    hipMemcpy(activeVertexPos.Peek(), this->activeVertexPos_D.Peek(), 7*activeCellCnt*sizeof(unsigned int),
//            hipMemcpyDeviceToHost);
//    hipMemcpy(vertexIdxOffsets.Peek(), this->vertexIdxOffs_D.Peek(), 7*activeCellCnt*sizeof(unsigned int),
//            hipMemcpyDeviceToHost);
//    for (int i = 0; i < 7*this->activeCellCnt; ++i) {
//        printf("#%i: active vertexPos %f %f %f (state = %u)\n", i,
//                activeVertexPos.Peek()[i].x,
//                activeVertexPos.Peek()[i].y,
//                activeVertexPos.Peek()[i].z,
//                vertexStates.Peek()[i]);
//    }

//    for (int i = 0; i < 7*this->activeCellCnt; ++i) {
//        printf("#%i: vertex index offset %u (state %u)\n",i,
//                vertexIdxOffsets.Peek()[i],
//                vertexStates.Peek()[i]);
//    }
    // END DEBUG


    /* Get number of active vertices */

    this->vertexCnt =
            this->vertexStates_D.GetAt(7*this->activeCellCnt-1) +
            this->vertexIdxOffs_D.GetAt(7*this->activeCellCnt-1);
    if (!CheckForCudaError()) {
        return false;
    }

//    printf("Vertex Cnt %u\n", this->vertexCnt);

    /* Create vertex buffer object and register with CUDA */

    // Create empty vbo to hold vertex data for the surface
    if (!this->InitVertexDataVBO(this->vertexCnt)) {
        return false;
    }

    // Register memory with CUDA
    if (!CudaSafeCall(hipGraphicsGLRegisterBuffer(
            &this->vertexDataResource, this->vboVtxData,
            cudaGraphicsMapFlagsNone))) {
        return false;
    }

    // Get mapped pointer to the vbo
    float *vboPt;
    size_t vboSize;
    if (!CudaSafeCall(hipGraphicsMapResources(1, &this->vertexDataResource, 0))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsResourceGetMappedPointer(
            reinterpret_cast<void**>(&vboPt), // The mapped pointer
            &vboSize,             // The size of the accessible data
            this->vertexDataResource))) {                   // The mapped resource
        // Unmap CUDA graphics resource
        if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->vertexDataResource))) {
            return false;
        }
        return false;
    }

    // Init with zeros
    if (!CudaSafeCall(hipMemset(vboPt, 0, vboSize))) {
        // Unmap CUDA graphics resource
        if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->vertexDataResource))) {
            return false;
        }
        if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
            return false;
        }
        return false;
    }

//    printf("Got VBO of size %u\n", vboSize);


    /* Compact list of vertex positions (keep only active vertices) */

    if (!CudaSafeCall(CompactActiveVertexPositions(
            vboPt,
            this->vertexStates_D.Peek(),
            this->vertexIdxOffs_D.Peek(),
            this->activeVertexPos_D.Peek(),
            this->activeCellCnt,
            this->vertexDataOffsPos,  // Array data byte offset
            this->vertexDataStride    // Array data element size
            ))) {
        // Unmap CUDA graphics resource
        if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->vertexDataResource))) {
            return false;
        }
        if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
            return false;
        }
        return false;
    }

//    // DEBUG Print vertex positions
//    HostArr<float> vertexPos;
//    vertexPos.Validate(this->vertexCnt*this->vertexDataStride);
//    hipMemcpy(vertexPos.Peek(), vboPt, this->vertexCnt*this->vertexDataStride*sizeof(float),
//            hipMemcpyDeviceToHost);
//    for (int i = 0; i < this->vertexCnt; ++i) {
//        printf("#%i: vertexPos %f %f %f\n", i, vertexPos.Peek()[9*i+0],
//                vertexPos.Peek()[9*i+1], vertexPos.Peek()[9*i+2]);
//    }
//    // END DEBUG

    // Unmap CUDA graphics resource
    if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->vertexDataResource))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
        return false;
    }

    return true;
}


/*
 * DeformableGPUSurfaceMT::computeTriangles
 */
bool GPUSurfaceMT::ComputeTriangles(
        float *volume_D,
        int3 volDim,
        float3 volOrg,
        float3 volDelta,
        float isovalue) {

    if (!this->vertexDataReady) { // We need vertex data to generate triangles
        return false;
    }

    size_t triangleVtxCnt;

    /* Calc vertex index map */

    if (!CudaSafeCall(this->vertexMap_D.Validate(this->vertexCnt))) {
        return false;
    }
    if (!CudaSafeCall(this->vertexMapInv_D.Validate(7*this->activeCellCnt))) {
        return false;
    }
    if (!CudaSafeCall(this->vertexMapInv_D.Set(0xff))) {
        return false;
    }
    if (!CudaSafeCall(CalcVertexMap(
            this->vertexMap_D.Peek(),
            this->vertexMapInv_D.Peek(),
            this->vertexIdxOffs_D.Peek(),
            this->vertexStates_D.Peek(),
            this->activeCellCnt))) {
        return false;
    }

//    // DEBUG Print vertex map
//    HostArr<unsigned int> vertexMap;
//    vertexMap.Validate(this->vertexCnt);
//    vertexMap_D.CopyToHost(vertexMap.Peek());
//    for (int i = 0; i < this->vertexMap_D.GetCount(); ++i) {
//        printf("Vertex mapping %i: %u\n", i, vertexMap.Peek()[i]);
//    }
//    // END DEBUG
//
//    // DEBUG Print vertex map
//    HostArr<unsigned int> vertexMapInv;
//    vertexMapInv.Validate(this->vertexMapInv_D.GetCount());
//    vertexMapInv_D.CopyToHost(vertexMapInv.Peek());
//    for (int i = 0; i < this->vertexMapInv_D.GetCount(); ++i) {
//        printf("Inverse Vertex mapping %i: %u\n", i, vertexMapInv.Peek()[i]);
//    }
//    // END DEBUG


    /* Flag tetrahedrons */

    if (!CudaSafeCall(this->verticesPerTetrahedron_D.Validate(6*this->activeCellCnt))) return false;
    if (!CudaSafeCall(FlagTetrahedrons(
            this->verticesPerTetrahedron_D.Peek(),
            this->cubeMap_D.Peek(),
            isovalue,
            this->activeCellCnt,
            volume_D))) {
        return false;
    }


    /* Scan tetrahedrons */

    if (!CudaSafeCall(this->tetrahedronVertexOffsets_D.Validate(6*activeCellCnt))) return false;
    if (!CudaSafeCall(GetTetrahedronVertexOffsets(
            this->tetrahedronVertexOffsets_D.Peek(),
            this->verticesPerTetrahedron_D.Peek(),
            activeCellCnt*6))) {
        return false;
    }


    /* Get triangle vertex count */

    triangleVtxCnt =
            this->tetrahedronVertexOffsets_D.GetAt(activeCellCnt*6-1) +
            this->verticesPerTetrahedron_D.GetAt(activeCellCnt*6-1);
    if (!CheckForCudaError()) {
        return false;
    }

//    printf("Triangle cnt %u\n", triangleVtxCnt);

    this->triangleCnt = triangleVtxCnt/3;

    /* Create vertex buffer object and register with CUDA */

    // Create empty vbo to hold the triangle indices
    if (!this->InitTriangleIdxVBO(this->triangleCnt)) {
        return false;
    }

    // Register memory with CUDA
    if (!CudaSafeCall(hipGraphicsGLRegisterBuffer(
            &this->triangleIdxResource,
            this->vboTriangleIdx,
            cudaGraphicsMapFlagsNone))) {
        return false;
    }

    // Get mapped pointer to the vbo
    unsigned int *vboTriangleIdxPt;
    size_t vboTriangleIdxSize;
    if (!CudaSafeCall(hipGraphicsMapResources(1, &this->triangleIdxResource, 0))) {

        // Unmap CUDA graphics resource
        if (!CudaSafeCall(hipGraphicsUnregisterResource(this->triangleIdxResource))) {
            return false;
        }
        return false;
    }

    if (!CudaSafeCall(hipGraphicsResourceGetMappedPointer(
            reinterpret_cast<void**>(&vboTriangleIdxPt), // The mapped pointer
            &vboTriangleIdxSize,             // The size of the accessible data
            this->triangleIdxResource))) {                   // The mapped resource

        // Unmap CUDA graphics resource
        if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->triangleIdxResource))) {
            return false;
        }
        if (!CudaSafeCall(hipGraphicsUnregisterResource(this->triangleIdxResource))) {
            return false;
        }
        return false;
    }


    /* Generate triangles */

    if (!CudaSafeCall(hipMemset(vboTriangleIdxPt, 0x00, vboTriangleIdxSize))) {

        // Unmap CUDA graphics resource
        if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->triangleIdxResource))) {
            return false;
        }
        if (!CudaSafeCall(hipGraphicsUnregisterResource(this->triangleIdxResource))) {
            return false;
        }
        return false;
    }

    if (!CudaSafeCall(GetTrianglesIdx(
            this->tetrahedronVertexOffsets_D.Peek(),
            this->cubeMap_D.Peek(),
            this->cubeMapInv_D.Peek(),
            isovalue,
            this->activeCellCnt*6,
            this->activeCellCnt,
            vboTriangleIdxPt,
            this->vertexMapInv_D.Peek(),
            volume_D))) {

        // Unmap CUDA graphics resource
        if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->triangleIdxResource))) {
            return false;
        }
        if (!CudaSafeCall(hipGraphicsUnregisterResource(this->triangleIdxResource))) {
            return false;
        }
        return false;
    }

    // Unmap CUDA graphics resource
    if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->triangleIdxResource))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsUnregisterResource(this->triangleIdxResource))) {
        return false;
    }
    return true;
}


/*
 * GPUSurfaceMT::computeVertexNormals
 */
bool GPUSurfaceMT::ComputeNormals(
        float *volume_D,
        int3 volDim,
        float3 volOrg,
        float3 volDelta,
        float isovalue) {

    using vislib::sys::Log;

    if (!this->triangleIdxReady) { // We need the triangles mesh info
        Log::DefaultLog.WriteMsg(Log::LEVEL_ERROR,
                "%s: triangles not computed",
                this->ClassName());
        return false;
    }

    CheckForCudaErrorSync();

    /* Init grid parameters */

    if (!CudaSafeCall(InitVolume_surface_generation(
            make_uint3(volDim.x, volDim.y, volDim.z),
            volOrg,
            volDelta))) {

        Log::DefaultLog.WriteMsg(Log::LEVEL_ERROR,
                "%s: could not init device constants",
                this->ClassName());

        return false;
    }

//        printf("Init volume surface generation\n");
//        printf("grid size  %u %u %u\n", volDim[0], volDim[1], volDim[2]);
//        printf("grid org   %f %f %f\n", volWSOrg[0], volWSOrg[1], volWSOrg[2]);
//        printf("grid delta %f %f %f\n", volWSDelta[0], volWSDelta[1], volWSDelta[2]);

    // Register memory with CUDA
    if (!CudaSafeCall(hipGraphicsGLRegisterBuffer(
            &this->vertexDataResource, this->vboVtxData,
            cudaGraphicsMapFlagsNone))) {

        Log::DefaultLog.WriteMsg(Log::LEVEL_ERROR,
                "%s: could not register vertex buffer",
                this->ClassName());

        return false;
    }

    // Get mapped pointer to the vbo
    float *vboPt;
    size_t vboSize;
    if (!CudaSafeCall(hipGraphicsMapResources(1, &this->vertexDataResource, 0))) {
        Log::DefaultLog.WriteMsg(Log::LEVEL_ERROR,
                "%s: could not map resources",
                this->ClassName());
        return false;
    }
    if (!CudaSafeCall(hipGraphicsResourceGetMappedPointer(
            reinterpret_cast<void**>(&vboPt), // The mapped pointer
            &vboSize,             // The size of the accessible data
            this->vertexDataResource))) {
        Log::DefaultLog.WriteMsg(Log::LEVEL_ERROR,
                "%s: could not acquire mapped pointer",
                this->ClassName());
        return false;
    }



//    int cnt = 0;
//    // DEBUG Print vertex map
//    HostArr<unsigned int> vertexMap;
//    vertexMap.Validate(this->vertexCnt);
//    if (!CudaSafeCall(vertexMap_D.CopyToHost(vertexMap.Peek()))) {
//        return false;
//    }
//    for (int i = 0; i < this->vertexMap_D.GetCount(); ++i) {
//        printf("Vertex mapping %i: %u\n", i, vertexMap.Peek()[i]);
////        cnt += vertexMap.Peek()[i];
//    }
//    // END DEBUG
//
//    // DEBUG Print vertex map
//    HostArr<unsigned int> vertexMapInv;
//    vertexMapInv.Validate(this->vertexMapInv_D.GetCount());
//    if (!CudaSafeCall(vertexMapInv_D.CopyToHost(vertexMapInv.Peek()))) {
//        return false;
//    }
//    for (int i = 0; i < this->vertexMapInv_D.GetCount(); ++i) {
//        printf("Inverse Vertex mapping %i: %u\n", i, vertexMapInv.Peek()[i]);
////        cnt += vertexMapInv.Peek()[i];
//    }
//    // END DEBUG

//    printf("active vertex count %u\n", this->vertexCnt);
//    printf("active cube count %u\n", this->activeCellCnt);
//    printf("normals vbo %u\n", vboSize);
//    printf("vertexMap size %u\n", this->vertexMap_D.GetCount());
//    printf("vertexMapInv size %u\n", this->vertexMapInv_D.GetCount());
//    printf("cubeMap_D size %u\n", this->cubeMap_D.GetCount());
//    printf("cubeMapInv_D size %u\n", this->cubeMapInv_D.GetCount());

//        // DEBUG Print buffer content
//        HostArr<float> vertexBuffer;
//        vertexBuffer.Validate(this->vertexDataStride*this->vertexCnt*sizeof(float));
//        if (!CudaSafeCall(hipMemcpy(vertexBuffer.Peek(), vboPt,
//                this->vertexDataStride*this->vertexCnt*sizeof(float), hipMemcpyDeviceToHost))) {
//            return false;
//        }
//        for (int i = 0; i < this->vertexCnt; ++i) {
//    //        if (uint(abs(vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+0]))>= this->vertexCnt) {
//            printf("%i: pos %f %f %f, normal %f %f %f, texcoord %f %f %f\n", i,
//                    vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsPos+0],
//                    vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsPos+1],
//                    vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsPos+2],
//                    vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+0],
//                    vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+1],
//                    vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+2],
//                    vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsTexCoord+0],
//                    vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsTexCoord+1],
//                    vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsTexCoord+2],
//                    this->vertexCnt);
//    //        }
//        }
//        vertexBuffer.Release();
//        // end DEBUG

    if (!CudaSafeCall(ComputeVertexNormals(
            vboPt,
            this->vertexMap_D.Peek(),
            this->vertexMapInv_D.Peek(),
            this->cubeMap_D.Peek(),
            this->cubeMapInv_D.Peek(),
            volume_D,
            isovalue,
            this->vertexCnt,
            this->vertexDataOffsPos,
            this->vertexDataOffsNormal,
            this->vertexDataStride))) {

        return false;
    }

//    // DEBUG Print normals
//    HostArr<float> vertexBuffer;
//    vertexBuffer.Validate(this->vertexDataStride*this->vertexCnt*sizeof(float));
//    if (!CudaSafeCall(hipMemcpy(vertexBuffer.Peek(), vboPt,
//            this->vertexDataStride*this->vertexCnt*sizeof(float), hipMemcpyDeviceToHost))) {
//        return false;
//    }
//    for (int i = 0; i < this->vertexCnt; i+=3) {
////        if (uint(abs(vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+0]))>= this->vertexCnt) {
//                    printf("%i: pos %f %f %f, normal %f %f %f, texcoord %f %f %f\n", i,
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsPos+0],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsPos+1],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsPos+2],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+0],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+1],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+2],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsTexCoord+0],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsTexCoord+1],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsTexCoord+2],
//                            this->vertexCnt);
////        }
//    }
//    vertexBuffer.Release();
//    // end DEBUG

    // Unmap CUDA graphics resource
    if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->vertexDataResource))) {
        Log::DefaultLog.WriteMsg(Log::LEVEL_ERROR,
                "%s: could not unmap resources",
                this->ClassName());
        return false;
    }
    if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
        Log::DefaultLog.WriteMsg(Log::LEVEL_ERROR,
                "%s: could not unregister buffers",
                this->ClassName());
        return false;
    }

    return true;
}


/*
 * DeformableGPUSurfaceMT::ComputeTexCoords
 */
bool GPUSurfaceMT::ComputeTexCoords(float minCoords[3], float maxCoords[3]) {
    if (!this->triangleIdxReady) { // We need the triangles mesh info
        return false;
    }

    // Register memory with CUDA
    if (!CudaSafeCall(hipGraphicsGLRegisterBuffer(
            &this->vertexDataResource, this->vboVtxData,
            cudaGraphicsMapFlagsNone))) {
        return false;
    }

    // Get mapped pointer to the vbo
    float *vboPt;
    size_t vboSize;
    CudaSafeCall(hipGraphicsMapResources(1, &this->vertexDataResource, 0));
    CudaSafeCall(hipGraphicsResourceGetMappedPointer(
            reinterpret_cast<void**>(&vboPt), // The mapped pointer
            &vboSize,             // The size of the accessible data
            this->vertexDataResource));                   // The mapped resource

    if (!CudaSafeCall(ComputeVertexTexCoords(
            vboPt,
            minCoords[0],
            minCoords[1],
            minCoords[2],
            maxCoords[0],
            maxCoords[1],
            maxCoords[2],
            this->vertexCnt,
            this->vertexDataOffsPos,
            this->vertexDataOffsTexCoord,
            this->vertexDataStride))) {

        // Unmap CUDA graphics resource
        if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->vertexDataResource))) {
            return false;
        }
        if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
            return false;
        }

        return false;
    }

    // Unmap CUDA graphics resource
    if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->vertexDataResource))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
        return false;
    }

    return true;
}


/*
 * GPUSurfaceMT::Rotate
 */
bool GPUSurfaceMT::Rotate(float rotMat[9]) {
    CudaDevArr<float> rotate_D;

    // Register memory with CUDA
    if (!CudaSafeCall(hipGraphicsGLRegisterBuffer(
            &this->vertexDataResource, this->vboVtxData,
            cudaGraphicsMapFlagsNone))) {
        return false;
    }

    // Get mapped pointer to the vbo
    float *vboPt;
    size_t vboSize;
    if (!CudaSafeCall(hipGraphicsMapResources(1, &this->vertexDataResource, 0))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsResourceGetMappedPointer(
            reinterpret_cast<void**>(&vboPt), // The mapped pointer
            &vboSize,                         // The size of the accessible data
            this->vertexDataResource))) {     // The mapped resource
        return false;
    }

    // Rotate for best fit
    rotate_D.Validate(9);
    if (!CudaSafeCall(hipMemcpy((void *)rotate_D.Peek(), &rotMat[0],
            9*sizeof(float), hipMemcpyHostToDevice))) {
        return false;
    }
    if (!CudaSafeCall(RotatePos(
            vboPt,
            this->vertexDataStride,
            this->vertexDataOffsPos,
            rotate_D.Peek(),
            vertexCnt))) {
        return false;
    }

    // Clean up
    rotate_D.Release();

    // Unmap CUDA graphics resource
    if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->vertexDataResource))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
        return false;
    }

    return true;
}


/*
 * GPUSurfaceMT::SortTrianglesByCamDist
 */
bool GPUSurfaceMT::SortTrianglesByCamDist(float camPos[3]) {

    if (!CudaSafeCall(this->triangleCamDistance_D.Validate(triangleCnt))) {
        return false;
    }

    // Register memory with CUDA
    if (!CudaSafeCall(hipGraphicsGLRegisterBuffer(
            &this->vertexDataResource, this->vboVtxData,
            cudaGraphicsMapFlagsNone))) {
        return false;
    }
    // Register memory with CUDA
    if (!CudaSafeCall(hipGraphicsGLRegisterBuffer(
            &this->triangleIdxResource,
            this->vboTriangleIdx,
            cudaGraphicsMapFlagsNone))) {
        return false;
    }

    // We need both cuda graphics resources to be mapped at the same time
    hipGraphicsResource *cudaToken[2];
    cudaToken[0] = this->vertexDataResource;
    cudaToken[1] = this->triangleIdxResource;
    if (!CudaSafeCall(hipGraphicsMapResources(2, cudaToken, 0))) {
        return false;
    }

    // Get mapped pointers to the vertex data and the triangle indices
    float *vboPt;
    uint *vboTriangleIdxPt;
    size_t vboSize, vboTriangleIdxSize;
    if (!CudaSafeCall(hipGraphicsResourceGetMappedPointer(
            reinterpret_cast<void**>(&vboPt), // The mapped pointer
            &vboSize,                         // The size of the accessible data
            cudaToken[0]))) {                 // The mapped resource
        return false;
    }
    if (!CudaSafeCall(hipGraphicsResourceGetMappedPointer(
            reinterpret_cast<void**>(&vboTriangleIdxPt), // The mapped pointer
            &vboTriangleIdxSize,              // The size of the accessible data
            cudaToken[1]))) {                 // The mapped resource
        return false;
    }

    if (!CudaSafeCall(SortTrianglesByCamDistance(
            vboPt,
            this->vertexDataStride,
            this->vertexDataOffsPos,
            make_float3(camPos[0], camPos[1], camPos[2]),
            vboTriangleIdxPt,
            this->triangleCnt,
            this->triangleCamDistance_D.Peek()))) {

        // Unmap CUDA graphics resource
        if (!CudaSafeCall(hipGraphicsUnmapResources(2, cudaToken, 0))) {
            return false;
        }
        if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
            return false;
        }
        if (!CudaSafeCall(hipGraphicsUnregisterResource(this->triangleIdxResource))) {
            return false;
        }

        return false;
    }

    // Unmap CUDA graphics resource
    if (!CudaSafeCall(hipGraphicsUnmapResources(2, cudaToken, 0))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsUnregisterResource(this->triangleIdxResource))) {
        return false;
    }


    return true;
}


/*
 * GPUSurfaceMT::Translate
 */
bool GPUSurfaceMT::Translate(float transVec[3]) {

    // Register memory with CUDA
    if (!CudaSafeCall(hipGraphicsGLRegisterBuffer(
            &this->vertexDataResource, this->vboVtxData,
            cudaGraphicsMapFlagsNone))) {
        return false;
    }

    // Get mapped pointer to the vbo
    float *vboPt;
    size_t vboSize;
    if (!CudaSafeCall(hipGraphicsMapResources(1, &this->vertexDataResource, 0))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsResourceGetMappedPointer(
            reinterpret_cast<void**>(&vboPt), // The mapped pointer
            &vboSize,                         // The size of the accessible data
            this->vertexDataResource))) {     // The mapped resource
        return false;
    }

    // Move vertex positions to origin (with respect to centroid)
    if (!CudaSafeCall(TranslatePos(
            vboPt,
            this->vertexDataStride,
            this->vertexDataOffsPos,
            make_float3(transVec[0], transVec[0], transVec[0]),
            this->vertexCnt))) {
        return false;
    }

    // Unmap CUDA graphics resource
    if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->vertexDataResource, 0))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
        return false;
    }

    return true;
}


/*
 * GPUSurfaceMT::operator=
 */
GPUSurfaceMT& GPUSurfaceMT::operator=(const GPUSurfaceMT &rhs) {
    AbstractGPUSurface::operator=(rhs);

    // Copy GPU memory

    CudaSafeCall(this->cubeStates_D.Validate(rhs.cubeStates_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->cubeStates_D.Peek(),
            rhs.cubeStates_D.PeekConst(),
            this->cubeStates_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->cubeOffsets_D.Validate(rhs.cubeOffsets_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->cubeOffsets_D.Peek(),
            rhs.cubeOffsets_D.PeekConst(),
            this->cubeOffsets_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->cubeMap_D.Validate(rhs.cubeMap_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->cubeMap_D.Peek(),
            rhs.cubeMap_D.PeekConst(),
            this->cubeMap_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->cubeMapInv_D.Validate(rhs.cubeMapInv_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->cubeMapInv_D.Peek(),
            rhs.cubeMapInv_D.PeekConst(),
            this->cubeMapInv_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->vertexStates_D.Validate(rhs.vertexStates_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->vertexStates_D.Peek(),
            rhs.vertexStates_D.PeekConst(),
            this->vertexStates_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->activeVertexPos_D.Validate(rhs.activeVertexPos_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->activeVertexPos_D.Peek(),
            rhs.activeVertexPos_D.PeekConst(),
            this->activeVertexPos_D.GetCount()*sizeof(float3),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->vertexIdxOffs_D.Validate(rhs.vertexIdxOffs_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->vertexIdxOffs_D.Peek(),
            rhs.vertexIdxOffs_D.PeekConst(),
            this->vertexIdxOffs_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->vertexMap_D.Validate(rhs.vertexMap_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->vertexMap_D.Peek(),
            rhs.vertexMap_D.PeekConst(),
            this->vertexMap_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->vertexMapInv_D.Validate(rhs.vertexMapInv_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->vertexMapInv_D.Peek(),
            rhs.vertexMapInv_D.PeekConst(),
            this->vertexMapInv_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->vertexNeighbours_D.Validate(rhs.vertexNeighbours_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->vertexNeighbours_D.Peek(),
            rhs.vertexNeighbours_D.PeekConst(),
            this->vertexNeighbours_D.GetCount()*sizeof(int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->verticesPerTetrahedron_D.Validate(rhs.verticesPerTetrahedron_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->verticesPerTetrahedron_D.Peek(),
            rhs.verticesPerTetrahedron_D.PeekConst(),
            this->verticesPerTetrahedron_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->tetrahedronVertexOffsets_D.Validate(rhs.tetrahedronVertexOffsets_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->tetrahedronVertexOffsets_D.Peek(),
            rhs.tetrahedronVertexOffsets_D.PeekConst(),
            this->tetrahedronVertexOffsets_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

//    CudaSafeCall(this->triangleCamDistance_D.Validate(rhs.triangleCamDistance_D.GetCount()));
//    CudaSafeCall(hipMemcpy(
//            this->triangleCamDistance_D.Peek(),
//            rhs.triangleCamDistance_D.PeekConst(),
//            this->triangleCamDistance_D.GetCount()*sizeof(float),
//            hipMemcpyDeviceToDevice));

    // The number of active cells
    this->activeCellCnt = rhs.activeCellCnt;

    /// Flag whether the neighbors have been computed
    this->neighboursReady = rhs.neighboursReady;

    return *this;

}


/*
 * GPUSurfaceMT::ComputeConnectivity
 */
bool GPUSurfaceMT::ComputeConnectivity(
        float *volume_D,
        int3 volDim,
        float3 volOrg,
        float3 volDelta,
        float isovalue) {

    CheckForCudaErrorSync();

    using namespace vislib::sys;
    /* Init grid parameters for all files */

    if (!CudaSafeCall(InitVolume(
            make_uint3(volDim.x, volDim.y, volDim.z),
            volOrg,
            volDelta))) {

        Log::DefaultLog.WriteMsg(Log::LEVEL_INFO,
                "%s: could not init device constants",
                this->ClassName());
        return false;
    }

    CheckForCudaErrorSync();

    if (!CudaSafeCall(InitVolume_surface_generation(
            make_uint3(volDim.x, volDim.y, volDim.z),
            volOrg,
            volDelta))) {
        Log::DefaultLog.WriteMsg(Log::LEVEL_INFO,
                "%s: could not init device constants",
                this->ClassName());
        return false;
    }

    /* Compute neighbours */

    CheckForCudaErrorSync();

    if (!CudaSafeCall(vertexNeighbours_D.Validate(this->vertexCnt*18))) {
        Log::DefaultLog.WriteMsg(Log::LEVEL_INFO,
                "%s: could not allocate device memory",
                this->ClassName());
        return false;
    }
    CheckForCudaErrorSync();
    //if (!CudaSafeCall(vertexNeighbours_D.Set(-1))) {
    if (!CudaSafeCall(vertexNeighbours_D.Set(0xff))) {
        Log::DefaultLog.WriteMsg(Log::LEVEL_INFO,
                "%s: could not init device memory",
                this->ClassName());
        return false;
    }
    CheckForCudaErrorSync();
    if (!CudaSafeCall(ComputeVertexConnectivity(
            this->vertexNeighbours_D.Peek(),
            this->vertexStates_D.Peek(),
            this->vertexMap_D.Peek(),
            this->vertexMapInv_D.Peek(),
            this->cubeMap_D.Peek(),
            this->cubeMapInv_D.Peek(),
            this->cubeStates_D.Peek(),
            this->vertexCnt,
            volume_D,
            isovalue))) {

//        // DEBUG Print neighbour indices
//        HostArr<int> vertexNeighbours;
//        vertexNeighbours.Validate(vertexNeighbours_D.GetCount());
//        vertexNeighbours_D.CopyToHost(vertexNeighbours.Peek());
//        for (int i = 0; i < vertexNeighbours_D.GetCount()/18; ++i) {
//            printf("Neighbours vtx #%i: ", i);
//            for (int j = 0; j < 18; ++j) {
//                printf("%i ", vertexNeighbours.Peek()[i*18+j]);
//            }
//            printf("\n");
//        }
//        // END DEBUG

        Log::DefaultLog.WriteMsg(Log::LEVEL_INFO,
                "%s: could not compute neighbors",
                this->ClassName());

        return false;
    }

    CheckForCudaErrorSync();

    this->neighboursReady = true;
    return true;
}


/*
 * GPUSurfaceMT::Release
 */
void GPUSurfaceMT::Release() {
    CudaSafeCall(this->cubeStates_D.Release());
    CudaSafeCall(this->cubeOffsets_D.Release());
    CudaSafeCall(this->cubeMap_D.Release());
    CudaSafeCall(this->cubeMapInv_D.Release());
    CudaSafeCall(this->vertexStates_D.Release());
    CudaSafeCall(this->activeVertexPos_D.Release());
    CudaSafeCall(this->vertexIdxOffs_D.Release());
    CudaSafeCall(this->vertexMap_D.Release());
    CudaSafeCall(this->vertexMapInv_D.Release());
    CudaSafeCall(this->vertexNeighbours_D.Release());
    CudaSafeCall(this->verticesPerTetrahedron_D.Release());
    CudaSafeCall(this->tetrahedronVertexOffsets_D.Release());
    CudaSafeCall(this->triangleCamDistance_D.Release());
}

/**
 * Returns a 1D grid definition based on the given threadsPerBlock value.
 *
 * @param size             The minimum number of threads
 * @param threadsPerBlock  The number of threads per block
 * @return The grid dimensions
 */
extern "C" dim3 GPUSurfaceMT::Grid(const unsigned int size, const int threadsPerBlock) {
    //TODO: remove hardcoded hardware capabilities :(
    // see: http://code.google.com/p/thrust/source/browse/thrust/detail/backend/cuda/arch.inl
    //   and http://code.google.com/p/thrust/source/browse/thrust/detail/backend/cuda/detail/safe_scan.inl
    //   for refactoring.
    // Get maximum grid size of CUDA device.
    //hipDevice_t device;
    //hipDeviceGet(&device, 0);
    //CUdevprop deviceProps;
    //cuDeviceGetProperties(&deviceProps, device);
    //this->gridSize = dim3(deviceProps.maxGridSize[0],
    //  deviceProps.maxGridSize[1],
    //  deviceProps.maxGridSize[2]);
    const dim3 maxGridSize(65535, 65535, 0);
    const int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    dim3 grid(blocksPerGrid, 1, 1);

    return grid;
}

#endif // WITH_CUDA
