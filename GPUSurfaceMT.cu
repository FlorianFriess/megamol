#include "hip/hip_runtime.h"
//
// GPUSurfaceMT.cpp
//
// Copyright (C) 2013 by University of Stuttgart (VISUS).
// All rights reserved.
//
// Created on : Sep 17, 2013
// Author     : scharnkn
//

#include "stdafx.h"
#include "GPUSurfaceMT.h"

#ifdef WITH_CUDA

#include "cuda_error_check.h"
//#include "ComparativeSurfacePotentialRenderer.cuh"
#include "HostArr.h"
#include "sort_triangles.cuh"
#include "CUDAGrid.cuh"
#include "cuda_helper.h"

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>

using namespace megamol;
using namespace megamol::protein;

// The number of threads per block used in GenerateTriangles_D
#define GET_TRIANGLE_IDX_BLOCKSIZE 128

// Shut up eclipse syntax error highlighting
#ifdef __CDT_PARSER__
#define __device__
#define __global__
#define __shared__
#define __constant__
#define __host__
#endif

/**
 * @return Returns the thread index based on the current CUDA grid dimensions
 */
inline __device__ uint GetThreadIdx() {
    return __umul24(__umul24(blockIdx.y, gridDim.x) + blockIdx.x, blockDim.x) +
            threadIdx.x;
}

/**
 * 'Safe' inverse sqrt, that prevents dividing by zero
 *
 * @param x The input value
 * @return The inverse sqrt if x>0, 0.0 otherwise
 */
inline __device__ float safeRsqrtf(float x) {
    if (x > 0.0) {
        return 1.0f/sqrtf(x);
    } else {
        return 0.0f;
    }
}

/**
 * 'Safe' normalize function for float3 that uses safe rsqrt
 *
 * @param v The input vector to be normalized
 * @return The normalized vector v
 */
inline __device__ float safeInvLength(float3 v) {
    return safeRsqrtf(dot(v, v));
}

/**
 * 'Safe' normalize function for float2 that uses safe rsqrt
 *
 * @param v The input vector to be normalized
 * @return The normalized vector v
 */
inline __device__ float2 safeNormalize(float2 v) {
    float invLen = safeRsqrtf(dot(v, v));
    return v * invLen;
}

/**
 * 'Safe' normalize function for float3 that uses safe rsqrt
 *
 * @param v The input vector to be normalized
 * @return The normalized vector v
 */
inline __device__ float3 safeNormalize(float3 v) {
    float invLen = safeRsqrtf(dot(v, v));
    return v * invLen;
}

/**
 * Setup mapping from the list containing all cells to the list containing only
 * active cells.
 *
 * @param[out] cubeMap_D     The mapping from the cell list to the active cells'
 *                           list
 * @param[out] cubeMapInv_D  The mapping from the active cells' list to the
 *                           global cell list
 * @param[in]  cubeOffs_D    Index of the cells in the active cell's list
 * @param[in]  cubeStates_D  The flags of the cells
 * @param[in]  cubeCount     The number of cells to be processed
 */
// TODO cubemapInv_D is pointless, since it contains the same information as
//      cubeOffs_D
__global__ void CalcCubeMap_D(
        uint* cubeMap_D,     // output
        uint* cubeMapInv_D,  // output
        uint* cubeOffs_D,    // input
        uint* cubeStates_D,  // input
        uint cubeCount) {    // input

    const uint cubeIndex = ::GetThreadIdx();
    if (cubeIndex >= cubeCount) {
        return;
    }

    if(cubeStates_D[cubeIndex] != 0) {
        // Map from active cubes list to cube index
        cubeMap_D[cubeOffs_D[cubeIndex]] = cubeIndex;
        cubeMapInv_D[cubeIndex] = cubeOffs_D[cubeIndex];
    }
}


/*
 * CalcCubeMap
 */
extern "C"
hipError_t CalcCubeMap(
        uint *cubeMap_D,
        uint *cubeMapInv_D,
        uint *cubeOffs_D,
        uint *cubeStates_D,
        uint cubeCount) {

#ifdef USE_TIMER
    float dt_ms;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipEventRecord(event1, 0);
#endif

    CalcCubeMap_D <<< Grid(cubeCount, 256), 256 >>> (
            cubeMap_D,
            cubeMapInv_D,
            cubeOffs_D,
            cubeStates_D,
            cubeCount);

#ifdef USE_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("CUDA time for 'CalcCubeMap_D' :                        %.10f sec\n",
            dt_ms/1000.0);
#endif

    return hipGetLastError();
}

__device__ __shared__ char tetrahedronTriangles_S[16][6];
__device__ __constant__ char tetrahedronTriangles[16][6] = {
    {-1, -1, -1, -1, -1, -1}, // #0
    { 0,  3,  2, -1, -1, -1}, // #1
    { 0,  1,  4, -1, -1, -1}, // #2
    { 1,  4,  2,  2,  4,  3}, // #3
    { 1,  2,  5, -1, -1, -1}, // #4
    { 0,  3,  5,  0,  5,  1}, // #5
    { 0,  2,  5,  0,  5,  4}, // #6
    { 5,  4,  3, -1, -1, -1}, // #7
    { 3,  4,  5, -1, -1, -1}, // #8
    { 4,  5,  0,  5,  2,  0}, // #9
    { 1,  5,  0,  5,  3,  0}, // #10
    { 5,  2,  1, -1, -1, -1}, // #11
    { 3,  4,  2,  2,  4,  1}, // #12
    { 4,  1,  0, -1, -1, -1}, // #13
    { 2,  3,  0, -1, -1, -1}, // #14
    {-1, -1, -1, -1, -1, -1}  // #15
};
inline __device__ void LoadTetrahedronTrianglesToSharedMemory() {
    // Load tetrahedron triangle table into shared memory.
    if (threadIdx.x < 16) {
        for (int i = 0; i < 6; ++i) {
            tetrahedronTriangles_S[threadIdx.x][i] = tetrahedronTriangles[threadIdx.x][i];
        }
    }
}

__device__ __shared__ unsigned char tetrahedronEdgeFlags_S[16];
__device__ __constant__ unsigned char tetrahedronEdgeFlags[16] = {
    0x00, 0x0d, 0x13, 0x1e, 0x26, 0x2b, 0x35, 0x38,
    0x38, 0x35, 0x2b, 0x26, 0x1e, 0x13, 0x0d, 0x00
};
__device__ __shared__ char tetrahedronEdgeConnections_S[6][2];
__device__ __constant__ char tetrahedronEdgeConnections[6][2] = {
    {0, 1},  {1, 2},  {2, 0},  {0, 3},  {1, 3},  {2, 3}
};
inline __device__
void LoadTetrahedronEdgeFlagsAndConnectionsToSharedMemory() {
    // Load tetrahedron edge flags into shared memory.
    if (threadIdx.x < 16) {
        tetrahedronEdgeFlags_S[threadIdx.x] = tetrahedronEdgeFlags[threadIdx.x];
    }
    // Load tetrahedron edge connection table into shared memory.
    if (threadIdx.x < 6) {
        tetrahedronEdgeConnections_S[threadIdx.x][0] = tetrahedronEdgeConnections[threadIdx.x][0];
        tetrahedronEdgeConnections_S[threadIdx.x][1] = tetrahedronEdgeConnections[threadIdx.x][1];
    }
}

// [tetrahedronIdx][vtx][tetrahedronEdge]
// -1 indicates undefined values
__device__ __shared__ int VertexIdxPerTetrahedronIdx_S[6][2][2];
__device__ __constant__ int VertexIdxPerTetrahedronIdx[6][2][2] = {
        {{ 0,  2}, { 6,  3}}, // Tetrahedron #0
        {{ 4,  2}, {-1, -1}}, // Tetrahedron #1
        {{ 1,  2}, {-1, -1}}, // Tetrahedron #2
        {{ 5,  2}, {-1, -1}}, // Tetrahedron #3
        {{ 2,  2}, {-1, -1}}, // Tetrahedron #4
        {{ 3,  2}, {-1, -1}}, // Tetrahedron #5
};
inline __device__ void LoadVertexIdxPerTetrahedronIdxToSharedMemory() {
    // Load cube vertex offsets into shared memory
    if (threadIdx.x < 6) {
        VertexIdxPerTetrahedronIdx_S[threadIdx.x][0][0] = VertexIdxPerTetrahedronIdx[threadIdx.x][0][0];
        VertexIdxPerTetrahedronIdx_S[threadIdx.x][0][1] = VertexIdxPerTetrahedronIdx[threadIdx.x][0][1];
        VertexIdxPerTetrahedronIdx_S[threadIdx.x][1][0] = VertexIdxPerTetrahedronIdx[threadIdx.x][1][0];
        VertexIdxPerTetrahedronIdx_S[threadIdx.x][1][1] = VertexIdxPerTetrahedronIdx[threadIdx.x][1][1];
    }
}


__device__ __shared__ uint tetrahedronsInACube_S[6][4];
__device__ __constant__ uint tetrahedronsInACube[6][4] = {
    {0, 5, 1, 6},
    {0, 1, 2, 6},
    {0, 2, 3, 6},
    {0, 3, 7, 6},
    {0, 7, 4, 6},
    {0, 4, 5, 6}
};
inline __device__ void LoadTetrahedronsInACubeToSharedMemory() {
    // Load cube vertex offsets into shared memory
    if (threadIdx.x < 6) {
        tetrahedronsInACube_S[threadIdx.x][0] = tetrahedronsInACube[threadIdx.x][0];
        tetrahedronsInACube_S[threadIdx.x][1] = tetrahedronsInACube[threadIdx.x][1];
        tetrahedronsInACube_S[threadIdx.x][2] = tetrahedronsInACube[threadIdx.x][2];
        tetrahedronsInACube_S[threadIdx.x][3] = tetrahedronsInACube[threadIdx.x][3];
    }
}


__device__ __shared__ uint cubeVertexOffsets_S[8][3];
__device__ __constant__ uint cubeVertexOffsets[8][3] = {
    {0, 0, 0},
    {1, 0, 0},
    {1, 1, 0},
    {0, 1, 0},
    {0, 0, 1},
    {1, 0, 1},
    {1, 1, 1},
    {0, 1, 1}
};
inline __device__ void LoadCubeOffsetsToSharedMemory() {
    // Load cube vertex offsets into shared memory
//    if (threadIdx.x < 32) {
//        const uint idx0 = clamp(int(threadIdx.x/8), 0, 7);
//        const uint idx1 = threadIdx.x%3;
//        cubeVertexOffsets_S[idx0][idx1] = cubeVertexOffsets[idx0][idx1];
//    }
    if (threadIdx.x < 8) {
        cubeVertexOffsets_S[threadIdx.x][0] = cubeVertexOffsets[threadIdx.x][0];
        cubeVertexOffsets_S[threadIdx.x][1] = cubeVertexOffsets[threadIdx.x][1];
        cubeVertexOffsets_S[threadIdx.x][2] = cubeVertexOffsets[threadIdx.x][2];
    }
}

/**
 * Computes flags for one tetrahedron. The flags define what vertices of the
 * tetrahedron are active.
 *
 * @param[in] cubeVertex0       The origin of the current cell
 * @param[in] tetrahedronIndex  Local index of the tetrahedron inside the cell
 * @param[in] thresholdValue    The isovalue that defines the isosurface
 * @param[in] volume_D          The volume the isosurface is extracted from
 *
 * @return The tetrahedron flags
 */
inline __device__ unsigned char TetrahedronFlags_D(
        uint3 cubeVertex0,
        int tetrahedronIndex,
        float thresholdValue,
        float *volume_D) {

    unsigned char flags = 0;
    // Loop through all four vertices of the tetrahedron
    for (int idx = 0; idx < 4; ++idx) {
        const uint3 cubeVertexOffset = make_uint3(
                cubeVertexOffsets_S[tetrahedronsInACube_S[tetrahedronIndex][idx]][0],
                cubeVertexOffsets_S[tetrahedronsInACube_S[tetrahedronIndex][idx]][1],
                cubeVertexOffsets_S[tetrahedronsInACube_S[tetrahedronIndex][idx]][2]);
        if(::SampleFieldAt_D<float>(cubeVertex0 + cubeVertexOffset, volume_D) <= thresholdValue) {
            flags |= 1 << static_cast<unsigned char>(idx);
        }
    }
    return flags;
}

/**
 * Compute vertex positions in active cells. Every active cell is associated
 * with 7 potentially active vertices. For all 7 vertices their active flag is
 * set and, if possible, the position is computed. One kernel processes one
 * tetrahedron
 *
 * @param[out] activeVertexIdx_D The flag that shows whether a vertex is active
 *                               ('1') or not ('0')
 * @param[out] activeVertexPos_D The position of active vertices
 * @param[in]  cubeMapInv_D      Mapping from the global cell list to the active
 *                               cells' list
 * @param[in]  isoval            The isovalue that defines the isosurface
 * @param[in]  activeCubeCnt     The number of active cells
 * @param[in]  volume_D          The volume the isosurface is extracted from
 */
__global__ void CalcVertexPositions_D(
        uint*activeVertexIdx_D,
        float3 *activeVertexPos_D,
        uint* cubeMap_D,
        float isoval,
        uint activeCubeCount,
        float *volume_D) {

    // Load LUTs to shared memory
    LoadCubeOffsetsToSharedMemory();
    LoadTetrahedronsInACubeToSharedMemory();
    LoadVertexIdxPerTetrahedronIdxToSharedMemory();
    LoadTetrahedronEdgeFlagsAndConnectionsToSharedMemory();
    __syncthreads();

    // Thread index (= active cube index)
    uint globalTetraIdx = ::GetThreadIdx();
    if (globalTetraIdx >= activeCubeCount*6) {
        return;
    }

    uint activeCubeIdx = globalTetraIdx/6;
    uint localTetraIdx = globalTetraIdx%6; // 0 ... 5

    // Compute cell origin
    const uint3 cellOrg = GetGridCoordsByCellIdx(cubeMap_D[activeCubeIdx]);

    // Get bitmap to classify the tetrahedron
    unsigned char tetrahedronFlags = TetrahedronFlags_D(cellOrg, localTetraIdx, isoval, volume_D);

    for (int i = 0; i < 2; ++i) {
        if (VertexIdxPerTetrahedronIdx_S[localTetraIdx][i][0] < 0) {
            continue;
        }
        uint localVtxIdx = VertexIdxPerTetrahedronIdx_S[localTetraIdx][i][0];
        uint edgeIdx = VertexIdxPerTetrahedronIdx_S[localTetraIdx][i][1];
        if (tetrahedronEdgeFlags_S[tetrahedronFlags] & (1 << static_cast<unsigned char>(edgeIdx))) {

            // Interpolate vertex position
            const uint3 v0 = cellOrg + make_uint3(
                    cubeVertexOffsets_S[tetrahedronsInACube_S[localTetraIdx][tetrahedronEdgeConnections_S[edgeIdx][0]]][0],
                    cubeVertexOffsets_S[tetrahedronsInACube_S[localTetraIdx][tetrahedronEdgeConnections_S[edgeIdx][0]]][1],
                    cubeVertexOffsets_S[tetrahedronsInACube_S[localTetraIdx][tetrahedronEdgeConnections_S[edgeIdx][0]]][2]);
            const uint3 v1 = cellOrg + make_uint3(
                    cubeVertexOffsets_S[tetrahedronsInACube_S[localTetraIdx][tetrahedronEdgeConnections_S[edgeIdx][1]]][0],
                    cubeVertexOffsets_S[tetrahedronsInACube_S[localTetraIdx][tetrahedronEdgeConnections_S[edgeIdx][1]]][1],
                    cubeVertexOffsets_S[tetrahedronsInACube_S[localTetraIdx][tetrahedronEdgeConnections_S[edgeIdx][1]]][2]);

            // Linear interpolation
            const float f0 = ::SampleFieldAt_D<float>(v0, volume_D);
            const float f1 = ::SampleFieldAt_D<float>(v1, volume_D);
            const float interpolator = (isoval - f0) / (f1 - f0);
            float3 vertex = lerp(make_float3(v0.x, v0.y, v0.z),
                    make_float3(v1.x, v1.y, v1.z), interpolator);

            // Save position and mark vertex index as 'active'
            activeVertexIdx_D[activeCubeIdx*7+localVtxIdx] = 1;
            activeVertexPos_D[activeCubeIdx*7+localVtxIdx] = TransformToWorldSpace(vertex);
        }
    }
}


extern "C"
hipError_t CalcVertexPositions(uint *vertexStates_D, float3 *activeVertexPos_D,
        uint *vertexIdxOffs_D, uint *cubeMap_D, uint activeCubeCount, float isoval,
        float *volume_D) {

    const uint threadPerBlock = 128;

#ifdef USE_TIMER
    float dt_ms;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipEventRecord(event1, 0);
#endif

    CalcVertexPositions_D <<< Grid(activeCubeCount*6, threadPerBlock ), threadPerBlock  >>> (
            vertexStates_D,
            activeVertexPos_D,
            cubeMap_D,
            isoval,
            activeCubeCount,
            volume_D
    );

#ifdef USE_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("CUDA time for 'CalcVertexPositions_D' :                %.10f sec\n",
            dt_ms/1000.0);
#endif

//    ::ComputePrefixSumExclusiveScan(
//            vertexStates_D,
//            vertexIdxOffs_D,
//            7*activeCubeCount-1);

    thrust::exclusive_scan(
            thrust::device_ptr<uint>(vertexStates_D),
            thrust::device_ptr<uint>(vertexStates_D + 7*activeCubeCount),
            thrust::device_ptr<uint>(vertexIdxOffs_D));

    return hipGetLastError();
}


/**
 * Setup mapping function from active vertex list to vertex list (based on
 * active cells).
 *
 * @param[out] vertexMap_D       Mapping from active vertex' list to global
 *                               vertex list
 * @param[out] vertexMapInv_D    Mapping from global vertex list to active
 *                               vertex' list
 * @param[in]  vertexIdxOffs_D   Offsets for vertex indices
 * @param[in]  activeVertexIdx_D Active vertex flags, '1' if vertex is active
 * @param[in]  vtxCount          The number of vertices
 */
// TODO vertexIdxOffs_D is pointless
__global__ void CalcVertexMap_D(
        uint* vertexMap_D,
        uint* vertexMapInv_D,
        uint* vertexIdxOffs_D,
        uint* activeVertexIdx_D,
        uint vtxCount) {

    const uint vtxIndex = ::GetThreadIdx();
    if (vtxIndex >= vtxCount) {
        return;
    }

    if(activeVertexIdx_D[vtxIndex] != 0) {
        // Map from active vertices list to vtx idx
        vertexMap_D[vertexIdxOffs_D[vtxIndex]] = vtxIndex;
        vertexMapInv_D[vtxIndex] = vertexIdxOffs_D[vtxIndex];
    }
}

extern "C"
hipError_t CalcVertexMap(uint *vertexMap_D, uint *vertexMapInv_D,
        uint *vertexIdxOffs_D, uint *vertexStates_D, uint activeCellsCount) {

    CalcVertexMap_D <<< Grid(7*activeCellsCount, 256), 256 >>> (
            vertexMap_D, vertexMapInv_D, vertexIdxOffs_D, vertexStates_D,
            7*activeCellsCount);

    return hipGetLastError();
}



__device__ __shared__ uint tetrahedronVertexCount_S[16];
__device__ __constant__ uint tetrahedronVertexCount[16] = {
    0, 3, 3, 6, 3, 6, 6, 3,
    3, 6, 6, 3, 6, 3, 3, 0
};
inline __device__ void LoadTetrahedronVertexCountToSharedMemory() {
    // Load tetrahedron vertex count into shared memory.
    if (threadIdx.x < 16) {
        tetrahedronVertexCount_S[threadIdx.x] = tetrahedronVertexCount[threadIdx.x];
    }
}


// Returns unsigned char from 00000 to 111111 describing the activity of the
// vertices inside the cube of index cubeId.
__device__
unsigned char CubeFlags(uint cubeId, uint *activeVertexIdx_D) {
    unsigned char flags = 0x00;
    for(int idx = 0; idx < 6; ++idx) {
        if(activeVertexIdx_D[6*cubeId] == 1) {
            flags |= 1 << static_cast<unsigned char>(idx);
        }
    }
    return flags;
}

// Maps the index based on the tetrahedron index and the edge
// index to a global vertex index based on cubeIdx*7 + localIdx. The layout is
// {cubeIdxOffs.x, cubeIdxOffs.y, cubeIdxOffs.z, localIdx}
__shared__ __device__ int TetrahedronEdgeVertexIdxOffset_S[6][6][4];
__constant__ __device__ int TetrahedronEdgeVertexIdxOffset[6][6][4] = {
    {{0, 0, 0, 3}, {1, 0, 0, 2}, {0, 0, 0, 0}, {0, 0, 0, 6}, {1, 0, 1, 1}, {1, 0, 0, 5}}, // Tetrahedron #0
    {{0, 0, 0, 0}, {1, 0, 0, 1}, {0, 0, 0, 4}, {0, 0, 0, 6}, {1, 0, 0, 5}, {1, 1, 0, 2}}, // Tetrahedron #1
    {{0, 0, 0, 4}, {0, 1, 0, 0}, {0, 0, 0, 1}, {0, 0, 0, 6}, {1, 1, 0, 2}, {0, 1, 0, 3}}, // Tetrahedron #2
    {{0, 0, 0, 1}, {0, 1, 0, 2}, {0, 0, 0, 5}, {0, 0, 0, 6}, {0, 1, 0, 3}, {0, 1, 1, 0}}, // Tetrahedron #3
    {{0, 0, 0, 5}, {0, 0, 1, 1}, {0, 0, 0, 2}, {0, 0, 0, 6}, {0, 1, 1, 0}, {0, 0, 1, 4}}, // Tetrahedron #4
    {{0, 0, 0, 2}, {0, 0, 1, 0}, {0, 0, 0, 3}, {0, 0, 0, 6}, {0, 0, 1, 4}, {1, 0, 1, 1}}  // Tetrahedron #5
};
__device__ void LoadTetrahedronEdgeVertexIdxOffsetToSharedMemory() {
//    if (threadIdx.x < 6) {
//        for (int i = 0; i < 6; ++i) {
//            TetrahedronEdgeVertexIdxOffset_S[threadIdx.x][i][0] = TetrahedronEdgeVertexIdxOffset[threadIdx.x][i][0];
//            TetrahedronEdgeVertexIdxOffset_S[threadIdx.x][i][1] = TetrahedronEdgeVertexIdxOffset[threadIdx.x][i][1];
//            TetrahedronEdgeVertexIdxOffset_S[threadIdx.x][i][2] = TetrahedronEdgeVertexIdxOffset[threadIdx.x][i][2];
//            TetrahedronEdgeVertexIdxOffset_S[threadIdx.x][i][3] = TetrahedronEdgeVertexIdxOffset[threadIdx.x][i][3];
//        }
//    }
    if (threadIdx.x < 64) {
        const uint idx0 = clamp(int(threadIdx.x/6), 0, 5);
        const uint idx1 = threadIdx.x%6;
        TetrahedronEdgeVertexIdxOffset_S[idx0][idx1][0] = TetrahedronEdgeVertexIdxOffset[idx0][idx1][0];
        TetrahedronEdgeVertexIdxOffset_S[idx0][idx1][1] = TetrahedronEdgeVertexIdxOffset[idx0][idx1][1];
        TetrahedronEdgeVertexIdxOffset_S[idx0][idx1][2] = TetrahedronEdgeVertexIdxOffset[idx0][idx1][2];
        TetrahedronEdgeVertexIdxOffset_S[idx0][idx1][3] = TetrahedronEdgeVertexIdxOffset[idx0][idx1][3];
    }
}

// Describes connections inside the tetrahedron for every tetrahedron edge
// based on the tetrahedron edge flags (0-16)
__constant__ __device__ unsigned char TetrahedronEdgeConnections[16][6] = {
// edges   #0      #1      #2      #3      #4      #5
        {0x00, 0x00, 0x00, 0x00, 0x00, 0x00}, // vertices active 0000 #00 (= #15)
        {0x0C, 0x00, 0x09, 0x05, 0x00, 0x00}, // vertices active 0001 #01 (= #14)
        {0x12, 0x11, 0x00, 0x00, 0x03, 0x00}, // vertices active 0010 #02 (= #13)
        {0x00, 0x14, 0x1A, 0x14, 0x0E, 0x00}, // vertices active 0011 #03 (= #12)
        {0x00, 0x24, 0x22, 0x00, 0x00, 0x06}, // vertices active 0100 #04 (= #11)
        {0x2A, 0x21, 0x00, 0x21, 0x00, 0x0B}, // vertices active 0101 #05 (= #10)
        {0x34, 0x00, 0x21, 0x00, 0x21, 0x15}, // vertices active 0110 #06 (= #09)
        {0x00, 0x00, 0x00, 0x30, 0x28, 0x18}, // vertices active 0111 #07 (= #08)
        {0x00, 0x00, 0x00, 0x30, 0x28, 0x18}, // vertices active 1000 #08 (= #07)
        {0x34, 0x00, 0x21, 0x00, 0x21, 0x15}, // vertices active 1001 #09 (= #06)
        {0x2A, 0x21, 0x00, 0x21, 0x00, 0x0B}, // vertices active 1010 #10 (= #05)
        {0x00, 0x24, 0x22, 0x00, 0x00, 0x06}, // vertices active 1011 #11 (= #04)
        {0x00, 0x14, 0x1A, 0x14, 0x0E, 0x00}, // vertices active 1100 #12 (= #03)
        {0x12, 0x11, 0x00, 0x00, 0x03, 0x00}, // vertices active 1101 #13 (= #02)
        {0x0C, 0x00, 0x09, 0x05, 0x00, 0x00}, // vertices active 1110 #14 (= #01)
        {0x00, 0x00, 0x00, 0x00, 0x00, 0x00}  // vertices active 1111 #15 (= #00)
};

// Contains all neighbouring tetrahedrons of a vertex (v0-v6), defined by
// a global cube offset and a local tetrahedron index. Values beyond [-1, 1]
// indicate undefined values
__shared__ __device__ int VertexNeighbouringTetrahedrons_S[7][6][4];
__constant__ __device__ int VertexNeighbouringTetrahedrons[7][6][4] = {
        {{ 0,  0,  0,  0}, { 0,  0,  0,  1}, { 0, -1,  0,  2}, { 0, -1, -1,  3}, { 0, -1, -1,  4}, { 0,  0, -1,  5}}, // v0
        {{-1,  0, -1,  0}, {-1,  0,  0,  1}, { 0,  0,  0,  2}, { 0,  0,  0,  3}, { 0,  0, -1,  4}, {-1,  0, -1,  5}}, // v1
        {{-1,  0,  0,  0}, {-1, -1,  0,  1}, {-1, -1,  0,  2}, { 0, -1,  0,  3}, { 0,  0,  0,  4}, { 0,  0,  0,  5}}, // v2
        {{ 0,  0,  0,  0}, {99, 99, 99,  1}, { 0, -1,  0,  2}, { 0, -1,  0,  3}, {99, 99, 99,  4}, { 0,  0,  0,  5}}, // v3
        {{99, 99, 99,  0}, { 0,  0,  0,  1}, { 0,  0,  0,  2}, {99, 99, 99,  3}, { 0,  0, -1,  4}, { 0,  0, -1,  5}}, // v4
        {{-1,  0,  0,  0}, {-1,  0,  0,  1}, {99, 99, 99,  2}, { 0,  0,  0,  3}, { 0,  0,  0,  4}, {99, 99, 99,  5}}, // v5
        {{ 0,  0,  0,  0}, { 0,  0,  0,  1}, { 0,  0,  0,  2}, { 0,  0,  0,  3}, { 0,  0,  0,  4}, { 0,  0,  0,  5}}  // v6
};
__device__ void LoadVertexNeighbouringTetrahedronsToSharedMemory() {
//    if (threadIdx.x < 7) {
//        for (int i = 0; i < 6; ++i) {
//            VertexNeighbouringTetrahedrons_S[threadIdx.x][i][0] = VertexNeighbouringTetrahedrons[threadIdx.x][i][0];
//            VertexNeighbouringTetrahedrons_S[threadIdx.x][i][1] = VertexNeighbouringTetrahedrons[threadIdx.x][i][1];
//            VertexNeighbouringTetrahedrons_S[threadIdx.x][i][2] = VertexNeighbouringTetrahedrons[threadIdx.x][i][2];
//            VertexNeighbouringTetrahedrons_S[threadIdx.x][i][3] = VertexNeighbouringTetrahedrons[threadIdx.x][i][3];
//        }
//    }
    if (threadIdx.x < 64) {
        const uint idx0 = clamp(int(threadIdx.x/7), 0, 6);
        const uint idx1 = threadIdx.x%6;
        VertexNeighbouringTetrahedrons_S[idx0][idx1][0] = VertexNeighbouringTetrahedrons[idx0][idx1][0];
        VertexNeighbouringTetrahedrons_S[idx0][idx1][1] = VertexNeighbouringTetrahedrons[idx0][idx1][1];
        VertexNeighbouringTetrahedrons_S[idx0][idx1][2] = VertexNeighbouringTetrahedrons[idx0][idx1][2];
        VertexNeighbouringTetrahedrons_S[idx0][idx1][3] = VertexNeighbouringTetrahedrons[idx0][idx1][3];
    }
}

// Contains the edge index every vertex (v0-v6) has inside its adjacent
// tetrahedrons, -1 indicates undefined values
__constant__ __device__ int VertexNeighbouringTetrahedronsOwnEdgeIdx[7][6] = {
        { 2,  0,  1,  5,  4,  1}, // v0
        { 4,  1,  2,  0,  1,  5}, // v1
        { 1,  5,  4,  1,  2,  0}, // v2
        { 0, -1,  5,  4, -1,  2}, // v3
        {-1,  2,  0, -1,  5,  4}, // v4
        { 5,  4, -1,  2,  0, -1}, // v5
        { 3,  3,  3,  3,  3,  3}  // v6
};

// Defines the neighbour index for all possible connected edges for all vertices
// -1 indicates, that there is no connection possible (self)
__constant__ __device__ int TetrahedronToNeighbourIdx[7][6][6] = {

        {{ 0,  1, -1,  2,  3,  4}, {-1,  5,  6,  2,  4,  7}, { 8, -1,  9, 10,  1,  0},
         {11, 12, 13, 14, 15, -1}, {13,  9, 16, 14, -1,  8}, {12, -1, 15, 17,  6,  5}}, // #v0

        {{ 0,  1,  2,  3, -1,  4}, { 5, -1,  6,  7,  8,  9}, {10, 11, -1, 12, 13, 14},
         {-1,  9,  8, 12, 14, 15}, { 4, -1,  1, 16, 11, 10}, {17,  5,  0,  3,  6, -1}}, // #v1

        {{ 0, -1,  1,  2,  3,  4}, { 5,  6,  7,  8,  9, -1}, { 7,  1, 10,  8, -1,  0},
         { 6, -1,  9, 11, 12, 13}, { 4,  3, -1, 14, 15, 16}, {-1, 13, 12, 14, 16, 17}}, // #v2

        {{-1,  0,  1,  2,  3,  4}, {-1, -1, -1, -1, -1, -1}, { 5,  1,  6,  7,  0, -1},
         { 6,  8,  9,  7, -1, 10}, {-1, -1, -1, -1, -1, -1}, { 8, 10, -1,  2, 11,  3}}, // #v3

        {{-1, -1, -1, -1, -1, -1}, { 0,  1, -1,  2,  3,  4}, {-1,  5,  6,  2,  4,  7},
         {-1, -1, -1, -1, -1, -1}, { 8,  6,  9,  10, 5, -1}, { 9,  0, 11, 10, -1,  1}}, // #v4

        {{ 0,  1,  2,  3,  4, -1}, { 2,  5,  6,  3, -1,  7}, {-1, -1, -1, -1, -1, -1},
         { 5,  7, -1,  8,  9, 10}, {-1,  4,  1,  8, 10, 11}, {-1, -1, -1, -1, -1, -1}}, // #v5

        {{ 0,  1,  2, -1,  3,  4}, { 2,  5,  6, -1,  4,  7}, { 6,  8,  9, -1,  7, 10},
         { 9, 11, 12, -1, 10, 13}, {12, 14, 15, -1, 13, 16}, {15, 17,  0, -1, 16,  3}}, // #v6
};


inline __device__
void LoadTetrahedronsInACube() {
    // Load tetrahedron vertex index to cube index map into shared memory.
    if (threadIdx.x < 6) {
        for (int i = 0; i < 4; ++i) {
            tetrahedronsInACube_S[threadIdx.x][i] = tetrahedronsInACube[threadIdx.x][i];
        }
    }
}

/*
 * freudenthal_subdiv::GetTetrahedronEdgeVertexIdxOffset
 */
inline __device__
uint GetTetrahedronEdgeVertexIdx(uint activeCubeIndex, uint tetrahedronIdx, uint edgeIdx, uint *cubeMap, uint *cubeMapInv) {
    uint cubeIdx = cubeMap[activeCubeIndex];
    uint offset = (gridSize_D.x-1)*(
            (gridSize_D.y-1)*TetrahedronEdgeVertexIdxOffset[tetrahedronIdx][edgeIdx][2] // Global cube index
            + TetrahedronEdgeVertexIdxOffset[tetrahedronIdx][edgeIdx][1])           // Global cube index
            + TetrahedronEdgeVertexIdxOffset[tetrahedronIdx][edgeIdx][0];
    uint cubeIdxNew = cubeMapInv[cubeIdx + offset];
    return 7*cubeIdxNew + TetrahedronEdgeVertexIdxOffset[tetrahedronIdx][edgeIdx][3];
}


/**
 * Writes all active vertex positions to a compacted array.
 * @param[out] vertexPos_D       The array with the compacted positions
 * @param[in]  vertexStates_D    Contains flags that show the activity of the
 *                               vertices
 * @param[in]  vertexIdxOffs_D   The index of the vertex in the compacted vertex
 *                               list
 * @param[in]  activeVertexPos_D The array with non-compacted vertex positions
 * @param[in]  vertexCount       The number of vertices (active and non-active)
 * @param[in]  outputArrOffs     The output buffer offset to store vertex
 *                               positions
 * @param[in]  outputArrDataSize The output buffer stride
 */
__global__
void CompactActiveVertexPositions_D(
        float *vertexPos_D,
        uint *vertexStates_D,
        uint *vertexIdxOffs_D,
        float3 *activeVertexPos_D,
        uint vertexCount,
        uint outputArrOffs,
        uint outputArrDataSize) {

    // Thread index (= vertex index)
    uint idx = GetThreadIdx();
    if (idx >= vertexCount) {
        return;
    }

    if (vertexStates_D[idx] == 1) {
        vertexPos_D[outputArrDataSize*vertexIdxOffs_D[idx]+outputArrOffs+0] = activeVertexPos_D[idx].x;
        vertexPos_D[outputArrDataSize*vertexIdxOffs_D[idx]+outputArrOffs+1] = activeVertexPos_D[idx].y;
        vertexPos_D[outputArrDataSize*vertexIdxOffs_D[idx]+outputArrOffs+2] = activeVertexPos_D[idx].z;
    }

}

extern "C"
hipError_t CompactActiveVertexPositions(float *vertexPos_D, uint *vertexStates_D,
        uint *vertexIdxOffs_D, float3 *activeVertexPos_D, uint activeCellCount,
        uint outputArrOffs, uint outputArrDataSize) {

#ifdef USE_TIMER
    float dt_ms;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipEventRecord(event1, 0);
#endif

    CompactActiveVertexPositions_D <<< Grid(activeCellCount*7, 256), 256 >>> (
            vertexPos_D, vertexStates_D, vertexIdxOffs_D, activeVertexPos_D,
            activeCellCount*7, outputArrOffs, outputArrDataSize);

#ifdef USE_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("CUDA time for 'CompactActiveVertexPositions_D' :       %.10f sec\n",
            dt_ms/1000.0);
#endif

    return hipGetLastError();
}


/**
 * Determines for every tetrahedron the number of active vertices. This is
 * necessary to get the actual triangle count.
 *
 * @param[out] verticesPerTetrahedron_D The number of vertices in every
 *                                      tetrahedron (can be either 0, 3 or 6)
 * @param[in]  cubeMap_D                mapping from active cells' list to
 *                                      global cell list
 * @param[in]  thresholdValue           The isovalue
 * @param[in]  activeCubeCount          The number of active cells
 * @param[in]  volume_D                 The volume the isosurface us extracted
 *                                      from
 */
__global__ void FlagTetrahedrons_D(
        uint* verticesPerTetrahedron_D,
        uint* cubeMap_D,
        float thresholdValue,
        uint activeCubeCount,
        float *volume_D) {

    const uint activeCubeIndex = GetThreadIdx();

    LoadCubeOffsetsToSharedMemory();
    LoadTetrahedronsInACubeToSharedMemory();
    __syncthreads();

    // Prevent non-power of two writes.
    if (activeCubeIndex >= activeCubeCount) {
        return;
    }
    const uint3 cubeVertex0 = GetGridCoordsByCellIdx(cubeMap_D[activeCubeIndex]);
    // Classify all tetrahedrons in a cube.
    for (int tetrahedronIndex = 0; tetrahedronIndex < 6; ++tetrahedronIndex) {
        // Compute tetrahedron flags.
        unsigned char tetrahedronFlags = TetrahedronFlags_D(cubeVertex0, tetrahedronIndex, thresholdValue, volume_D);
        // Store number of vertices.
        verticesPerTetrahedron_D[activeCubeIndex * 6 + tetrahedronIndex] = tetrahedronVertexCount[tetrahedronFlags];
    }
}

extern "C"
hipError_t FlagTetrahedrons(
        uint *verticesPerTetrahedron_D,
        uint *cubeMap_D,
        float isoval,
        uint activeCellCount,
        float *volume_D) {

#ifdef USE_TIMER
    float dt_ms;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipEventRecord(event1, 0);
#endif

    FlagTetrahedrons_D <<< Grid(activeCellCount, 256), 256 >>> (
            verticesPerTetrahedron_D,
            cubeMap_D,
            isoval,
            activeCellCount,
            volume_D);

#ifdef USE_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("CUDA time for 'FlagTetrahedrons_D' :       %.10f sec\n",
            dt_ms/1000.0);
#endif

    return hipGetLastError();
}


extern "C"
hipError_t GetTetrahedronVertexOffsets(
        uint *tetrahedronVertexOffsets_D,
        uint *verticesPerTetrahedron_D,
        uint tetrahedronCount) {


    thrust::exclusive_scan(
            thrust::device_ptr<uint>(verticesPerTetrahedron_D),
            thrust::device_ptr<uint>(verticesPerTetrahedron_D + tetrahedronCount),
            thrust::device_ptr<uint>(tetrahedronVertexOffsets_D));

//    ::ComputePrefixSumExclusiveScan(
//            verticesPerTetrahedron_D,
//            tetrahedronVertexOffsets_D,
//            tetrahedronCount); // TODO This is unintuitive

    return hipGetLastError();
}


/**
 * Obtains the vertex indices for all triangles.
 *
 * @param[in]  vertexOffsets_D Offsets for vertex indices
 * @param[in]  cubeMap_D       Mapping from active cells's list to global cell list
 * @param[in]  cubeMapInv_D    Inverse mapping to cubeMap_D
 * @param[in]  thresholdValue  The isovalue
 * @param[in]  tetrahedronCount The number of tetrahedrons
 * @param[in]  activeCubeCount The number of active cells
 * @param[out] triangleVertexIdx_D The triangles' vertex indices
 * @param[in]  vertexMapInv_D Inverse mapping to vertexOffsets_D
 * @param[in]  volume_D The volume the isosurface is extracted from
 */
__global__
void GetTrianglesIdx_D(
        uint* vertexOffsets_D,
        uint* cubeMap_D,
        uint* cubeMapInv_D,
        float thresholdValue,
        uint tetrahedronCount,
        uint activeCubeCount,
        uint *triangleVertexIdx_D,
        uint *vertexMapInv_D,
        float *volume_D) {

    const uint id = GetThreadIdx();
    const uint activeCubeIndex = id / 6;
    const int tetrahedronIndex = id % 6;


    // Load tables from constant to shared memory
    LoadCubeOffsetsToSharedMemory();
    LoadTetrahedronsInACubeToSharedMemory();
    __syncthreads();

    // Prevent non-power of two writes.
    if (id >= tetrahedronCount) {
        return;
    }

    const uint3 cubeVertex0 = GetGridCoordsByCellIdx(cubeMap_D[activeCubeIndex]);

    // Get bitmap to classify the tetrahedron
    unsigned char tetrahedronFlags = TetrahedronFlags_D(cubeVertex0,
            tetrahedronIndex, thresholdValue, volume_D);

    // Skip inactive tetrahedrons
    if (tetrahedronFlags == 0x00 || tetrahedronFlags == 0x0F) {
        return;
    }
    __shared__ uint edgeVertexIdx[6 * GET_TRIANGLE_IDX_BLOCKSIZE];

    // Find intersection of the surface with each edge.
    for (int edgeIndex = 0; edgeIndex < 6; edgeIndex++) {
        // Test if edge intersects with surface.
        if (tetrahedronEdgeFlags[tetrahedronFlags] & (1 << static_cast<unsigned char>(edgeIndex)))  {
            edgeVertexIdx[threadIdx.x * 6 + edgeIndex] =
                    GetTetrahedronEdgeVertexIdx(activeCubeIndex,
                            tetrahedronIndex, edgeIndex, cubeMap_D, cubeMapInv_D);
        }
    }

    __syncthreads();

    // Write vertices.
    for (int triangleIndex = 0; triangleIndex < 2; triangleIndex++) {
        if (tetrahedronTriangles[tetrahedronFlags][3 * triangleIndex] >= 0) {
            for (int cornerIndex = 0; cornerIndex < 3; cornerIndex++) {
                int edgeIndex = threadIdx.x * 6 + tetrahedronTriangles[tetrahedronFlags][3 * triangleIndex + cornerIndex];
                uint vertexOffset = vertexOffsets_D[id] + 3 * triangleIndex + cornerIndex;
                triangleVertexIdx_D[vertexOffset] = vertexMapInv_D[edgeVertexIdx[edgeIndex]];
                //triangleVertexIdx_D[vertexOffset] = edgeVertexIdx[edgeIndex];
            }
        }
    }

    __syncthreads();
}


extern "C"
hipError_t GetTrianglesIdx(uint *tetrahedronVertexOffsets_D, uint *cubeMap_D,
        uint *cubeMapInv_D,
        float isoval,
        uint tetrahedronCount,
        uint activeCellCount,
        uint *triangleVertexIdx_D,
        uint *vertexMapInv_D,
        float *volume_D) {

#ifdef USE_TIMER
    float dt_ms;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipEventRecord(event1, 0);
#endif

    // TODO Use dnymic shared memory here
    GetTrianglesIdx_D <<< Grid(tetrahedronCount, GET_TRIANGLE_IDX_BLOCKSIZE), GET_TRIANGLE_IDX_BLOCKSIZE >>> (
            tetrahedronVertexOffsets_D,
            cubeMap_D,
            cubeMapInv_D,
            isoval,
            tetrahedronCount,
            activeCellCount,
            triangleVertexIdx_D,
            vertexMapInv_D,
            volume_D);

#ifdef USE_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("CUDA time for 'GetTrianglesIdx_D' :                    %.10f sec\n",
            dt_ms/1000.0);
#endif

    return hipGetLastError();
}


/**
 * Identifies neighbouring vertices for all vertices and stores them as vertex
 * indices. -1 indicates invalid neighbours
 *
 * @param[out] vertexNeighbours_D The vertex connectivity information
 * @param[in]  activeVertexIdx_D  Array with vertex activity flags
 * @param[in]  activeVertexCnt    The number of vertices
 * @param[in]  vertexMap_D        Vertex mapping from active to global index
 * @param[in]  vertexMapInv_D     Inverse mapping to vertexMap_D
 * @param[in]  cubeMap_D          Mapping from active cells's list to global
 *                                cell list
 * @param[in]  cubeMapInv_D       Inverse mapping to cubeMap_D
 * @param[in]  cubeStates_D       Flags active cells
 * @param[in]  volume_D           The volume the isosurface is extracted from
 * @param[in]  isoval             The isovalue that defines the isosurface
 */
__global__
void ComputeVertexConnectivity_D(
        int *vertexNeighbours_D,
        uint *activeVertexIdx_D,
        uint activeVertexCnt,
        uint *vertexMap_D,
        uint *vertexMapInv_D,
        uint *cubeMap_D,
        uint *cubeMapInv_D,
        uint *cubeStates_D,
        float *volume_D,
        float isoval) {

    // Get different indices
    uint idx = ::GetThreadIdx();
    uint activeVertexIdx = idx/6;
    uint i = idx - __umul24(activeVertexIdx, 6); // == idx%6;

    /* 1. Load LUTs to shared memory */

    // Note: We have 6 warps per block (each warp with 32 threads)

    // Load cube vertex offsets into shared memory
    // Use warp #0
//    LoadCubeOffsetsToSharedMemory();
    if (threadIdx.x < 32) {
        const uint idx0 = clamp(int(threadIdx.x), 0, 7);
        cubeVertexOffsets_S[idx0][0] = cubeVertexOffsets[idx0][0];
        cubeVertexOffsets_S[idx0][1] = cubeVertexOffsets[idx0][1];
        cubeVertexOffsets_S[idx0][2] = cubeVertexOffsets[idx0][2];
    }

    // Load cube vertex offsets into shared memory
    // Use warp #1
    //LoadTetrahedronsInACubeToSharedMemory();
    if (threadIdx.x >= 32 && threadIdx.x < 64) {
        const uint idx0 = clamp(int(threadIdx.x-32), 0, 7);
        tetrahedronsInACube_S[idx0][0] = tetrahedronsInACube[idx0][0];
        tetrahedronsInACube_S[idx0][1] = tetrahedronsInACube[idx0][1];
        tetrahedronsInACube_S[idx0][2] = tetrahedronsInACube[idx0][2];
        tetrahedronsInACube_S[idx0][3] = tetrahedronsInACube[idx0][3];
    }

    // Load vertex indices of neighbouring tetrahedrons to shared memory
    // Use warps #2 and #3
    //LoadVertexNeighbouringTetrahedronsToSharedMemory();
    if (threadIdx.x >= 64 && threadIdx.x < 128) { // We need 2 warps here
        const uint idx0 = clamp(int((threadIdx.x-64)/7), 0, 6);
        const uint idx1 = threadIdx.x%6;
        VertexNeighbouringTetrahedrons_S[idx0][idx1][0] = VertexNeighbouringTetrahedrons[idx0][idx1][0];
        VertexNeighbouringTetrahedrons_S[idx0][idx1][1] = VertexNeighbouringTetrahedrons[idx0][idx1][1];
        VertexNeighbouringTetrahedrons_S[idx0][idx1][2] = VertexNeighbouringTetrahedrons[idx0][idx1][2];
        VertexNeighbouringTetrahedrons_S[idx0][idx1][3] = VertexNeighbouringTetrahedrons[idx0][idx1][3];
    }

    // Use vertex index offset to shared memory
    // Use warps #4 and #5
    //LoadTetrahedronEdgeVertexIdxOffsetToSharedMemory();
    if (threadIdx.x >= 128 && threadIdx.x < 192) {
        const uint idx0 = clamp(int((threadIdx.x-128)/6), 0, 5);
        const uint idx1 = threadIdx.x%6;
        TetrahedronEdgeVertexIdxOffset_S[idx0][idx1][0] = TetrahedronEdgeVertexIdxOffset[idx0][idx1][0];
        TetrahedronEdgeVertexIdxOffset_S[idx0][idx1][1] = TetrahedronEdgeVertexIdxOffset[idx0][idx1][1];
        TetrahedronEdgeVertexIdxOffset_S[idx0][idx1][2] = TetrahedronEdgeVertexIdxOffset[idx0][idx1][2];
        TetrahedronEdgeVertexIdxOffset_S[idx0][idx1][3] = TetrahedronEdgeVertexIdxOffset[idx0][idx1][3];
    }

    // Use first warp of every block to load vertex data for 6 threads
    // from global device memory to shared memory

    __shared__ int VertIdxGlobal_S[32];
    __shared__ int VertIdxLocal_S[32];
    __shared__ uint CellIdx_S[32];
    __shared__ uint3 CellOrg_S[32];

    if (threadIdx.x < 32) {
        //uint vtxIdx = clamp(32*blockIdx.x+threadIdx.x, uint(0), activeVertexCnt-1);
        uint vtxIdx = 32*blockIdx.x+threadIdx.x;

        if (vtxIdx < activeVertexCnt) {
            //VertIdxGlobal_S[threadIdx.x] = static_cast<int>(vertexMap_D[vtxIdx]); <-- this is actually slower
            VertIdxGlobal_S[threadIdx.x] = static_cast<int>(vertexMap_D[32*blockIdx.x+threadIdx.x]);
            CellIdx_S[threadIdx.x]       = VertIdxGlobal_S[threadIdx.x]/7;
            VertIdxLocal_S[threadIdx.x]  = VertIdxGlobal_S[threadIdx.x] - CellIdx_S[threadIdx.x]*7;
            CellOrg_S[threadIdx.x]       = ::GetGridCoordsByCellIdx(cubeMap_D[CellIdx_S[threadIdx.x]]);
        }
    }

    __syncthreads(); // Sync to make sure all writing operations are done!

    if (activeVertexIdx >= activeVertexCnt) {
        return;
    }

    // Load vertex data for current adjacent tetrahedron to registers
    const uint sharedMemoryIdx = threadIdx.x/6;
    int vertIdx = VertIdxGlobal_S[sharedMemoryIdx];
    int v       = VertIdxLocal_S[sharedMemoryIdx];
    uint cubeId = CellIdx_S[sharedMemoryIdx];
    uint3 cellOrg = CellOrg_S[sharedMemoryIdx];

    if (cellOrg.x >= gridSize_D.x-2) return;
    if (cellOrg.y >= gridSize_D.y-2) return;
    if (cellOrg.z >= gridSize_D.z-2) return;
    if (cellOrg.x <= 0) return;
    if (cellOrg.y <= 0) return;
    if (cellOrg.z <= 0) return;

    //--- From here on everything depends on 'i' -----------------------------//

    unsigned char terahedronFlagsTmp;
    unsigned char connectionFlags;
    uint ownEdgeIdx;

    // From here on stuff that depends on 'i'
    if (VertexNeighbouringTetrahedrons_S[v][i][0] == 99) return;

    // Get origin of the cell containing the adjacent tetrahedron
    int3 cellOrgTemp = make_int3(
            cellOrg.x + VertexNeighbouringTetrahedrons_S[v][i][0],
            cellOrg.y + VertexNeighbouringTetrahedrons_S[v][i][1],
            cellOrg.z + VertexNeighbouringTetrahedrons_S[v][i][2]);

    // Get tetrahedron flags of the adjacent tetrahedron
    terahedronFlagsTmp = TetrahedronFlags_D(
            make_uint3(cellOrgTemp.x, cellOrgTemp.y, cellOrgTemp.z),
            VertexNeighbouringTetrahedrons_S[v][i][3], isoval, volume_D);

    // Edge index of this vertex in the adjacent tetrahedron
    ownEdgeIdx = VertexNeighbouringTetrahedronsOwnEdgeIdx[v][i];

    // Look up connections
    connectionFlags = TetrahedronEdgeConnections[terahedronFlagsTmp][ownEdgeIdx];

    // Loop through possible connections
    for(int j = 0; j < 6; ++j) {
        if (connectionFlags & (1 << static_cast<unsigned char>(j))) {
            int3 tempOffs = make_int3(
                    TetrahedronEdgeVertexIdxOffset_S[i][j][0],
                    TetrahedronEdgeVertexIdxOffset_S[i][j][1],
                    TetrahedronEdgeVertexIdxOffset_S[i][j][2]);
            int3 neighbourVertexIdxOffs = cellOrgTemp + tempOffs;
            int vertexIdx =
                    static_cast<int>(cubeMapInv_D[GetCellIdxByGridCoords(neighbourVertexIdxOffs)]*7) +
                    TetrahedronEdgeVertexIdxOffset_S[i][j][3];
            vertexNeighbours_D[18*activeVertexIdx+TetrahedronToNeighbourIdx[v][i][j]] = vertexMapInv_D[vertexIdx];
            //vertexNeighbours_D[18*activeVertexIdx+TetrahedronToNeighbourIdx[v][i][j]] = vertexIdx;
        }


    }

}

extern "C"
hipError_t ComputeVertexConnectivity(int *vertexNeighbours_D, uint *vertexStates_D,
        uint *vertexMap_D, uint *vertexMapInv_D, uint *cubeMap_D,
        uint *cubeMapInv_D, uint *cubeStates_D, uint activeVertexCnt, float *volume_D,
        float isoval) {

    const uint blockSize = 192; // == 6 * 32, 32 = warpsize

#ifdef USE_TIMER
    float dt_ms;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipEventRecord(event1, 0);
#endif

//    hipDeviceProp_t devProp;
//    hipGetDeviceProperties(&devProp, 0);
//    printf("Shared memory per block %u bytes\n", devProp.sharedMemPerBlock);
//    printf("Number of blocks %u\n", Grid(activeVertexCnt*6, blockSize).x);

//    CheckForCudaErrorSync();

    ComputeVertexConnectivity_D <<< Grid(activeVertexCnt*6, blockSize), blockSize >>> (
            vertexNeighbours_D,
            vertexStates_D,
            activeVertexCnt,
            vertexMap_D,
            vertexMapInv_D,
            cubeMap_D,
            cubeMapInv_D,
            cubeStates_D,
            volume_D,
            isoval);

//    CheckForCudaErrorSync();

#ifdef USE_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("CUDA time for 'ComputeVertexConnectivity_D' :          %.10f sec\n",
            dt_ms/1000.0);
#endif

    return hipGetLastError();
}

__constant__ __device__ uint TriangleCrossPoductVtxIdx[3][2] = {
        {1, 2}, {1, 0}, {0, 2}
};

__global__
void ComputeVertexNormals_D(
        float *dataBuffer_D,
        uint *vertexMap_D,
        uint *vertexMapInv_D,
        uint *cubeMap_D,
        uint *cubeMapInv_D,
        float *volume_D,
        float isoval,
        uint activeVertexCnt,
        uint arrDataOffsPos,
        uint arrDataOffsNormals,
        uint arrDataSize){

    // Get thread index
    uint activeVertexIdx = GetThreadIdx();
    LoadCubeOffsetsToSharedMemory();
    LoadTetrahedronsInACubeToSharedMemory();
    __syncthreads();
    if (activeVertexIdx >= activeVertexCnt) {
        return;
    }
    int vertIdx = static_cast<int>(vertexMap_D[activeVertexIdx]);

    int v = vertIdx%7; // We have 7 vertices per cube
    int cubeId = vertIdx/7;
    unsigned char terahedronFlagsTmp;
    uint ownEdgeIdx;

    // Omit border cells
    uint3 cellOrgUint = GetGridCoordsByCellIdx(cubeMap_D[cubeId]);
    int3 cellOrg = make_int3(cellOrgUint.x, cellOrgUint.y, cellOrgUint.z);
    if (cellOrg.x >= gridSize_D.x-2) return;
    if (cellOrg.y >= gridSize_D.y-2) return;
    if (cellOrg.z >= gridSize_D.z-2) return;
    if (cellOrg.x <= 0) return;
    if (cellOrg.y <= 0) return;
    if (cellOrg.z <= 0) return;


    float3 normal = make_float3(0.0, 0.0, 0.0);
    float3 pos = make_float3(
            dataBuffer_D[arrDataSize*activeVertexIdx+arrDataOffsPos+0],
            dataBuffer_D[arrDataSize*activeVertexIdx+arrDataOffsPos+1],
            dataBuffer_D[arrDataSize*activeVertexIdx+arrDataOffsPos+2]);

    int maxIdx = 0;

    // Loop through all adjacent tetrahedrons
    for(int tetrahedronIdx = 0; tetrahedronIdx < 6; ++tetrahedronIdx) {

        // Check whether =? 99 (is tetrahedron neighbour)
        if(VertexNeighbouringTetrahedrons[v][tetrahedronIdx][0] == 99) continue;

        // Get origin of the cell containing the adjacent tetrahedron
        int3 cellOrgTemp = make_int3(
                cellOrg.x + VertexNeighbouringTetrahedrons[v][tetrahedronIdx][0],
                cellOrg.y + VertexNeighbouringTetrahedrons[v][tetrahedronIdx][1],
                cellOrg.z + VertexNeighbouringTetrahedrons[v][tetrahedronIdx][2]);


        // Get tetrahedron flags of the adjacent tetrahedron
        terahedronFlagsTmp = TetrahedronFlags_D(
                make_uint3(cellOrgTemp.x, cellOrgTemp.y, cellOrgTemp.z),
                VertexNeighbouringTetrahedrons[v][tetrahedronIdx][3], isoval, volume_D);


        // Edge index of this vertex in the adjacent tetrahedron
        ownEdgeIdx = VertexNeighbouringTetrahedronsOwnEdgeIdx[v][tetrahedronIdx];

        // Loop both possible triangles
        for(int triangleIdx = 0; triangleIdx < 2; ++triangleIdx) {
            if(tetrahedronTriangles[terahedronFlagsTmp][3*triangleIdx+0] < 0) {
                continue;
            }

            for(int vtx = 0; vtx < 3; vtx++) {

                if(tetrahedronTriangles[terahedronFlagsTmp][triangleIdx*3 + vtx] == ownEdgeIdx) {

                    uint edgeIdx0 = tetrahedronTriangles[terahedronFlagsTmp][triangleIdx*3 + (vtx+1)%3];
                    uint edgeIdx1 = tetrahedronTriangles[terahedronFlagsTmp][triangleIdx*3 + (vtx+2)%3];



                    int3 cubeIdx0 = make_int3(
                            TetrahedronEdgeVertexIdxOffset[tetrahedronIdx][edgeIdx0][0],
                            TetrahedronEdgeVertexIdxOffset[tetrahedronIdx][edgeIdx0][1],
                            TetrahedronEdgeVertexIdxOffset[tetrahedronIdx][edgeIdx0][2]) + cellOrgTemp;

                    int3 cubeIdx1 = make_int3(
                            TetrahedronEdgeVertexIdxOffset[tetrahedronIdx][edgeIdx1][0],
                            TetrahedronEdgeVertexIdxOffset[tetrahedronIdx][edgeIdx1][1],
                            TetrahedronEdgeVertexIdxOffset[tetrahedronIdx][edgeIdx1][2]) + cellOrgTemp;

                    int vertexIdx0 = vertexMapInv_D[cubeMapInv_D[GetCellIdxByGridCoords(cubeIdx0)]*7 +
                                                    TetrahedronEdgeVertexIdxOffset[tetrahedronIdx][edgeIdx0][3]];

                    int vertexIdx1 = vertexMapInv_D[cubeMapInv_D[GetCellIdxByGridCoords(cubeIdx1)]*7 +
                                                    TetrahedronEdgeVertexIdxOffset[tetrahedronIdx][edgeIdx1][3]];

                    maxIdx = max(maxIdx, max(vertexIdx0, vertexIdx1));

                    float3 pos0, pos1;
                    pos0 = make_float3(
                            dataBuffer_D[arrDataSize*vertexIdx0+arrDataOffsPos+0],
                            dataBuffer_D[arrDataSize*vertexIdx0+arrDataOffsPos+1],
                            dataBuffer_D[arrDataSize*vertexIdx0+arrDataOffsPos+2]);
                    pos1 = make_float3(
                            dataBuffer_D[arrDataSize*vertexIdx1+arrDataOffsPos+0],
                            dataBuffer_D[arrDataSize*vertexIdx1+arrDataOffsPos+1],
                            dataBuffer_D[arrDataSize*vertexIdx1+arrDataOffsPos+2]);
                    float3 vec0 = safeNormalize(pos0 - pos);
                    float3 vec1 = safeNormalize(pos1 - pos);

                    normal += cross(vec0, vec1);
                }
            }
        }
    }

    normal = safeNormalize(normal);
    dataBuffer_D[arrDataSize*activeVertexIdx+arrDataOffsNormals+0] = normal.x;
    dataBuffer_D[arrDataSize*activeVertexIdx+arrDataOffsNormals+1] = normal.y;
    dataBuffer_D[arrDataSize*activeVertexIdx+arrDataOffsNormals+2] = normal.z;
}

extern "C"
hipError_t ComputeVertexNormals(
        float *dataBuffer_D,
        uint *vertexMap_D,
        uint *vertexMapInv_D,
        uint *cubeMap_D,
        uint *cubeMapInv_D,
        float *volume_D,
        float isoval,
        uint activeVertexCnt,
        uint arrDataOffsPos,
        uint arrDataOffsNormals,
        uint arrDataSize) {

#ifdef USE_TIMER
    float dt_ms;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipEventRecord(event1, 0);
#endif

    ComputeVertexNormals_D <<< Grid(activeVertexCnt, 256), 256 >>> (
            dataBuffer_D,
            vertexMap_D,
            vertexMapInv_D,
            cubeMap_D,
            cubeMapInv_D,
            volume_D,
            isoval,
            activeVertexCnt,
            arrDataOffsPos,
            arrDataOffsNormals,
            arrDataSize);


#ifdef USE_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("CUDA time for 'ComputeVertexNormals_D' :               %.10f sec\n",
            dt_ms/1000.0);
#endif

    //return hipGetLastError();
    return hipDeviceSynchronize(); // Appearently this is necessary
}

__global__
void ComputeVertexTexCoords_D(float *dataBuff_D,
        float volMinX, float volMinY, float volMinZ,
        float volMaxX, float volMaxY, float volMaxZ,
        uint activeVertexCnt,
        uint arrDataOffsPos,
        uint arrDataOffsTexCoords,
        uint arrDataSize) {

    // Get thread index
    uint activeVertexIdx = GetThreadIdx();
    if (activeVertexIdx >= activeVertexCnt) {
        return;
    }

    dataBuff_D[arrDataSize*activeVertexIdx+arrDataOffsTexCoords+0] =
            (dataBuff_D[arrDataSize*activeVertexIdx+arrDataOffsPos+0] - volMinX) / (volMaxX-volMinX);
    dataBuff_D[arrDataSize*activeVertexIdx+arrDataOffsTexCoords+1] =
            (dataBuff_D[arrDataSize*activeVertexIdx+arrDataOffsPos+1] - volMinY) / (volMaxY-volMinY);
    dataBuff_D[arrDataSize*activeVertexIdx+arrDataOffsTexCoords+2] =
            (dataBuff_D[arrDataSize*activeVertexIdx+arrDataOffsPos+2] - volMinZ) / (volMaxZ-volMinZ);
}

extern "C"
hipError_t ComputeVertexTexCoords(float *dataBuff_D,
        float volMinX, float volMinY, float volMinZ,
        float volMaxX, float volMaxY, float volMaxZ,
        uint activeVertexCnt,
        uint arrDataOffsPos,
        uint arrDataOffsTexCoords,
        uint arrDataSize) {


#ifdef USE_TIMER
    float dt_ms;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipEventRecord(event1, 0);
#endif

    // Calc difference field using the iso value
    ComputeVertexTexCoords_D <<< Grid(activeVertexCnt, 256), 256 >>> (
            dataBuff_D,
            volMinX, volMinY, volMinZ,
            volMaxX, volMaxY, volMaxZ,
                    activeVertexCnt,
                    arrDataOffsPos,
                    arrDataOffsTexCoords,
                    arrDataSize);

#ifdef USE_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("CUDA time for 'ComputeVertexTexCoords_D' :             %.10f sec\n",
            dt_ms/1000.0);
#endif

    return hipGetLastError();
}


/**
 * Setup mapping function from active vertex list to vertex list (based on
 * active cells).
 *
 * @param[out] vertexMap_D       Mapping from active vertex' list to global
 *                               vertex list
 * @param[out] vertexMapInv_D    Mapping from global vertex list to active
 *                               vertex' list
 * @param[in]  vertexIdxOffs_D   Offsets for vertex indices
 * @param[in]  activeVertexIdx_D Active vertex flags, '1' if vertex is active
 * @param[in]  vtxCount          The number of vertices
 */
// TODO vertexIdxOffs_D is pointless
__global__ void CalcVertexMapTODO_D(
        uint* vertexMap_D,
        uint* vertexMapInv_D,
        uint* vertexIdxOffs_D,
        uint* activeVertexIdx_D,
        uint vtxCount) {

    const uint vtxIndex = ::GetThreadIdx();
    if (vtxIndex >= vtxCount) {
        return;
    }

    if(activeVertexIdx_D[vtxIndex] != 0) {
        // Map from active vertices list to vtx idx
        vertexMap_D[vertexIdxOffs_D[vtxIndex]] = vtxIndex;
        vertexMapInv_D[vtxIndex] = vertexIdxOffs_D[vtxIndex];
    }
}

__global__
void TranslatePos_D(float *vertexData_D, uint vertexDataStride,
        uint vertexDataOffsPos, float3 translation, uint vertexCnt) {

    const uint idx = GetThreadIdx();
    if (idx >= vertexCnt) {
        return;
    }
    const uint vertexDataIdx = vertexDataStride*idx+vertexDataOffsPos;

    vertexData_D[vertexDataIdx+0] += translation.x;
    vertexData_D[vertexDataIdx+1] += translation.y;
    vertexData_D[vertexDataIdx+2] += translation.z;
}

extern "C"
hipError_t TranslatePos(float *vertexData_D, uint vertexDataStride,
        uint vertexDataOffsPos, float3 translation, uint vertexCnt) {

#ifdef USE_TIMER
    //Create events
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    //Record events around kernel launch
    hipEventRecord(event1, 0); //where 0 is the default stream
#endif

    // Initialize triangle index array
    TranslatePos_D <<< Grid(vertexCnt, 256), 256 >>> (vertexData_D,
            vertexDataStride, vertexDataOffsPos, translation, vertexCnt);

#ifdef USE_TIMER
    hipEventRecord(event2, 0);
    // Synchronize
    hipEventSynchronize(event1); //optional
    hipEventSynchronize(event2); //wait for the event to be executed!
    // Calculate time
    float dt_ms;
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("Kernel execution time 'Translateos_D': %f sec\n", dt_ms/1000.0f);
#endif

    return hipGetLastError();
}

__global__
void RotatePos_D(float *vertexData_D, uint vertexDataStride,
        uint vertexDataOffsPos, float *rotation_D, uint vertexCnt) {

    const uint idx = GetThreadIdx();
    if (idx >= vertexCnt) {
        return;
    }

    const uint vertexDataIdx = vertexDataStride*idx+vertexDataOffsPos;

    float xtemp, ytemp, ztemp;
    xtemp = rotation_D[0] * vertexData_D[vertexDataIdx+0] +
            rotation_D[3] * vertexData_D[vertexDataIdx+1] +
            rotation_D[6] * vertexData_D[vertexDataIdx+2];
    ytemp = rotation_D[1] * vertexData_D[vertexDataIdx+0] +
            rotation_D[4] * vertexData_D[vertexDataIdx+1] +
            rotation_D[7] * vertexData_D[vertexDataIdx+2];
    ztemp = rotation_D[2] * vertexData_D[vertexDataIdx+0] +
            rotation_D[5] * vertexData_D[vertexDataIdx+1] +
            rotation_D[8] * vertexData_D[vertexDataIdx+2];
    vertexData_D[vertexDataIdx+0] = xtemp;
    vertexData_D[vertexDataIdx+1] = ytemp;
    vertexData_D[vertexDataIdx+2] = ztemp;
}

extern "C"
hipError_t RotatePos(float *vertexData_D, uint vertexDataStride,
        uint vertexDataOffsPos, float *rotation_D, uint vertexCnt) {

#ifdef USE_TIMER
    //Create events
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    //Record events around kernel launch
    hipEventRecord(event1, 0); //where 0 is the default stream
#endif

    // Initialize triangle index array
    RotatePos_D <<< Grid(vertexCnt, 256), 256 >>> (vertexData_D,
            vertexDataStride, vertexDataOffsPos, rotation_D, vertexCnt);
#ifdef USE_TIMER
    hipEventRecord(event2, 0);
    // Synchronize
    hipEventSynchronize(event1); //optional
    hipEventSynchronize(event2); //wait for the event to be executed!
    // Calculate time
    float dt_ms;
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("Kernel execution time 'RotatePos_D': %f sec\n", dt_ms/1000.0f);
#endif

    return hipGetLastError();
}


/**
 * Identifies neighbouring vertices for all vertices and stores them as vertex
 * indices. -1 indicates invalid neighbours
 *
 * @param[out] vertexNeighbours_D The vertex connectivity information
 * @param[in]  activeVertexIdx_D  Array with vertex activity flags
 * @param[in]  activeVertexCnt    The number of vertices
 * @param[in]  vertexMap_D        Vertex mapping from active to global index
 * @param[in]  vertexMapInv_D     Inverse mapping to vertexMap_D
 * @param[in]  cubeMap_D          Mapping from active cells's list to global
 *                                cell list
 * @param[in]  cubeMapInv_D       Inverse mapping to cubeMap_D
 * @param[in]  cubeStates_D       Flags active cells
 * @param[in]  volume_D           The volume the isosurface is extracted from
 * @param[in]  isoval             The isovalue that defines the isosurface
 */
__global__
void ComputeVertexConnectivityTODO_D(
        int *vertexNeighbours_D,
        uint *activeVertexIdx_D,
        uint activeVertexCnt,
        uint *vertexMap_D,
        uint *vertexMapInv_D,
        uint *cubeMap_D,
        uint *cubeMapInv_D,
        uint *cubeStates_D,
        float *volume_D,
        float isoval) {

    // Get different indices
    uint idx = ::GetThreadIdx();
    uint activeVertexIdx = idx/6;
    uint i = idx - __umul24(activeVertexIdx, 6); // == idx%6;

    /* 1. Load LUTs to shared memory */

    // Note: We have 6 warps per block (each warp with 32 threads)

    // Load cube vertex offsets into shared memory
    // Use warp #0
//    LoadCubeOffsetsToSharedMemory();
    if (threadIdx.x < 32) {
        const uint idx0 = clamp(int(threadIdx.x), 0, 7);
        cubeVertexOffsets_S[idx0][0] = cubeVertexOffsets[idx0][0];
        cubeVertexOffsets_S[idx0][1] = cubeVertexOffsets[idx0][1];
        cubeVertexOffsets_S[idx0][2] = cubeVertexOffsets[idx0][2];
    }

    // Load cube vertex offsets into shared memory
    // Use warp #1
    //LoadTetrahedronsInACubeToSharedMemory();
    if (threadIdx.x >= 32 && threadIdx.x < 64) {
        const uint idx0 = clamp(int(threadIdx.x-32), 0, 7);
        tetrahedronsInACube_S[idx0][0] = tetrahedronsInACube[idx0][0];
        tetrahedronsInACube_S[idx0][1] = tetrahedronsInACube[idx0][1];
        tetrahedronsInACube_S[idx0][2] = tetrahedronsInACube[idx0][2];
        tetrahedronsInACube_S[idx0][3] = tetrahedronsInACube[idx0][3];
    }

    // Load vertex indices of neighbouring tetrahedrons to shared memory
    // Use warps #2 and #3
    //LoadVertexNeighbouringTetrahedronsToSharedMemory();
    if (threadIdx.x >= 64 && threadIdx.x < 128) { // We need 2 warps here
        const uint idx0 = clamp(int((threadIdx.x-64)/7), 0, 6);
        const uint idx1 = threadIdx.x%6;
        VertexNeighbouringTetrahedrons_S[idx0][idx1][0] = VertexNeighbouringTetrahedrons[idx0][idx1][0];
        VertexNeighbouringTetrahedrons_S[idx0][idx1][1] = VertexNeighbouringTetrahedrons[idx0][idx1][1];
        VertexNeighbouringTetrahedrons_S[idx0][idx1][2] = VertexNeighbouringTetrahedrons[idx0][idx1][2];
        VertexNeighbouringTetrahedrons_S[idx0][idx1][3] = VertexNeighbouringTetrahedrons[idx0][idx1][3];
    }

    // Use vertex index offset to shared memory
    // Use warps #4 and #5
    //LoadTetrahedronEdgeVertexIdxOffsetToSharedMemory();
    if (threadIdx.x >= 128 && threadIdx.x < 192) {
        const uint idx0 = clamp(int((threadIdx.x-128)/6), 0, 5);
        const uint idx1 = threadIdx.x%6;
        TetrahedronEdgeVertexIdxOffset_S[idx0][idx1][0] = TetrahedronEdgeVertexIdxOffset[idx0][idx1][0];
        TetrahedronEdgeVertexIdxOffset_S[idx0][idx1][1] = TetrahedronEdgeVertexIdxOffset[idx0][idx1][1];
        TetrahedronEdgeVertexIdxOffset_S[idx0][idx1][2] = TetrahedronEdgeVertexIdxOffset[idx0][idx1][2];
        TetrahedronEdgeVertexIdxOffset_S[idx0][idx1][3] = TetrahedronEdgeVertexIdxOffset[idx0][idx1][3];
    }

    // Use first warp of every block to load vertex data for 6 threads
    // from global device memory to shared memory

    __shared__ int VertIdxGlobal_S[32];
    __shared__ int VertIdxLocal_S[32];
    __shared__ uint CellIdx_S[32];
    __shared__ uint3 CellOrg_S[32];

    if (threadIdx.x < 32) {
        //uint vtxIdx = clamp(32*blockIdx.x+threadIdx.x, uint(0), activeVertexCnt-1);
        uint vtxIdx = 32*blockIdx.x+threadIdx.x;

        if (vtxIdx < activeVertexCnt) {
            //VertIdxGlobal_S[threadIdx.x] = static_cast<int>(vertexMap_D[vtxIdx]); <-- this is actually slower
            VertIdxGlobal_S[threadIdx.x] = static_cast<int>(vertexMap_D[32*blockIdx.x+threadIdx.x]);
            CellIdx_S[threadIdx.x]       = VertIdxGlobal_S[threadIdx.x]/7;
            VertIdxLocal_S[threadIdx.x]  = VertIdxGlobal_S[threadIdx.x] - CellIdx_S[threadIdx.x]*7;
            CellOrg_S[threadIdx.x]       = ::GetGridCoordsByCellIdx(cubeMap_D[CellIdx_S[threadIdx.x]]);
        }
    }

    __syncthreads(); // Sync to make sure all writing operations are done!

    if (activeVertexIdx >= activeVertexCnt) {
        return;
    }

    // Load vertex data for current adjacent tetrahedron to registers
    const uint sharedMemoryIdx = threadIdx.x/6;
    int vertIdx = VertIdxGlobal_S[sharedMemoryIdx];
    int v       = VertIdxLocal_S[sharedMemoryIdx];
    uint cubeId = CellIdx_S[sharedMemoryIdx];
    uint3 cellOrg = CellOrg_S[sharedMemoryIdx];

    if (cellOrg.x >= gridSize_D.x-2) return;
    if (cellOrg.y >= gridSize_D.y-2) return;
    if (cellOrg.z >= gridSize_D.z-2) return;
    if (cellOrg.x <= 0) return;
    if (cellOrg.y <= 0) return;
    if (cellOrg.z <= 0) return;

    //--- From here on everything depends on 'i' -----------------------------//

    unsigned char terahedronFlagsTmp;
    unsigned char connectionFlags;
    uint ownEdgeIdx;

    // From here on stuff that depends on 'i'
    if (VertexNeighbouringTetrahedrons_S[v][i][0] == 99) return;

    // Get origin of the cell containing the adjacent tetrahedron
    int3 cellOrgTemp = make_int3(
            cellOrg.x + VertexNeighbouringTetrahedrons_S[v][i][0],
            cellOrg.y + VertexNeighbouringTetrahedrons_S[v][i][1],
            cellOrg.z + VertexNeighbouringTetrahedrons_S[v][i][2]);

    // Get tetrahedron flags of the adjacent tetrahedron
    terahedronFlagsTmp = TetrahedronFlags_D(
            make_uint3(cellOrgTemp.x, cellOrgTemp.y, cellOrgTemp.z),
            VertexNeighbouringTetrahedrons_S[v][i][3], isoval, volume_D);

    // Edge index of this vertex in the adjacent tetrahedron
    ownEdgeIdx = VertexNeighbouringTetrahedronsOwnEdgeIdx[v][i];

    // Look up connections
    connectionFlags = TetrahedronEdgeConnections[terahedronFlagsTmp][ownEdgeIdx];

    // Loop through possible connections
    for(int j = 0; j < 6; ++j) {
        if (connectionFlags & (1 << static_cast<unsigned char>(j))) {
            int3 tempOffs = make_int3(
                    TetrahedronEdgeVertexIdxOffset_S[i][j][0],
                    TetrahedronEdgeVertexIdxOffset_S[i][j][1],
                    TetrahedronEdgeVertexIdxOffset_S[i][j][2]);
            int3 neighbourVertexIdxOffs = cellOrgTemp + tempOffs;
            int vertexIdx =
                    static_cast<int>(cubeMapInv_D[GetCellIdxByGridCoords(neighbourVertexIdxOffs)]*7) +
                    TetrahedronEdgeVertexIdxOffset_S[i][j][3];
            vertexNeighbours_D[18*activeVertexIdx+TetrahedronToNeighbourIdx[v][i][j]] = vertexMapInv_D[vertexIdx];
            //vertexNeighbours_D[18*activeVertexIdx+TetrahedronToNeighbourIdx[v][i][j]] = vertexIdx;
        }


    }

}


__global__ void FlagGridCells_D(
        uint* activeCellFlag_D,  // Output
        float *volume_D,         // Input
        float isoval,            // Input
        uint cubeCount) {        // Input

    const uint cellIdx = ::GetThreadIdx();

    if (cellIdx >= cubeCount) {
        return;
    }

    const uint3 cellOrg = ::GetGridCoordsByCellIdx(cellIdx);

    // Put into registers by the compiler since the array size is constant
    const float cellVertexOffsets[8][3] = {
        {0, 0, 0},
        {1, 0, 0},
        {1, 1, 0},
        {0, 1, 0},
        {0, 0, 1},
        {1, 0, 1},
        {1, 1, 1},
        {0, 1, 1}
    };

    // Add vertex states of a cube (0: inactive, 1: active)
    float volSample = ::SampleFieldAt_D<float>(cellOrg, volume_D);
    unsigned char cubeFlags = static_cast<uint>(volSample <= isoval);

#pragma unroll
    for (int v = 1; v < 8; ++v) {
        const uint3 pos = make_uint3(
                cellOrg.x + cellVertexOffsets[v][0],
                cellOrg.y + cellVertexOffsets[v][1],
                cellOrg.z + cellVertexOffsets[v][2]);
        volSample = ::SampleFieldAt_D<float>(pos, volume_D);
        cubeFlags |= static_cast<uint>(volSample <= isoval) * (1 << v);
    }

    // Reduce vertex states to one cube state
    activeCellFlag_D[cellIdx] = min(cubeFlags % 255, 1);
}


/*
 * GPUSurfaceMT::GPUSurfaceMT
 */
GPUSurfaceMT::GPUSurfaceMT() : AbstractGPUSurface() , neighboursReady(false) {
}


/*
 * GPUSurfaceMT::GPUSurfaceMT
 */
GPUSurfaceMT::GPUSurfaceMT(const GPUSurfaceMT& other) : AbstractGPUSurface(other) {

    // Copy GPU memory

    CudaSafeCall(this->cubeStates_D.Validate(other.cubeStates_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->cubeStates_D.Peek(),
            other.cubeStates_D.PeekConst(),
            this->cubeStates_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->cubeOffsets_D.Validate(other.cubeOffsets_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->cubeOffsets_D.Peek(),
            other.cubeOffsets_D.PeekConst(),
            this->cubeOffsets_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->cubeMap_D.Validate(other.cubeMap_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->cubeMap_D.Peek(),
            other.cubeMap_D.PeekConst(),
            this->cubeMap_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->cubeMapInv_D.Validate(other.cubeMapInv_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->cubeMapInv_D.Peek(),
            other.cubeMapInv_D.PeekConst(),
            this->cubeMapInv_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->vertexStates_D.Validate(other.vertexStates_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->vertexStates_D.Peek(),
            other.vertexStates_D.PeekConst(),
            this->vertexStates_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->activeVertexPos_D.Validate(other.activeVertexPos_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->activeVertexPos_D.Peek(),
            other.activeVertexPos_D.PeekConst(),
            this->activeVertexPos_D.GetCount()*sizeof(float3),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->vertexIdxOffs_D.Validate(other.vertexIdxOffs_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->vertexIdxOffs_D.Peek(),
            other.vertexIdxOffs_D.PeekConst(),
            this->vertexIdxOffs_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->vertexMap_D.Validate(other.vertexMap_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->vertexMap_D.Peek(),
            other.vertexMap_D.PeekConst(),
            this->vertexMap_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->vertexMapInv_D.Validate(other.vertexMapInv_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->vertexMapInv_D.Peek(),
            other.vertexMapInv_D.PeekConst(),
            this->vertexMapInv_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->vertexNeighbours_D.Validate(other.vertexNeighbours_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->vertexNeighbours_D.Peek(),
            other.vertexNeighbours_D.PeekConst(),
            this->vertexNeighbours_D.GetCount()*sizeof(int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->verticesPerTetrahedron_D.Validate(other.verticesPerTetrahedron_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->verticesPerTetrahedron_D.Peek(),
            other.verticesPerTetrahedron_D.PeekConst(),
            this->verticesPerTetrahedron_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->tetrahedronVertexOffsets_D.Validate(other.tetrahedronVertexOffsets_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->tetrahedronVertexOffsets_D.Peek(),
            other.tetrahedronVertexOffsets_D.PeekConst(),
            this->tetrahedronVertexOffsets_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->triangleCamDistance_D.Validate(other.triangleCamDistance_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->triangleCamDistance_D.Peek(),
            other.triangleCamDistance_D.PeekConst(),
            this->triangleCamDistance_D.GetCount()*sizeof(float),
            hipMemcpyDeviceToDevice));

    // The number of active cells
    this->activeCellCnt = other.activeCellCnt;

    // Check whether neighbors have been computed
    this->neighboursReady = other.neighboursReady;
}


/*
 * GPUSurfaceMT::~GPUSurfaceMT
 */
GPUSurfaceMT::~GPUSurfaceMT() {
}


/*
 * DeformableGPUSurfaceMT::ComputeVertexPositions
 */
bool GPUSurfaceMT::ComputeVertexPositions(
        float *volume_D,
        int3 volDim,
        float3 volOrg,
        float3 volDelta,
        float isovalue) {

//    printf("Compute vertex positions\n");

    using vislib::sys::Log;

    size_t gridCellCnt = (volDim.x-1)*(volDim.y-1)*(volDim.z-1);


    /* Init grid parameters */

    // Init constant device params
    if (!initGridParams(volDim, volOrg, volDelta)) {
        Log::DefaultLog.WriteMsg(Log::LEVEL_ERROR,
                "%s: could not init constant device params",
                this->ClassName());
        return false;
    }

    ::CheckForCudaErrorSync();

//    if (!CudaSafeCall(InitVolume(
//            make_uint3(volDim.x, volDim.y, volDim.z),
//            volOrg,
//            volDelta))) {
//        return false;
//    }
//
//    if (!CudaSafeCall(InitVolume_surface_generation(
//            make_uint3(volDim.x, volDim.y, volDim.z),
//            volOrg,
//            volDelta))) {
//        return false;
//    }

//    printf("ComputeVertexPositions: Grid dims %u %u %u\n", volDim.x, volDim.y, volDim.z);
//    printf("ComputeVertexPositions: cell count %u\n", gridCellCnt);

//    // DEBUG Print volume
//    HostArr<float> volume;
//    volume.Validate(volDim.x*volDim.y*volDim.z);
//    hipMemcpy(volume.Peek(),volume_D,sizeof(float)*volDim.x*volDim.y*volDim.z, hipMemcpyDeviceToHost);
//    for (int i = 0; i < volDim.x*volDim.y*volDim.z;++i) {
//        printf("volume %i %f\n", i, volume.Peek()[i]);
//    }
//    volume.Release();
//    // End DEBUG


    /* Find active grid cells */

    if (!CudaSafeCall(this->cubeStates_D.Validate(gridCellCnt))) {
        return false;
    }
    if (!CudaSafeCall(this->cubeOffsets_D.Validate(gridCellCnt))) {
        return false;
    }
    if (!CudaSafeCall(this->cubeStates_D.Set(0x00))) {
        return false;
    }
    if (!CudaSafeCall(this->cubeOffsets_D.Set(0x00))) {
        return false;
    }

    ::CheckForCudaErrorSync();

#ifdef USE_TIMER
    hipEvent_t event1, event2;
    float dt_ms;
    //Create events
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    //Record events around kernel launch
    hipEventRecord(event1, 0); //where 0 is the default stream
#endif

    // Classify cells
    FlagGridCells_D <<< Grid(gridCellCnt, 256), 256 >>> (
            cubeStates_D.Peek(),
            volume_D,
            isovalue,
            gridCellCnt);

    ::CheckForCudaErrorSync();

#ifdef USE_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1); //optional
    hipEventSynchronize(event2); //wait for the event to be executed!
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("CUDA time for 'FlagGridCells_D' :                      %.10f sec\n",
            dt_ms/1000.0);
#endif

    thrust::exclusive_scan(
            thrust::device_ptr<uint>(this->cubeStates_D.Peek()),
            thrust::device_ptr<uint>(this->cubeStates_D.Peek() + gridCellCnt),
            thrust::device_ptr<uint>(this->cubeOffsets_D.Peek()));

//    // DEBUG Print Cube states and offsets
//    HostArr<unsigned int> cubeStates;
//    HostArr<unsigned int> cubeOffsets;
//    cubeStates.Validate(gridCellCnt);
//    cubeOffsets.Validate(gridCellCnt);
//    this->cubeStates_D.CopyToHost(cubeStates.Peek());
//    this->cubeOffsets_D.CopyToHost(cubeOffsets.Peek());
//    for (int i = 0; i < gridCellCnt; ++i) {
//        printf ("Cell %i: state %u, offs %u\n", i, cubeStates.Peek()[i],
//                cubeOffsets.Peek()[i]);
//    }
//    // END DEBUG


    /* Get number of active grid cells */

    this->activeCellCnt =
            this->cubeStates_D.GetAt(gridCellCnt-1) +
            this->cubeOffsets_D.GetAt(gridCellCnt-1);
    if (!CheckForCudaError()) {
        return false;
    }


//    printf("ComputeVertexPositions: active cell count %u\n", activeCellCnt); // DEBUG
//    printf("Reduction %f\n", 1.0 - static_cast<float>(activeCellCnt)/
//            static_cast<float>(gridCellCnt)); // DEBUG


    /* Prepare cube map */

    if (!CudaSafeCall(this->cubeMapInv_D.Validate(gridCellCnt))) {
        return false;
    }
    if (!CudaSafeCall(this->cubeMapInv_D.Set(0x00))) {
        return false;
    }
    if (!CudaSafeCall(this->cubeMap_D.Validate(this->activeCellCnt))) {
        return false;
    }
    if (!CudaSafeCall(CalcCubeMap(
            this->cubeMap_D.Peek(),
            this->cubeMapInv_D.Peek(),
            this->cubeOffsets_D.Peek(),
            this->cubeStates_D.Peek(),
            gridCellCnt))) {
        return false;
    }

//
//    // DEBUG Cube map
//    HostArr<unsigned int> cubeMap;
//    HostArr<unsigned int> cubeMapInv;
//    cubeMap.Validate(activeCellCnt);
//    cubeMapInv.Validate(gridCellCnt);
//    cubeMapInv_D.CopyToHost(cubeMapInv.Peek());
//    cubeMap_D.CopyToHost(cubeMap.Peek());
//    for (int i = 0; i < gridCellCnt; ++i) {
//        printf ("Cell %i: cubeMapInv %u\n", i, cubeMapInv.Peek()[i]);
//    }
//    for (int i = 0; i < activeCellCnt; ++i) {
//        printf ("Cell %i: cubeMap %u\n", i, cubeMap.Peek()[i]);
//    }
//    // END DEBUG


    /* Get vertex positions */

    if (!CudaSafeCall(this->vertexStates_D.Validate(7*this->activeCellCnt))) {
        return false;
    }
    if (!CudaSafeCall(this->activeVertexPos_D.Validate(7*this->activeCellCnt))) {
        return false;
    }
    if (!CudaSafeCall(this->vertexIdxOffs_D.Validate(7*this->activeCellCnt))) {
        return false;
    }
    if (!CudaSafeCall(this->vertexStates_D.Set(0x00))) {
        return false;
    }
    if (!CudaSafeCall(this->activeVertexPos_D.Set(0x00))) {
        return false;
    }
    if (!CudaSafeCall(this->vertexIdxOffs_D.Set(0x00))) {
        return false;
    }
    if (!CudaSafeCall(CalcVertexPositions(
            this->vertexStates_D.Peek(),
            this->activeVertexPos_D.Peek(),
            this->vertexIdxOffs_D.Peek(),
            this->cubeMap_D.Peek(),
            this->activeCellCnt,
            isovalue,
            volume_D))) {
        return false;
    }

//    // DEBUG Print active vertex positions
//    HostArr<float3> activeVertexPos;
//    HostArr<unsigned int> vertexStates;
//    HostArr<unsigned int> vertexIdxOffsets;
//    activeVertexPos.Validate(7*this->activeCellCnt);
//    vertexIdxOffsets.Validate(7*this->activeCellCnt);
//    vertexStates.Validate(7*this->activeCellCnt);
//    hipMemcpy(vertexStates.Peek(), this->vertexStates_D.Peek(), 7*this->activeCellCnt*sizeof(unsigned int),
//            hipMemcpyDeviceToHost);
//    hipMemcpy(activeVertexPos.Peek(), this->activeVertexPos_D.Peek(), 7*this->activeCellCnt*sizeof(float3),
//            hipMemcpyDeviceToHost);
//    hipMemcpy(vertexIdxOffsets.Peek(), this->vertexIdxOffs_D.Peek(), 7*this->activeCellCnt*sizeof(unsigned int),
//            hipMemcpyDeviceToHost);
//    for (int i = 0; i < 7*this->activeCellCnt; ++i) {
//        printf("#%i: active vertexPos %f %f %f (state = %u)\n", i,
//                activeVertexPos.Peek()[i].x,
//                activeVertexPos.Peek()[i].y,
//                activeVertexPos.Peek()[i].z,
//                vertexStates.Peek()[i]);
//    }
//
////    for (int i = 0; i < 7*this->activeCellCnt; ++i) {
////        printf("#%i: vertex index offset %u (state %u)\n",i,
////                vertexIdxOffsets.Peek()[i],
////                vertexStates.Peek()[i]);
////    }
//    // END DEBUG


    /* Get number of active vertices */

    this->vertexCnt =
            this->vertexStates_D.GetAt(7*this->activeCellCnt-1) +
            this->vertexIdxOffs_D.GetAt(7*this->activeCellCnt-1);
    if (!CheckForCudaError()) {
        return false;
    }

//    printf("ComputeVertexPositions: vertex Cnt %u\n", this->vertexCnt);

    /* Create vertex buffer object and register with CUDA */

    // Create empty vbo to hold vertex data for the surface
    if (!this->InitVertexDataVBO(this->vertexCnt)) {
        return false;
    }

    // Register memory with CUDA
    if (!CudaSafeCall(hipGraphicsGLRegisterBuffer(
            &this->vertexDataResource, this->vboVtxData,
            cudaGraphicsMapFlagsNone))) {
        return false;
    }

    // Get mapped pointer to the vbo
    float *vboPt;
    size_t vboSize;
    if (!CudaSafeCall(hipGraphicsMapResources(1, &this->vertexDataResource, 0))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsResourceGetMappedPointer(
            reinterpret_cast<void**>(&vboPt), // The mapped pointer
            &vboSize,             // The size of the accessible data
            this->vertexDataResource))) {                   // The mapped resource
        // Unmap CUDA graphics resource
        if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->vertexDataResource))) {
            return false;
        }
        return false;
    }

    // Init with zeros
    if (!CudaSafeCall(hipMemset(vboPt, 0, vboSize))) {
        // Unmap CUDA graphics resource
        if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->vertexDataResource))) {
            return false;
        }
        if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
            return false;
        }
        return false;
    }

//    printf("Got VBO of size %u\n", vboSize);


    /* Compact list of vertex positions (keep only active vertices) */

    if (!CudaSafeCall(CompactActiveVertexPositions(
            vboPt,
            this->vertexStates_D.Peek(),
            this->vertexIdxOffs_D.Peek(),
            this->activeVertexPos_D.Peek(),
            this->activeCellCnt,
            this->vertexDataOffsPos,  // Array data byte offset
            this->vertexDataStride    // Array data element size
            ))) {
        // Unmap CUDA graphics resource
        if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->vertexDataResource))) {
            return false;
        }
        if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
            return false;
        }
        return false;
    }

//    // DEBUG Print vertex positions
//    HostArr<float> vertexPos;
//    vertexPos.Validate(this->vertexCnt*this->vertexDataStride);
//    hipMemcpy(vertexPos.Peek(), vboPt, this->vertexCnt*this->vertexDataStride*sizeof(float),
//            hipMemcpyDeviceToHost);
//    for (int i = 0; i < this->vertexCnt; ++i) {
//        printf("#%i: vertexPos %f %f %f\n", i, vertexPos.Peek()[9*i+0],
//                vertexPos.Peek()[9*i+1], vertexPos.Peek()[9*i+2]);
//    }
//    // END DEBUG

    // Unmap CUDA graphics resource
    if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->vertexDataResource))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
        return false;
    }

    return true;
}


/*
 * DeformableGPUSurfaceMT::computeTriangles
 */
bool GPUSurfaceMT::ComputeTriangles(
        float *volume_D,
        int3 volDim,
        float3 volOrg,
        float3 volDelta,
        float isovalue) {

    using namespace vislib::sys;

//    printf("VERTEX COUNT %u\n", this->vertexCnt);

    /* Init grid parameters */

    // Init constant device params
    if (!initGridParams(volDim, volOrg, volDelta)) {
        Log::DefaultLog.WriteMsg(Log::LEVEL_ERROR,
                "%s: could not init constant device params",
                this->ClassName());
        return false;
    }

    ::CheckForCudaErrorSync();

//    // DEBUG print grid data
//    int3 gridSize;
//    float3 gridOrg;
//    float3 gridDelta;
//    hipMemcpyFromSymbol(&gridSize, HIP_SYMBOL(gridSize_D), sizeof(int3),0,hipMemcpyDeviceToHost);
//    hipMemcpyFromSymbol(&gridDelta, HIP_SYMBOL(gridDelta_D), sizeof(float3),0,hipMemcpyDeviceToHost);
//    hipMemcpyFromSymbol(&gridOrg, HIP_SYMBOL(gridOrg_D), sizeof(float3),0,hipMemcpyDeviceToHost);
//    printf("HOST gridSize  %i %i %i\n",gridSize.x,gridSize.y,gridSize.z);
//    printf("HOST gridOrg   %f %f %f\n",gridOrg.x,gridOrg.y,gridOrg.z);
//    printf("HOST gridDelta %f %f %f\n",gridDelta.x,gridDelta.y,gridDelta.z);
//    printf("SHOULD BE gridSize: %i %i %i\n", volDim.x,volDim.y,volDim.z);
//    printf("SHOULD BE gridDelta: %f %f %f\n", volDelta.x,volDelta.y,volDelta.z);
//    printf("SHOULD BE gridOrg: %f %f %f\n", volOrg.x,volOrg.y,volOrg.z);
//    // END DEBUG

    if (!this->vertexDataReady) { // We need vertex data to generate triangles
        return false;
    }

    size_t triangleVtxCnt;

    /* Calc vertex index map */

    if (!CudaSafeCall(this->vertexMap_D.Validate(this->vertexCnt))) {
        return false;
    }
    if (!CudaSafeCall(this->vertexMapInv_D.Validate(7*this->activeCellCnt))) {
        return false;
    }
    if (!CudaSafeCall(this->vertexMapInv_D.Set(0xff))) {
        return false;
    }

    ::CheckForCudaErrorSync();

    CalcVertexMapTODO_D <<< Grid(7*this->activeCellCnt, 256), 256 >>> ( // TODO rename
            this->vertexMap_D.Peek(),
            this->vertexMapInv_D.Peek(),
            this->vertexIdxOffs_D.Peek(),
            this->vertexStates_D.Peek(),
             7*this->activeCellCnt);

    // DEBUG Print vertex map
    HostArr<unsigned int> vertexMap;
    vertexMap.Validate(this->vertexCnt);
    vertexMap_D.CopyToHost(vertexMap.Peek());
//    for (int i = 0; i < this->vertexMap_D.GetCount(); ++i) {
//        printf("Vertex mapping %i: %u\n", i, vertexMap.Peek()[i]);
//    }
    // END DEBUG

    // DEBUG Print vertex map
    HostArr<unsigned int> vertexMapInv;
    vertexMapInv.Validate(this->vertexMapInv_D.GetCount());
    vertexMapInv_D.CopyToHost(vertexMapInv.Peek());
//    for (int i = 0; i < this->vertexMapInv_D.GetCount(); ++i) {
//        printf("Inverse Vertex mapping %i: %u\n", i, vertexMapInv.Peek()[i]);
//    }
//    for (int i = 0; i < this->vertexCnt; ++i) {
//        printf("MAPPING %i: %u\n", i, vertexMapInv.Peek()[vertexMap.Peek()[i]]);
//    }
    // END DEBUG

    ::CheckForCudaErrorSync();


    /* Flag tetrahedrons */

    if (!CudaSafeCall(this->verticesPerTetrahedron_D.Validate(6*this->activeCellCnt))) return false;
    if (!CudaSafeCall(FlagTetrahedrons(
            this->verticesPerTetrahedron_D.Peek(),
            this->cubeMap_D.Peek(),
            isovalue,
            this->activeCellCnt,
            volume_D))) {
        return false;
    }

    ::CheckForCudaErrorSync();


    /* Scan tetrahedrons */

    if (!CudaSafeCall(this->tetrahedronVertexOffsets_D.Validate(6*this->activeCellCnt))) return false;
    if (!CudaSafeCall(GetTetrahedronVertexOffsets(
            this->tetrahedronVertexOffsets_D.Peek(),
            this->verticesPerTetrahedron_D.Peek(),
            this->activeCellCnt*6))) {
        return false;
    }

    ::CheckForCudaErrorSync();


    /* Get triangle vertex count */

    triangleVtxCnt =
            this->tetrahedronVertexOffsets_D.GetAt(activeCellCnt*6-1) +
            this->verticesPerTetrahedron_D.GetAt(activeCellCnt*6-1);
    if (!CheckForCudaError()) {
        return false;
    }

    ::CheckForCudaErrorSync();

//    printf("Triangle cnt %u\n", triangleVtxCnt);

    this->triangleCnt = triangleVtxCnt/3;

    /* Create vertex buffer object and register with CUDA */

    // Create empty vbo to hold the triangle indices
    if (!this->InitTriangleIdxVBO(this->triangleCnt)) {
        return false;
    }

    // Register memory with CUDA
    if (!CudaSafeCall(hipGraphicsGLRegisterBuffer(
            &this->triangleIdxResource,
            this->vboTriangleIdx,
            cudaGraphicsMapFlagsNone))) {
        return false;
    }

    // Get mapped pointer to the vbo
    unsigned int *vboTriangleIdxPt;
    size_t vboTriangleIdxSize;
    if (!CudaSafeCall(hipGraphicsMapResources(1, &this->triangleIdxResource, 0))) {

        // Unmap CUDA graphics resource
        if (!CudaSafeCall(hipGraphicsUnregisterResource(this->triangleIdxResource))) {
            return false;
        }
        return false;
    }

    if (!CudaSafeCall(hipGraphicsResourceGetMappedPointer(
            reinterpret_cast<void**>(&vboTriangleIdxPt), // The mapped pointer
            &vboTriangleIdxSize,             // The size of the accessible data
            this->triangleIdxResource))) {                   // The mapped resource

        // Unmap CUDA graphics resource
        if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->triangleIdxResource))) {
            return false;
        }
        if (!CudaSafeCall(hipGraphicsUnregisterResource(this->triangleIdxResource))) {
            return false;
        }
        return false;
    }


    /* Generate triangles */

    if (!CudaSafeCall(hipMemset(vboTriangleIdxPt, 0x00, vboTriangleIdxSize))) {

        // Unmap CUDA graphics resource
        if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->triangleIdxResource))) {
            return false;
        }
        if (!CudaSafeCall(hipGraphicsUnregisterResource(this->triangleIdxResource))) {
            return false;
        }
        return false;
    }

    ::CheckForCudaErrorSync();

    if (!CudaSafeCall(GetTrianglesIdx(
            this->tetrahedronVertexOffsets_D.Peek(),
            this->cubeMap_D.Peek(),
            this->cubeMapInv_D.Peek(),
            isovalue,
            this->activeCellCnt*6,
            this->activeCellCnt,
            vboTriangleIdxPt,
            this->vertexMapInv_D.Peek(),
            volume_D))) {

        // Unmap CUDA graphics resource
        if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->triangleIdxResource))) {
            return false;
        }
        if (!CudaSafeCall(hipGraphicsUnregisterResource(this->triangleIdxResource))) {
            return false;
        }
        return false;
    }

    ::CheckForCudaErrorSync();

//    // DEBUG Printf triangle indices
//    HostArr<unsigned int> triangleIdx;
//    triangleIdx.Validate(this->triangleCnt*3);
//    hipMemcpy(triangleIdx.Peek(), vboTriangleIdxPt, sizeof(unsigned int)*this->triangleCnt*3, hipMemcpyDeviceToHost);
//    for (int i = 0; i < this->triangleCnt; ++i) {
////    for (int i = 45124; i < 45125; ++i) {
////                if ((vertexMapInv.Peek()[triangleIdx.Peek()[i*3+0]] > this->vertexCnt) ||
////                        (vertexMapInv.Peek()[triangleIdx.Peek()[i*3+1]] > this->vertexCnt)||
////                        (vertexMapInv.Peek()[triangleIdx.Peek()[i*3+2]] > this->vertexCnt)) {
//        if ((triangleIdx.Peek()[i*3+0] > this->vertexCnt) ||
//                (triangleIdx.Peek()[i*3+1] > this->vertexCnt)||
//                (triangleIdx.Peek()[i*3+2] > this->vertexCnt)) {
////            printf("Gen: vertex index idx %i: %u %u %u (vtxCnt %u)\n", i,
////                    vertexMapInv.Peek()[triangleIdx.Peek()[i*3+0]],
////                    vertexMapInv.Peek()[triangleIdx.Peek()[i*3+1]],
////                    vertexMapInv.Peek()[triangleIdx.Peek()[i*3+2]],
////                    this->vertexCnt);
//
//            printf("Gen: vertex index idx %i: %u %u %u (vtxCnt %u)\n", i,
//                    triangleIdx.Peek()[i*3+0],
//                    triangleIdx.Peek()[i*3+1],
//                    triangleIdx.Peek()[i*3+2],
//                    this->vertexCnt);
//        }
//    }
//    // END DEBUG

    // Unmap CUDA graphics resource
    if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->triangleIdxResource))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsUnregisterResource(this->triangleIdxResource))) {
        return false;
    }
    return true;
}


/*
 * GPUSurfaceMT::computeVertexNormals
 */
bool GPUSurfaceMT::ComputeNormals(
        float *volume_D,
        int3 volDim,
        float3 volOrg,
        float3 volDelta,
        float isovalue) {

    using vislib::sys::Log;

    if (!this->triangleIdxReady) { // We need the triangles mesh info
        Log::DefaultLog.WriteMsg(Log::LEVEL_ERROR,
                "%s: triangles not computed",
                this->ClassName());
        return false;
    }

    /* Init grid parameters */

    // Init constant device params
    if (!initGridParams(volDim, volOrg, volDelta)) {
        Log::DefaultLog.WriteMsg(Log::LEVEL_ERROR,
                "%s: could not init constant device params",
                this->ClassName());
        return false;
    }

//    if (!CudaSafeCall(InitVolume_surface_generation(
//            make_uint3(volDim.x, volDim.y, volDim.z),
//            volOrg,
//            volDelta))) {
//
//        Log::DefaultLog.WriteMsg(Log::LEVEL_ERROR,
//                "%s: could not init device constants",
//                this->ClassName());
//
//        return false;
//    }

//        printf("Init volume surface generation\n");
//        printf("grid size  %u %u %u\n", volDim[0], volDim[1], volDim[2]);
//        printf("grid org   %f %f %f\n", volWSOrg[0], volWSOrg[1], volWSOrg[2]);
//        printf("grid delta %f %f %f\n", volWSDelta[0], volWSDelta[1], volWSDelta[2]);

    // Register memory with CUDA
    if (!CudaSafeCall(hipGraphicsGLRegisterBuffer(
            &this->vertexDataResource, this->vboVtxData,
            cudaGraphicsMapFlagsNone))) {

        Log::DefaultLog.WriteMsg(Log::LEVEL_ERROR,
                "%s: could not register vertex buffer",
                this->ClassName());

        return false;
    }

    // Get mapped pointer to the vbo
    float *vboPt;
    size_t vboSize;
    if (!CudaSafeCall(hipGraphicsMapResources(1, &this->vertexDataResource, 0))) {
        Log::DefaultLog.WriteMsg(Log::LEVEL_ERROR,
                "%s: could not map resources",
                this->ClassName());
        return false;
    }
    if (!CudaSafeCall(hipGraphicsResourceGetMappedPointer(
            reinterpret_cast<void**>(&vboPt), // The mapped pointer
            &vboSize,             // The size of the accessible data
            this->vertexDataResource))) {
        Log::DefaultLog.WriteMsg(Log::LEVEL_ERROR,
                "%s: could not acquire mapped pointer",
                this->ClassName());
        return false;
    }



//    int cnt = 0;
//    // DEBUG Print vertex map
//    HostArr<unsigned int> vertexMap;
//    vertexMap.Validate(this->vertexCnt);
//    if (!CudaSafeCall(vertexMap_D.CopyToHost(vertexMap.Peek()))) {
//        return false;
//    }
//    for (int i = 0; i < this->vertexMap_D.GetCount(); ++i) {
//        printf("Vertex mapping %i: %u\n", i, vertexMap.Peek()[i]);
////        cnt += vertexMap.Peek()[i];
//    }
//    // END DEBUG
//
//    // DEBUG Print vertex map
//    HostArr<unsigned int> vertexMapInv;
//    vertexMapInv.Validate(this->vertexMapInv_D.GetCount());
//    if (!CudaSafeCall(vertexMapInv_D.CopyToHost(vertexMapInv.Peek()))) {
//        return false;
//    }
//    for (int i = 0; i < this->vertexMapInv_D.GetCount(); ++i) {
//        printf("Inverse Vertex mapping %i: %u\n", i, vertexMapInv.Peek()[i]);
////        cnt += vertexMapInv.Peek()[i];
//    }
//    // END DEBUG

//    printf("active vertex count %u\n", this->vertexCnt);
//    printf("active cube count %u\n", this->activeCellCnt);
//    printf("normals vbo %u\n", vboSize);
//    printf("vertexMap size %u\n", this->vertexMap_D.GetCount());
//    printf("vertexMapInv size %u\n", this->vertexMapInv_D.GetCount());
//    printf("cubeMap_D size %u\n", this->cubeMap_D.GetCount());
//    printf("cubeMapInv_D size %u\n", this->cubeMapInv_D.GetCount());

//        // DEBUG Print buffer content
//        HostArr<float> vertexBuffer;
//        vertexBuffer.Validate(this->vertexDataStride*this->vertexCnt*sizeof(float));
//        if (!CudaSafeCall(hipMemcpy(vertexBuffer.Peek(), vboPt,
//                this->vertexDataStride*this->vertexCnt*sizeof(float), hipMemcpyDeviceToHost))) {
//            return false;
//        }
//        for (int i = 0; i < this->vertexCnt; ++i) {
//    //        if (uint(abs(vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+0]))>= this->vertexCnt) {
//            printf("%i: pos %f %f %f, normal %f %f %f, texcoord %f %f %f\n", i,
//                    vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsPos+0],
//                    vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsPos+1],
//                    vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsPos+2],
//                    vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+0],
//                    vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+1],
//                    vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+2],
//                    vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsTexCoord+0],
//                    vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsTexCoord+1],
//                    vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsTexCoord+2],
//                    this->vertexCnt);
//    //        }
//        }
//        vertexBuffer.Release();
//        // end DEBUG

    if (!CudaSafeCall(ComputeVertexNormals(
            vboPt,
            this->vertexMap_D.Peek(),
            this->vertexMapInv_D.Peek(),
            this->cubeMap_D.Peek(),
            this->cubeMapInv_D.Peek(),
            volume_D,
            isovalue,
            this->vertexCnt,
            this->vertexDataOffsPos,
            this->vertexDataOffsNormal,
            this->vertexDataStride))) {

        return false;
    }

//    // DEBUG Print normals
//    HostArr<float> vertexBuffer;
//    vertexBuffer.Validate(this->vertexDataStride*this->vertexCnt*sizeof(float));
//    if (!CudaSafeCall(hipMemcpy(vertexBuffer.Peek(), vboPt,
//            this->vertexDataStride*this->vertexCnt*sizeof(float), hipMemcpyDeviceToHost))) {
//        return false;
//    }
//    for (int i = 0; i < this->vertexCnt; i+=3) {
////        if (uint(abs(vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+0]))>= this->vertexCnt) {
//                    printf("%i: pos %f %f %f, normal %f %f %f, texcoord %f %f %f\n", i,
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsPos+0],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsPos+1],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsPos+2],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+0],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+1],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsNormal+2],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsTexCoord+0],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsTexCoord+1],
//                            vertexBuffer.Peek()[this->vertexDataStride*i+this->vertexDataOffsTexCoord+2],
//                            this->vertexCnt);
////        }
//    }
//    vertexBuffer.Release();
//    // end DEBUG

    // Unmap CUDA graphics resource
    if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->vertexDataResource))) {
        Log::DefaultLog.WriteMsg(Log::LEVEL_ERROR,
                "%s: could not unmap resources",
                this->ClassName());
        return false;
    }
    if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
        Log::DefaultLog.WriteMsg(Log::LEVEL_ERROR,
                "%s: could not unregister buffers",
                this->ClassName());
        return false;
    }

    return true;
}


/*
 * DeformableGPUSurfaceMT::ComputeTexCoords
 */
bool GPUSurfaceMT::ComputeTexCoords(float minCoords[3], float maxCoords[3]) {
    if (!this->triangleIdxReady) { // We need the triangles mesh info
        return false;
    }

    // Register memory with CUDA
    if (!CudaSafeCall(hipGraphicsGLRegisterBuffer(
            &this->vertexDataResource, this->vboVtxData,
            cudaGraphicsMapFlagsNone))) {
        return false;
    }

    // Get mapped pointer to the vbo
    float *vboPt;
    size_t vboSize;
    CudaSafeCall(hipGraphicsMapResources(1, &this->vertexDataResource, 0));
    CudaSafeCall(hipGraphicsResourceGetMappedPointer(
            reinterpret_cast<void**>(&vboPt), // The mapped pointer
            &vboSize,             // The size of the accessible data
            this->vertexDataResource));                   // The mapped resource

    if (!CudaSafeCall(ComputeVertexTexCoords(
            vboPt,
            minCoords[0],
            minCoords[1],
            minCoords[2],
            maxCoords[0],
            maxCoords[1],
            maxCoords[2],
            this->vertexCnt,
            this->vertexDataOffsPos,
            this->vertexDataOffsTexCoord,
            this->vertexDataStride))) {

        // Unmap CUDA graphics resource
        if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->vertexDataResource))) {
            return false;
        }
        if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
            return false;
        }

        return false;
    }

    // Unmap CUDA graphics resource
    if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->vertexDataResource))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
        return false;
    }

    return true;
}


/*
 * GPUSurfaceMT::Rotate
 */
bool GPUSurfaceMT::Rotate(float rotMat[9]) {
    CudaDevArr<float> rotate_D;

    // Register memory with CUDA
    if (!CudaSafeCall(hipGraphicsGLRegisterBuffer(
            &this->vertexDataResource, this->vboVtxData,
            cudaGraphicsMapFlagsNone))) {
        return false;
    }

    // Get mapped pointer to the vbo
    float *vboPt;
    size_t vboSize;
    if (!CudaSafeCall(hipGraphicsMapResources(1, &this->vertexDataResource, 0))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsResourceGetMappedPointer(
            reinterpret_cast<void**>(&vboPt), // The mapped pointer
            &vboSize,                         // The size of the accessible data
            this->vertexDataResource))) {     // The mapped resource
        return false;
    }

    // Rotate for best fit
    rotate_D.Validate(9);
    if (!CudaSafeCall(hipMemcpy((void *)rotate_D.Peek(), &rotMat[0],
            9*sizeof(float), hipMemcpyHostToDevice))) {
        return false;
    }
    if (!CudaSafeCall(RotatePos(
            vboPt,
            this->vertexDataStride,
            this->vertexDataOffsPos,
            rotate_D.Peek(),
            vertexCnt))) {
        return false;
    }

    // Clean up
    rotate_D.Release();

    // Unmap CUDA graphics resource
    if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->vertexDataResource))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
        return false;
    }

    return true;
}


/*
 * GPUSurfaceMT::SortTrianglesByCamDist
 */
bool GPUSurfaceMT::SortTrianglesByCamDist(float camPos[3]) {

    if (!CudaSafeCall(this->triangleCamDistance_D.Validate(triangleCnt))) {
        return false;
    }

    // Register memory with CUDA
    if (!CudaSafeCall(hipGraphicsGLRegisterBuffer(
            &this->vertexDataResource, this->vboVtxData,
            cudaGraphicsMapFlagsNone))) {
        return false;
    }
    // Register memory with CUDA
    if (!CudaSafeCall(hipGraphicsGLRegisterBuffer(
            &this->triangleIdxResource,
            this->vboTriangleIdx,
            cudaGraphicsMapFlagsNone))) {
        return false;
    }

    // We need both cuda graphics resources to be mapped at the same time
    hipGraphicsResource *cudaToken[2];
    cudaToken[0] = this->vertexDataResource;
    cudaToken[1] = this->triangleIdxResource;
    if (!CudaSafeCall(hipGraphicsMapResources(2, cudaToken, 0))) {
        return false;
    }

    // Get mapped pointers to the vertex data and the triangle indices
    float *vboPt;
    uint *vboTriangleIdxPt;
    size_t vboSize, vboTriangleIdxSize;
    if (!CudaSafeCall(hipGraphicsResourceGetMappedPointer(
            reinterpret_cast<void**>(&vboPt), // The mapped pointer
            &vboSize,                         // The size of the accessible data
            cudaToken[0]))) {                 // The mapped resource
        return false;
    }
    if (!CudaSafeCall(hipGraphicsResourceGetMappedPointer(
            reinterpret_cast<void**>(&vboTriangleIdxPt), // The mapped pointer
            &vboTriangleIdxSize,              // The size of the accessible data
            cudaToken[1]))) {                 // The mapped resource
        return false;
    }

    if (!CudaSafeCall(SortTrianglesByCamDistance(
            vboPt,
            this->vertexDataStride,
            this->vertexDataOffsPos,
            make_float3(camPos[0], camPos[1], camPos[2]),
            vboTriangleIdxPt,
            this->triangleCnt,
            this->triangleCamDistance_D.Peek()))) {

        // Unmap CUDA graphics resource
        if (!CudaSafeCall(hipGraphicsUnmapResources(2, cudaToken, 0))) {
            return false;
        }
        if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
            return false;
        }
        if (!CudaSafeCall(hipGraphicsUnregisterResource(this->triangleIdxResource))) {
            return false;
        }

        return false;
    }

    // Unmap CUDA graphics resource
    if (!CudaSafeCall(hipGraphicsUnmapResources(2, cudaToken, 0))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsUnregisterResource(this->triangleIdxResource))) {
        return false;
    }


    return true;
}


/*
 * GPUSurfaceMT::Translate
 */
bool GPUSurfaceMT::Translate(float transVec[3]) {

    // Register memory with CUDA
    if (!CudaSafeCall(hipGraphicsGLRegisterBuffer(
            &this->vertexDataResource, this->vboVtxData,
            cudaGraphicsMapFlagsNone))) {
        return false;
    }

    // Get mapped pointer to the vbo
    float *vboPt;
    size_t vboSize;
    if (!CudaSafeCall(hipGraphicsMapResources(1, &this->vertexDataResource, 0))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsResourceGetMappedPointer(
            reinterpret_cast<void**>(&vboPt), // The mapped pointer
            &vboSize,                         // The size of the accessible data
            this->vertexDataResource))) {     // The mapped resource
        return false;
    }

    // Move vertex positions to origin (with respect to centroid)
    if (!CudaSafeCall(TranslatePos(
            vboPt,
            this->vertexDataStride,
            this->vertexDataOffsPos,
            make_float3(transVec[0], transVec[0], transVec[0]),
            this->vertexCnt))) {
        return false;
    }

    // Unmap CUDA graphics resource
    if (!CudaSafeCall(hipGraphicsUnmapResources(1, &this->vertexDataResource, 0))) {
        return false;
    }
    if (!CudaSafeCall(hipGraphicsUnregisterResource(this->vertexDataResource))) {
        return false;
    }

    return true;
}


/*
 * GPUSurfaceMT::operator=
 */
GPUSurfaceMT& GPUSurfaceMT::operator=(const GPUSurfaceMT &rhs) {
    AbstractGPUSurface::operator=(rhs);

    // Copy GPU memory

    CudaSafeCall(this->cubeStates_D.Validate(rhs.cubeStates_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->cubeStates_D.Peek(),
            rhs.cubeStates_D.PeekConst(),
            this->cubeStates_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->cubeOffsets_D.Validate(rhs.cubeOffsets_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->cubeOffsets_D.Peek(),
            rhs.cubeOffsets_D.PeekConst(),
            this->cubeOffsets_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->cubeMap_D.Validate(rhs.cubeMap_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->cubeMap_D.Peek(),
            rhs.cubeMap_D.PeekConst(),
            this->cubeMap_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->cubeMapInv_D.Validate(rhs.cubeMapInv_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->cubeMapInv_D.Peek(),
            rhs.cubeMapInv_D.PeekConst(),
            this->cubeMapInv_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->vertexStates_D.Validate(rhs.vertexStates_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->vertexStates_D.Peek(),
            rhs.vertexStates_D.PeekConst(),
            this->vertexStates_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->activeVertexPos_D.Validate(rhs.activeVertexPos_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->activeVertexPos_D.Peek(),
            rhs.activeVertexPos_D.PeekConst(),
            this->activeVertexPos_D.GetCount()*sizeof(float3),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->vertexIdxOffs_D.Validate(rhs.vertexIdxOffs_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->vertexIdxOffs_D.Peek(),
            rhs.vertexIdxOffs_D.PeekConst(),
            this->vertexIdxOffs_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->vertexMap_D.Validate(rhs.vertexMap_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->vertexMap_D.Peek(),
            rhs.vertexMap_D.PeekConst(),
            this->vertexMap_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->vertexMapInv_D.Validate(rhs.vertexMapInv_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->vertexMapInv_D.Peek(),
            rhs.vertexMapInv_D.PeekConst(),
            this->vertexMapInv_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->vertexNeighbours_D.Validate(rhs.vertexNeighbours_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->vertexNeighbours_D.Peek(),
            rhs.vertexNeighbours_D.PeekConst(),
            this->vertexNeighbours_D.GetCount()*sizeof(int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->verticesPerTetrahedron_D.Validate(rhs.verticesPerTetrahedron_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->verticesPerTetrahedron_D.Peek(),
            rhs.verticesPerTetrahedron_D.PeekConst(),
            this->verticesPerTetrahedron_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

    CudaSafeCall(this->tetrahedronVertexOffsets_D.Validate(rhs.tetrahedronVertexOffsets_D.GetCount()));
    CudaSafeCall(hipMemcpy(
            this->tetrahedronVertexOffsets_D.Peek(),
            rhs.tetrahedronVertexOffsets_D.PeekConst(),
            this->tetrahedronVertexOffsets_D.GetCount()*sizeof(unsigned int),
            hipMemcpyDeviceToDevice));

//    CudaSafeCall(this->triangleCamDistance_D.Validate(rhs.triangleCamDistance_D.GetCount()));
//    CudaSafeCall(hipMemcpy(
//            this->triangleCamDistance_D.Peek(),
//            rhs.triangleCamDistance_D.PeekConst(),
//            this->triangleCamDistance_D.GetCount()*sizeof(float),
//            hipMemcpyDeviceToDevice));

    // The number of active cells
    this->activeCellCnt = rhs.activeCellCnt;

    /// Flag whether the neighbors have been computed
    this->neighboursReady = rhs.neighboursReady;

    return *this;

}


/*
 * GPUSurfaceMT::ComputeConnectivity
 */
bool GPUSurfaceMT::ComputeConnectivity(
        float *volume_D,
        int3 volDim,
        float3 volOrg,
        float3 volDelta,
        float isovalue) {

    CheckForCudaErrorSync();

    using namespace vislib::sys;
    /* Init grid parameters for all files */

    // Init constant device params
    if (!initGridParams(volDim, volOrg, volDelta)) {
        Log::DefaultLog.WriteMsg(Log::LEVEL_ERROR,
                "%s: could not init constant device params",
                this->ClassName());
        return false;
    }

//    if (!CudaSafeCall(InitVolume(
//            make_uint3(volDim.x, volDim.y, volDim.z),
//            volOrg,
//            volDelta))) {
//
//        Log::DefaultLog.WriteMsg(Log::LEVEL_INFO,
//                "%s: could not init device constants",
//                this->ClassName());
//        return false;
//    }
//
//    CheckForCudaErrorSync();
//
//    if (!CudaSafeCall(InitVolume_surface_generation(
//            make_uint3(volDim.x, volDim.y, volDim.z),
//            volOrg,
//            volDelta))) {
//        Log::DefaultLog.WriteMsg(Log::LEVEL_INFO,
//                "%s: could not init device constants",
//                this->ClassName());
//        return false;
//    }

    /* Compute neighbours */

    CheckForCudaErrorSync();

    if (!CudaSafeCall(vertexNeighbours_D.Validate(this->vertexCnt*18))) {
        Log::DefaultLog.WriteMsg(Log::LEVEL_INFO,
                "%s: could not allocate device memory",
                this->ClassName());
        return false;
    }
    CheckForCudaErrorSync();
    //if (!CudaSafeCall(vertexNeighbours_D.Set(-1))) {
    if (!CudaSafeCall(vertexNeighbours_D.Set(0xff))) {
        Log::DefaultLog.WriteMsg(Log::LEVEL_INFO,
                "%s: could not init device memory",
                this->ClassName());
        return false;
    }
    CheckForCudaErrorSync();

    const uint blockSize = 192; // == 6 * 32, 32 = warpsize

#ifdef USE_TIMER
    float dt_ms;
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipEventRecord(event1, 0);
#endif

//    hipDeviceProp_t devProp;
//    hipGetDeviceProperties(&devProp, 0);
//    printf("Shared memory per block %u bytes\n", devProp.sharedMemPerBlock);
//    printf("Number of blocks %u\n", Grid(activeVertexCnt*6, blockSize).x);

//    CheckForCudaErrorSync();

    ComputeVertexConnectivityTODO_D <<< Grid(this->vertexCnt*6, blockSize), blockSize >>> (
            this->vertexNeighbours_D.Peek(),
            this->vertexStates_D.Peek(),
            this->vertexCnt,
            this->vertexMap_D.Peek(),
            this->vertexMapInv_D.Peek(),
            this->cubeMap_D.Peek(),
            this->cubeMapInv_D.Peek(),
            this->cubeStates_D.Peek(),
            volume_D,
            isovalue);

//    CheckForCudaErrorSync();

#ifdef USE_TIMER
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("CUDA time for 'ComputeVertexConnectivity_D' :          %.10f sec\n",
            dt_ms/1000.0);
#endif

//    if (!CudaSafeCall(ComputeVertexConnectivity(
//            this->vertexNeighbours_D.Peek(),
//            this->vertexStates_D.Peek(),
//            this->vertexMap_D.Peek(),
//            this->vertexMapInv_D.Peek(),
//            this->cubeMap_D.Peek(),
//            this->cubeMapInv_D.Peek(),
//            this->cubeStates_D.Peek(),
//            this->vertexCnt,
//            volume_D,
//            isovalue))) {
//
////        // DEBUG Print neighbour indices
////        HostArr<int> vertexNeighbours;
////        vertexNeighbours.Validate(vertexNeighbours_D.GetCount());
////        vertexNeighbours_D.CopyToHost(vertexNeighbours.Peek());
////        for (int i = 0; i < vertexNeighbours_D.GetCount()/18; ++i) {
////            printf("Neighbours vtx #%i: ", i);
////            for (int j = 0; j < 18; ++j) {
////                printf("%i ", vertexNeighbours.Peek()[i*18+j]);
////            }
////            printf("\n");
////        }
////        // END DEBUG
//
//        Log::DefaultLog.WriteMsg(Log::LEVEL_INFO,
//                "%s: could not compute neighbors",
//                this->ClassName());
//
//        return false;
//    }

    CheckForCudaErrorSync();

    this->neighboursReady = true;
    return true;
}


/*
 * GPUSurfaceMT::Release
 */
void GPUSurfaceMT::Release() {
    CudaSafeCall(this->cubeStates_D.Release());
    CudaSafeCall(this->cubeOffsets_D.Release());
    CudaSafeCall(this->cubeMap_D.Release());
    CudaSafeCall(this->cubeMapInv_D.Release());
    CudaSafeCall(this->vertexStates_D.Release());
    CudaSafeCall(this->activeVertexPos_D.Release());
    CudaSafeCall(this->vertexIdxOffs_D.Release());
    CudaSafeCall(this->vertexMap_D.Release());
    CudaSafeCall(this->vertexMapInv_D.Release());
    CudaSafeCall(this->vertexNeighbours_D.Release());
    CudaSafeCall(this->verticesPerTetrahedron_D.Release());
    CudaSafeCall(this->tetrahedronVertexOffsets_D.Release());
    CudaSafeCall(this->triangleCamDistance_D.Release());
}

#endif // WITH_CUDA
